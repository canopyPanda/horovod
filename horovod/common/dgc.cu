#include "hip/hip_runtime.h"
// DGC host function implementations
// by Yuechao Pan
// for NVIDIA

// past compile with following command:
// nvcc -std=c++11 -c -o dgc.cu.o horovod_nvidia/horovod/common/dgc.cu.cc      \
   -x cu -Xcompiler -fPIC -dlink --expt-extended-lambda -gencode=arch=compute_70,code=\"sm_70,compute_70\"

//#pragma once

#include <string>
#include <algorithm>
#include <chrono>
#include <thread>
#include <hiprand/hiprand_kernel.h>
//#include <thrust/sort.h>
//#include <thrust/execution_policy.h>
#include <hipcub/hipcub.hpp>
#include <mpi.h>
#include "dgc.h"
#include "dgc_kernel.cu.cc"

namespace horovod {
namespace dgc {

#define GUARD_CU2(op_name, op)                                                 \
{                                                                              \
  /*do {*/                                                                     \
    retval = (op);                                                             \
    if (retval != hipSuccess) {                                               \
      std::string error_message = std::string(__FILE__) + std::string(":")     \
        + std::to_string(__LINE__) + std::string("(")                          \
        + std::string(op_name) + std::string(") failed: ")                     \
        + hipGetErrorString(retval);                                          \
      fprintf(stderr, "%s\n", error_message.c_str());                          \
      fflush(stderr);                                                          \
      return retval;                                                           \
    }                                                                          \
  /*} while (false);*/                                                         \
}

#define GUARD_CU(op)                                                           \
{                                                                              \
  /*do {*/                                                                     \
    retval = (op);                                                             \
    if (retval != hipSuccess) {                                               \
      std::string error_message = std::string(__FILE__) + std::string(":")     \
        + std::to_string(__LINE__) + std::string(" failed: ")                  \
        + hipGetErrorString(retval);                                          \
      fprintf(stderr, "%s\n", error_message.c_str());                          \
      fflush(stderr);                                                          \
      return retval;                                                           \
    }                                                                          \
  /*} while (false);*/                                                         \
}

#define GUARD_NCCL2(op_name, op)                                               \
{                                                                              \
  /*do {*/                                                                     \
    auto nccl_result = (op);                                                   \
    if (nccl_result != ncclSuccess) {                                          \
      std::string error_message = std::string(__FILE__) + std::string(":")     \
        + std::to_string(__LINE__) + std::string("(")                          \
        + std::string(op_name) + std::string(") failed: ")                     \
        + ncclGetErrorString(nccl_result);                                     \
      fprintf(stderr, "%s\n", error_message.c_str());                          \
      fflush(stderr);                                                          \
      return hipErrorUnknown;                                                 \
    }                                                                          \
  /*} while (false);*/                                                         \
}

#define GUARD_MPI2(op_name, op)                                                \
{                                                                              \
  auto mpi_result = (op);                                                      \
  if (mpi_result != MPI_SUCCESS) {                                             \
    char  error_string[MPI_MAX_ERROR_STRING + 1];                              \
    error_string[MPI_MAX_ERROR_STRING] = 0;                                    \
    int   error_length = 0;                                                    \
    MPI_Error_string(mpi_result, error_string, &error_length);                 \
    std::string error_message = std::string(__FILE__) + std::string(":")       \
      + std::to_string(__LINE__) + std::string("(")                            \
      + std::string(op_name) + std::string(") failed: ")                       \
      + std::string(error_string);                                             \
    fprintf(stderr, "%s\n", error_message.c_str());                            \
    fflush(stderr);                                                            \
    return hipErrorUnknown;                                                   \
  }                                                                            \
}

// ****************************
// Memory management
// ****************************

enum Malloc_t
{
  Default,
  Host,
  Managed,
  Raw,
};

template <typename T>
hipError_t Free(
  T* &ptr,
  Malloc_t malloc_type = Malloc_t::Default)
{
  hipError_t retval = hipSuccess;
  if (ptr == NULL)
    return retval;

  printf("Freeing @ %p\n", ptr);
  if (malloc_type == Host) {
    GUARD_CU2("hipHostFree",
      hipHostFree(ptr));
  } else if (malloc_type == Default || malloc_type == Managed) {
    GUARD_CU2("hipFree",
      hipFree(ptr));
  } else if (malloc_type == Raw)
    free(ptr);

  printf("Freed @ %p\n", ptr);
  ptr = NULL;
  return retval;
}

template <typename T>
hipError_t Malloc(
  T* &ptr,
  size_t target,
  Malloc_t malloc_type = Malloc_t::Default,
  unsigned int flags = hipMemAttachGlobal,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;

  size_t size = target * sizeof(T);
  printf("Allocating %ld x %ld bytes on %s\n", target, sizeof(T),
     malloc_type == Default ? "Default" :
    (malloc_type == Host    ? "Host" :
    (malloc_type == Managed ? "Managed" : "Raw")));

  if (malloc_type == Default) {
    GUARD_CU2("hipMalloc",
      hipMalloc(&ptr, size));
  } else if (malloc_type == Host) {
    GUARD_CU2("hipHostMalloc",
      hipHostMalloc(&ptr, size));
  } else if (malloc_type == Managed) {
    GUARD_CU2("hipMallocManaged",
      hipMallocManaged(&ptr, size, flags));
  } else if (malloc_type == Raw)
    ptr = (T*)malloc(size);

  printf("Allocated %ld x %ld bytes @ %p\n", target, sizeof(T), ptr);
  return retval;
}

template <typename T, typename SizeT>
hipError_t Memcpy(
  T* dest,
  T* src,
  SizeT num_elements,
  Malloc_t malloc_type = Malloc_t::Default,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;
  if (num_elements == 0)
    return retval;
  if (dest == NULL || src == NULL)
    return retval;

  if (malloc_type != Raw)
  {
    if (stream == 0)
    {
      retval = hipMemcpyAsync(dest, src, sizeof(T) * num_elements,
        hipMemcpyDefault, stream);
    } else {
      retval = hipMemcpy(dest, src, sizeof(T) * num_elements,
        hipMemcpyDefault);
    }
  } else {
    memcpy(dest, src, sizeof(T) * num_elements);
  }
  return retval;
}

template <typename T, typename SizeT>
hipError_t Memset(
  T* ptr,
  int value,
  SizeT num_elements,
  Malloc_t malloc_type = Malloc_t::Default,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;
  if (num_elements == 0 || ptr == NULL)
    return retval;

  if (malloc_type != Malloc_t::Raw)
  {
    if (stream == 0)
    {
      retval = hipMemset(ptr, value, num_elements * sizeof(T));
    } else {
      retval = hipMemsetAsync(ptr, value, num_elements * sizeof(T), stream);
    }
  } else {
    memset(ptr, value, num_elements * sizeof(T));
  }

  return retval;
}

template <typename T, typename SizeT>
hipError_t GarenteeAllocation(
  T*      &ptr,
  SizeT   &allocated,
  size_t   target,
  Malloc_t malloc_type = Malloc_t::Default,
  unsigned int flags = hipMemAttachGlobal,
  hipStream_t stream = 0,
  bool     keep_content = false,
  bool     init_to_zero = false)
{
  hipError_t retval = hipSuccess;
  if (allocated >= target)
    return retval;

  //if (stream != 0)
  //{
  //  GUARD_CU2("hipStreamSynchronize",
  //    hipStreamSynchronize(stream));
  //}
  if (!keep_content)
  {
    auto temp_ptr = ptr;
    GUARD_CU(Free<T> (temp_ptr, malloc_type));
    GUARD_CU(Malloc(ptr, target, malloc_type, flags));
    if (init_to_zero)
    {
      GUARD_CU(Memset(ptr, 0, target, malloc_type, stream));
    }
  } else {
    T* temp_ptr = NULL;
    GUARD_CU(Malloc(temp_ptr, target, malloc_type, flags));
    GUARD_CU(Memcpy(temp_ptr, ptr, allocated, malloc_type, stream));
    if (init_to_zero)
    {
      GUARD_CU(Memset(temp_ptr + allocated, 0, target - allocated,
        malloc_type, stream));
    }
    GUARD_CU(Free(ptr, malloc_type));
    ptr = temp_ptr;
    temp_ptr = NULL;
  }
  allocated = target;
  return retval;
}

// ****************************
// DGC Functions
// ****************************

void DgcConfig::Set(std::string key, std::string value)
{
  if (key == "dgc_sparsity_warmup_epochs")
    warmup_epochs = std::stoi(value);

  else if (key == "dgc_init_sparsity")
    init_sparsity = std::stod(value);

  else if (key == "dgc_final_sparsity")
    final_sparsity = std::stod(value);

  else if (key == "dgc_sampling_rate")
    sampling_rate = std::stod(value);

  else if (key == "dgc_rand_seed")
    rand_seed = std::stoi(value);

  else if (key == "dgc_grid_size")
    grid_size = std::stoi(value);

  else if (key == "dgc_block_size")
    block_size = std::stoi(value);

  else if (key == "dgc_min_sampling_num")
    min_sampling_num = std::stoi(value);

  else if (key == "dgc_local_gradient_clipping")
  {
    if (value == "True")
      local_gradient_clipping = true;
    else if (value == "False")
      local_gradient_clipping = false;
  }

  else if (key == "dgc_clipping_threshold")
    clipping_threshold = std::stof(value);

  else if (key == "dgc_use_allreduce")
  {
    if (value == "True")
      use_allReduce = true;
    else if (value == "False")
      use_allReduce = false;
  }

  else if (key == "dgc_use_hierarchical_allreduce")
  {
    if (value == "True")
      use_hierarchical_allreduce = true;
    else if (value == "False")
      use_hierarchical_allreduce = false;
  }

  else if (key == "dgc_overlap_mask_allreduce")
  {
    if (value == "True")
      overlap_mask_allreduce = true;
    else if (value == "False")
      overlap_mask_allreduce = false;
  }

  else if (key == "dgc_learning_rate_decay_factor")
    learning_rate_decay_factor = std::stof(value);

  else if (key == "dgc_num_epochs_per_decay")
    num_epochs_per_decay = std::stof(value);

  else if (key == "dgc_min_learning_rate_factor")
    min_learning_rate_factor = std::stof(value);

  else if (key == "dgc_flush_steps")
    flush_steps = std::stoi(value);

  else if (key == "dgc_use_momentum_correction")
  {
    if (value == "True")
      use_momentum_correction = true;
    else if (value == "False")
      use_momentum_correction = false;
  }

  else if (key == "momentum")
    momentum = std::stof(value);

  else if (key == "num_examples_per_epoch")
    num_examples_per_epoch = std::stoi(value);

  else if (key == "batch_size")
    batch_size_per_gpu = std::stoi(value);

  //printf("%s = %s\n", key.c_str(), value.c_str());
}

template <typename T, typename SizeT, typename Compare>
hipError_t Sort(
  T           *elements,
  SizeT        num_elements,
  Compare      compare,
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  hipError_t retval = hipSuccess;

  if (malloc_type == Raw)
  {
    std::sort(elements, elements + num_elements, compare);
    return retval;
  }

  // Not using thrust for now;
  // if sort becomes performance bottleneck, change to cub
  // Note: thrust::sort hit a bug that produced illegal memory access
  //thrust::sort(thrust::cuda::par.on(stream),
  //  elements, elements + num_elements, compare);

  // Cub sorting
  bool temp_storage_allocated = false;
  if (temp_storage == NULL && temp_storage_bytes == NULL)
  {
    temp_storage = new char*;
    temp_storage[0] = NULL;
    temp_storage_bytes = new size_t;
    temp_storage_bytes[0] = 0;
    temp_storage_allocated = true;
  }

  size_t required_bytes = 0;
  GUARD_CU(hipcub::DeviceRadixSort::SortKeys(
    (char*)NULL, required_bytes,
    elements, elements,
    num_elements, 0, sizeof(T) * 8, stream));

  GUARD_CU(GarenteeAllocation(temp_storage[0],
    temp_storage_bytes[0], required_bytes, malloc_type, flags));
  //GUARD_CU2("hipDeviceSynchronize",
  //  hipDeviceSynchronize());

  GUARD_CU(hipcub::DeviceRadixSort::SortKeys(
    temp_storage[0], temp_storage_bytes[0],
    elements, elements,
    num_elements, 0, sizeof(T) * 8, stream));

  if (temp_storage_allocated)
  {
    GUARD_CU(Free(temp_storage[0], malloc_type));
    free(temp_storage);
    free(temp_storage_bytes);
    temp_storage = NULL;
    temp_storage_bytes = NULL;
    temp_storage_allocated = false;
  }

  return retval;
}

template <typename T, typename SizeT>
hipError_t Sort(
  T      *elements,
  SizeT   num_elements,
  hipStream_t stream = 0,
  Malloc_t malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  return Sort(elements, num_elements,
    [] __host__ __device__ (T a, T b){ return a < b;},
    stream, malloc_type, temp_storage, temp_storage_bytes, flags);
}

template <typename SizeT>
hipError_t Sort(
  ncclDataType_t nccl_type,
  void        *elements,
  SizeT        num_elements,
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  hipError_t retval = hipSuccess;

  switch (nccl_type)
  {
  case ncclFloat32:
    retval = Sort<float> ((float*)elements, num_elements,
      stream, malloc_type, temp_storage, temp_storage_bytes, flags);
    break;

  case ncclFloat64:
    retval = Sort<double> ((double*)elements, num_elements,
      stream, malloc_type, temp_storage, temp_storage_bytes, flags);
    break;

  case ncclInt32:
    retval = Sort<int32_t> ((int32_t*)elements, num_elements,
      stream, malloc_type, temp_storage, temp_storage_bytes, flags);
    break;

  case ncclInt64:
    retval = Sort<int64_t> ((int64_t*)elements, num_elements,
      stream, malloc_type, temp_storage, temp_storage_bytes, flags);
    break;

  default:
    break;
  }
  return retval;
}

template <typename T, typename SizeT, typename Compare>
hipError_t SegSort(
  T           *elements,
  SizeT        num_elements,
  SizeT       *seg_starts,
  int          num_segments,
  Compare      compare,
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  hipError_t retval = hipSuccess;

  if (malloc_type == Raw)
  {
    for (int i = 0; i < num_segments; i++)
      std::sort(elements + seg_starts[i], elements + seg_starts[i+1], compare);
    return retval;
  }

  // Not using thrust for now;
  // if sort becomes performance bottleneck, change to cub
  // Note: thrust::sort hit a bug that produced illegal memory access
  //thrust::sort(thrust::cuda::par.on(stream),
  //  elements, elements + num_elements, compare);

  // Cub sorting
  bool temp_storage_allocated = false;
  if (temp_storage == NULL && temp_storage_bytes == NULL)
  {
    temp_storage = new char*;
    temp_storage[0] = NULL;
    temp_storage_bytes = new size_t;
    temp_storage_bytes[0] = 0;
    temp_storage_allocated = true;
  }

  size_t required_bytes = 0;
  GUARD_CU(hipcub::DeviceSegmentedRadixSort::SortKeys(
    (char*)NULL, required_bytes,
    elements, elements, num_elements,
    num_segments, seg_starts, seg_starts + 1,
    0, sizeof(T) * 8, stream));

  GUARD_CU(GarenteeAllocation(temp_storage[0],
    temp_storage_bytes[0], required_bytes, malloc_type, flags));
  //GUARD_CU2("hipDeviceSynchronize",
  //  hipDeviceSynchronize());

  GUARD_CU(hipcub::DeviceSegmentedRadixSort::SortKeys(
    temp_storage[0], temp_storage_bytes[0],
    elements, elements, num_elements,
    num_segments, seg_starts, seg_starts + 1,
    0, sizeof(T) * 8, stream));

  if (temp_storage_allocated)
  {
    GUARD_CU(Free(temp_storage[0], malloc_type));
    free(temp_storage);
    free(temp_storage_bytes);
    temp_storage = NULL;
    temp_storage_bytes = NULL;
    temp_storage_allocated = false;
  }

  return retval;
}

template <typename T, typename SizeT>
hipError_t SegSort(
  T      *elements,
  SizeT   num_elements,
  SizeT  *seg_starts,
  int     num_segments,
  hipStream_t stream = 0,
  Malloc_t malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  return SegSort(elements, num_elements, seg_starts, num_segments,
    [] __host__ __device__ (T a, T b){ return a < b;},
    stream, malloc_type, temp_storage, temp_storage_bytes, flags);
}

template <typename T>
hipError_t ClipGradient(
  T          *gradients,
  //uint64_t   *layer_offsets,
  //int         num_layers,
  std::vector<std::pair<std::string, uint64_t> > &layers,
             // <name, #elements> of layers
  DgcConfig  &config,
  DgcState   &state,
  DgcToken   *token)
{
  hipError_t retval = hipSuccess;

  // skip first step, because total number of layers are unknown
  if (state.step == 0)
    return retval;

  int num_layers = layers.size();
  GUARD_CU(GarenteeAllocation(state.temp_storage, state.temp_storage_bytes,
    sizeof(T) * 2 * num_layers + sizeof(uint32_t) * (num_layers + 1)));
  GUARD_CU(GarenteeAllocation(token -> h_layer_starts,
    token -> h_layer_starts_allocated, num_layers + 1, Malloc_t::Host));
  uint32_t start_counter = 0;
  for (int i = 0; i < num_layers; i++)
  {
    token -> h_layer_starts[i] = start_counter;
    start_counter += layers[i].second;
  }
  token -> h_layer_starts[num_layers] = start_counter;

  T* sums         = (T*)(state.temp_storage);
  T* coefficients = (T*)(state.temp_storage + sizeof(T) * num_layers);
  uint32_t* layer_starts
    = (uint32_t*)(state.temp_storage + sizeof(T) * 2 * num_layers);
  auto stream     = config.stream;
  int  grid_size  = config.grid_size;
  int  block_size = config.block_size;
  auto clipping_threshold = config.clipping_threshold;

  GUARD_CU(Memset(sums, 0, num_layers, Malloc_t::Default, stream));
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(layer_starts, token -> h_layer_starts,
      sizeof(uint32_t) * (num_layers + 1), hipMemcpyHostToDevice, stream));

  // loop_kernel<<<grid_size, block_size, 0, stream>>>(layer_offsets[num_layers],
  //   [offsets, sums, gradients, num_layers] __device__ (const uint64_t &i)
  //   {
  //     int layer = binarySearch(offsets, 0, num_layers, i);
  //     //if (i < offsets[layer] || i >= offsets[layer + 1])
  //     //  printf("offset mismatch: i = %ld, layer = %d, offsets = %ld, %ld, %ld\n",
  //     //      i, layer, layer > 0 ? offsets[layer -1] : -1,
  //     //      offsets[layer], layer < num_layers ? offsets[layer + 1] : -1);
  //
  //     auto gradient = gradients[i];
  //     atomicAdd(sums + layer, gradient * gradient);
  //   });
  L2norm_kernel<<<grid_size, block_size, 0, stream>>>(
    gradients, layer_starts, num_layers, sums);

  int total_num_layers = state.layer_offset_bytes.size();
  uint64_t total_num_gradients = state.offset_byte_counter / sizeof(T);

  loop_kernel<<<grid_size, block_size, 0, stream>>>(num_layers,
    [sums, coefficients, total_num_layers, total_num_gradients,
    clipping_threshold, layer_starts]
    __device__ (const int &layer)
    {
      coefficients[layer] = clipping_threshold /
        // (sqrt(sums[layer] * total_num_gradients / (offsets[layer + 1] - offsets[layer])) + 1e-6);
        // (sqrt(sums[layer]) + 1e-6);
        (sqrt(sums[layer]) * total_num_layers + 1e-6);
        //(sqrt(sums[layer]) * total_num_gradients / (offsets[layer + 1] - offsets[layer]) + 1e-6);
      //printf("Layer %3d: L2 norm = %3.6f, #gradients = %6ld, coef = %3.6f\n",
      //  layer, sqrt(sums[layer]), (long)(offsets[layer+1] - offsets[layer]),
      //  coefficients[layer]);
    });

  loop_kernel<<<grid_size, block_size, 0, stream>>>(start_counter,
    [layer_starts, gradients, coefficients, num_layers]
    __device__ (const uint32_t &i)
    {
      int layer = binarySearch(layer_starts, 0, num_layers, i);
      auto coefficient = coefficients[layer];
      if (coefficient < 1)
        gradients[i] *= coefficient;
    });

  return retval;
}

hipError_t DgcToken::Init()
{
  hipError_t retval = hipSuccess;
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> dgc_finish    ), hipEventDisableTiming));
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> stream2_begin ), hipEventDisableTiming));
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> stream2_finish), hipEventDisableTiming));
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> stream3_begin ), hipEventDisableTiming));
  return retval;
}

hipError_t DgcToken::isFinished(bool &finished, int check)
{
  hipError_t retval = hipSuccess;
  if (this -> dgc_finished)
  {
    finished = true;
    return retval;
  }
  retval = hipEventQuery(this -> dgc_finish);
  if (retval == hipSuccess)
  {
    finished = true;
    this -> dgc_finished = true;
  } else if (retval == hipErrorNotReady)
  {
    finished = false;
    retval = hipSuccess;
  }
  return retval;
}

hipError_t MaskToken::Init()
{
  hipError_t retval = hipSuccess;
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> d2h_finish), hipEventDisableTiming));
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> h2d_finish), hipEventDisableTiming));
  return retval;
}

hipError_t MaskToken::isFinished(bool &finished, int check)
{
  hipError_t retval = hipSuccess;
  if (check == 0)
  {
    if (this -> d2h_finished)
    {
      finished = true;
      return retval;
    }
    retval = hipEventQuery(this -> d2h_finish);
    if (retval == hipSuccess)
    {
      finished = true;
      this -> d2h_finished = true;
    } else if (retval == hipErrorNotReady)
    {
      finished = false;
      retval = hipSuccess;
    }
  }

  else if (check == 1)
  {
    if (this -> mpi_finished)
    {
      finished = true;
      return retval;
    }
    if (!this -> mpi_started)
    {
      finished = false;
      return retval;
    }

    //printf("Checking MPI token\n");
    int flag;
    GUARD_MPI2("MPI_Test",
      MPI_Test(&(this -> mpi_request), &flag, MPI_STATUS_IGNORE));
    if (flag)
    {
      //printf("\t token = %p, received %ld masks from MPI, first 3: %#X, %#X, %#X\n",
      //  this, (long)this -> num_masks,
      //  this -> h_recv_masks[0], this -> h_recv_masks[1], this -> h_recv_masks[2]);
      finished = true;
      this -> mpi_finished = true;
      this -> mpi_started = false;
    } else {
      finished = false;
    }
  }

  else if (check == 2)
  {
    if (this -> h2d_finished)
    {
      finished = true;
      return retval;
    }
    retval = hipEventQuery(this -> h2d_finish);
    if (retval == hipSuccess)
    {
      finished = true;
      this -> h2d_finished = true;
    } else if (retval == hipErrorNotReady)
    {
      finished = false;
      retval = hipSuccess;
    }
  }

  return retval;
}

template <typename TokenT>
hipError_t GetToken(
  std::list<TokenT*> &free_queue,
  std::list<TokenT*> &busy_queue,
  TokenT* &token,
  int check = 0)
{
  hipError_t retval = hipSuccess;

  if (free_queue.size() != 0)
  {
    token = free_queue.front();
    free_queue.pop_front();
    return retval;
  }

  if (busy_queue.size() != 0 && check != -1)
  {
    auto first_token = busy_queue.front();
    bool finished = false;
    GUARD_CU(first_token -> isFinished(finished, check));
    if (finished)
    {
      token = first_token;
      busy_queue.pop_front();
      return retval;
    }

    //if (retval != hipErrorNotReady)
    //{
    //  GUARD_CU2("hipEventQuery", retval);
    //}
    //retval = hipSuccess;
  }

  token = new TokenT;
  GUARD_CU(token -> Init());
  return retval;
}

hipError_t TryPushMask(
  int             max_requests_allowed_waiting,
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  hipError_t retval = hipSuccess;
  if (max_requests_allowed_waiting != 0)
  {
    int total_num_layers = 0;
    for (auto &token : state.d2h_mask_queue)
    {
      total_num_layers += token -> num_layers;
    }
    if (total_num_layers >= state.layer_offset_bytes.size())
      max_requests_allowed_waiting = 0;
  }

  //while (!state.d2h_mask_queue.empty())
  while (state.d2h_mask_queue.size() > max_requests_allowed_waiting)
  {
    auto token = state.d2h_mask_queue.front();
    //bool finished = false;
    //GUARD_CU(token -> isFinished(finished, 0));
    //if (!finished)
    //  break;
    GUARD_CU2("hipEventSynchronize",
      hipEventSynchronize(token -> d2h_finish));
    token -> d2h_finished = true;

    //printf("%ld\t token = %p, %ld masks pushing to MPI\n",
    //  (long)state.step, token, (long)token -> num_masks);

    state.d2h_mask_queue.pop_front();
    GUARD_MPI2("MPI_Iallreduce",
      MPI_Iallreduce(token -> h_send_masks, token -> h_recv_masks,
        (int)token -> num_masks, PreDefinedValues<uint32_t>::getMpiDataType(), MPI_BOR,
        config.use_hierarchical_allreduce ? config.cross_comm : config.mpi_comm,
        &(token -> mpi_request)));
    token -> mpi_started  = true;
    token -> mpi_finished = false;
    //printf("%ld\t token = %p, %ld masks pushed to MPI, first 3: %#X, %#X, %#X\n",
    //  (long)state.step, token, (long)token -> num_masks,
    //  token -> h_send_masks[0], token -> h_send_masks[1], token -> h_send_masks[2]);
    state.mpi_mask_queue.push_back(token);
  }
  return retval;
}

// Main DGC routine
template <typename T, typename SizeT>
hipError_t GradientAllReduce(
  T              *input_gradients,     // GPU pointer to the input_gradients
  T              *output_gradients,     // GPU pointer to the output_gradients
  std::vector<std::pair<std::string, uint64_t> > &layers,
                                  // <name, #elements> of layers
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  hipError_t retval = hipSuccess;
  //SizeT num_samples  = 0;
  auto  block_size   = config.block_size;
  auto  grid_size    = config.grid_size;
  auto  stream       = config.stream;
  int   num_layers   = layers.size();
  SizeT num_gradients = 0;

  DgcToken *token = NULL;
  GUARD_CU(GetToken(state.free_tokens, state.busy_tokens, token));
  if (config.stream2 == 0)
  {
    int greatest_priority;
    GUARD_CU2("hipDeviceGetStreamPriorityRange",
      hipDeviceGetStreamPriorityRange(NULL, &greatest_priority));
    GUARD_CU2("hipStreamCreateWithPriority",
      hipStreamCreateWithPriority(&(config.stream2), hipStreamNonBlocking,
        greatest_priority));
    GUARD_CU2("hipStreamCreateWithPriority",
      hipStreamCreateWithPriority(&(config.stream3), hipStreamNonBlocking,
        greatest_priority));
    GUARD_CU2("hipStreamCreateWithPriority",
      hipStreamCreateWithPriority(&(config.stream4), hipStreamNonBlocking,
        greatest_priority));
  }
  auto stream2 = config.stream2;
  auto stream3 = config.stream3;
  auto stream4 = config.stream4;

  //GUARD_CU2("hipStreamSynchronize before",
  //  hipStreamSynchronize(stream));

  if (config.local_gradient_clipping)
    GUARD_CU(ClipGradient(input_gradients, layers, config, state, token));
  //GUARD_CU2("hipStreamSynchronize after clipping",
  //  hipStreamSynchronize(stream));

  // find which step is currently in and look for unallocated layers
  std::vector<std::pair<std::string, uint64_t> > layers_to_allocate;
  SizeT num_gradients_to_allocate = 0;
  for (auto &layer : layers)
  {
    auto name = layer.first;
    num_gradients += layer.second;
    // finds step number
    auto counter_it = state.step_counters.find(name);
    if (counter_it == state.step_counters.end())
      state.step_counters[name] = 0;
    else {
      auto step = counter_it -> second;
      counter_it -> second ++;
      if (state.step < step)
        state.step = step;
    }

    auto offset_it = state.layer_offset_bytes.find(name);
    if (offset_it == state.layer_offset_bytes.end()) {
      layers_to_allocate.push_back(std::make_pair(layer.first, layer.second));
      num_gradients_to_allocate += layer.second;
    }
  } // end of for layers

  // allocate new layers
  if (num_gradients_to_allocate > 0) {
    if (config.use_momentum_correction) {
      GUARD_CU(GarenteeAllocation(state.pervious_verlocity,
        state.pervious_verlocity_allocated,
        state.offset_byte_counter + sizeof(T) * num_gradients_to_allocate,
        Malloc_t::Default, hipMemAttachGlobal, stream, true, true));
      GUARD_CU(GarenteeAllocation(state.pervious_accumulated_verlocity,
        state.pervious_accumulated_verlocity_allocated,
        state.offset_byte_counter + sizeof(T) * num_gradients_to_allocate,
        Malloc_t::Default, hipMemAttachGlobal, stream, true, true));
    } else {
      GUARD_CU(GarenteeAllocation(state.pervious_accumulated_gradients,
        state.pervious_accumulated_gradients_allocated,
        state.offset_byte_counter + sizeof(T) * num_gradients_to_allocate,
        Malloc_t::Default, hipMemAttachGlobal, stream, true, true));
    }
    for (auto& layer : layers_to_allocate) {
      state.layer_offset_bytes[layer.first] = state.offset_byte_counter;
      state.offset_byte_counter += layer.second * sizeof(T);
    }
  }

  GUARD_CU(GarenteeAllocation(token -> h_layer_starts,
    token -> h_layer_starts_allocated, num_layers + 1, Malloc_t::Host));
  // find continous layers as chunks
  // <start, size, offset> of chunks
  std::vector<std::tuple<SizeT, SizeT, size_t> > chunks;
  size_t chunk_offset_bytes = state.layer_offset_bytes[layers.begin() -> first];
  SizeT  layer_start = 0;
  SizeT  chunk_start = 0;
  SizeT  chunk_size  = 0;
  for (int i = 0; i < num_layers; i++) {
    auto &layer = layers[i];
    token -> h_layer_starts[i] = layer_start;
    //printf("layer %d : %s [%ld, %ld)\n",
    //    i, layer.first.c_str(), layer_start, layer_start + layer.second);
    if (chunk_offset_bytes + chunk_size * sizeof(T) !=
      state.layer_offset_bytes[layer.first]) {
      // mismatch
      chunks.push_back(std::make_tuple(
        chunk_start, chunk_size, chunk_offset_bytes));
      chunk_size  = 0;
      chunk_start = layer_start;
      chunk_offset_bytes = state.layer_offset_bytes[layer.first];
    }

    chunk_size  += layer.second;
    layer_start += layer.second;
  } // end of for layers
  token -> h_layer_starts[num_layers] = layer_start;
  if (chunk_size != 0)
    chunks.push_back(std::make_tuple(
      chunk_start, chunk_size, chunk_offset_bytes));

  auto &layer_starts = state.layer_starts;
  GUARD_CU(GarenteeAllocation(state.layer_starts,
    state.layer_starts_allocated, num_layers + 1));
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(state.layer_starts, token -> h_layer_starts,
      sizeof(uint32_t) * (num_layers + 1), hipMemcpyHostToDevice, stream));

  // Memory allocation and type conversion
  if (config.use_momentum_correction) {
    GUARD_CU(GarenteeAllocation(state.verlocity,
      state.verlocity_allocated, num_gradients * sizeof(T)));
    GUARD_CU(GarenteeAllocation(state.accumulated_verlocity,
      state.accumulated_verlocity_allocated, num_gradients * sizeof(T)));
  } else {
    GUARD_CU(GarenteeAllocation(state.accumulated_gradients,
      state.accumulated_gradients_allocated, num_gradients * sizeof(T)));
  }
  T* verlocity = (T*)(state.verlocity);
  T* accumulated_verlocity = (T*)(state.accumulated_verlocity);
  T* accumulated_gradients = (T*)(state.accumulated_gradients);
  T* elements = NULL;

  if (config.use_momentum_correction) {
    // momentum correction by chunks
    for (auto& chunk : chunks) {
      SizeT chunk_start = std::get<0>(chunk);
      SizeT chunk_size  = std::get<1>(chunk);
      size_t chunk_offset = std::get<2>(chunk);

      T* pervious_verlocity
        = (T*)(state.pervious_verlocity + chunk_offset);
      T* pervious_accumulated_verlocity
        = (T*)(state.pervious_accumulated_verlocity + chunk_offset);
      auto &momentum = config.momentum;

      //printf("input_gradients = %p, gradient_chunk = [%ld, %ld), "
      //  "pervious_verlocity = %p, verlocity = %p, "
      //  "pervious_accumulated_verlocity = %p, accumulated_verlocity = %p\n",
      //  input_gradients, gradient_start_chunk,
      //  gradient_start_chunk + num_gradients_chunk,
      //  pervious_verlocity, verlocity,
      //  pervious_accumulated_verlocity, accumulated_verlocity);

      loop_kernel<<<grid_size, block_size, 0, stream>>>(chunk_size,
        [momentum, input_gradients, chunk_start,
        pervious_verlocity, verlocity,
        accumulated_verlocity, pervious_accumulated_verlocity]
        __device__ (const SizeT &i) {
          auto pos = i + chunk_start;
          auto u = pervious_verlocity[i] * momentum + input_gradients[pos];
          accumulated_verlocity[pos] = pervious_accumulated_verlocity[i] + u;
          verlocity[pos] = u;
        });
    }
    elements = accumulated_verlocity;
  }

  else {
    // accumulate gradients
    for (auto& chunk : chunks) {
      SizeT chunk_start = std::get<0>(chunk);
      SizeT chunk_size  = std::get<1>(chunk);
      size_t chunk_offset = std::get<2>(chunk);

      T* pervious_accumulated_gradients
        = (T*)(state.pervious_accumulated_gradients + chunk_offset);

      loop_kernel<<<grid_size, block_size, 0, stream>>>(chunk_size,
        [input_gradients, chunk_start,
        accumulated_gradients, pervious_accumulated_gradients]
        __device__ (const SizeT &i) {
          auto pos = i + chunk_start;
          auto g = pervious_accumulated_gradients[i] + input_gradients[pos];
          accumulated_gradients[pos] = g;
        });
    }
    elements = accumulated_gradients;
  }

  //GUARD_CU2("hipStreamSynchronize after momentum correction",
  //  hipStreamSynchronize(stream));
  bool to_overlap_mask = config.use_allReduce && config.overlap_mask_allreduce;
  if (to_overlap_mask) {
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream3_begin, stream));
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream3, token -> stream3_begin, 0));
  }

  // Determine the threshold
  uint64_t num_examples_per_step
    = config.batch_size_per_gpu * config.global_num_gpus;
  uint64_t steps_per_epoch
    = config.num_examples_per_epoch / num_examples_per_step;
  if (steps_per_epoch * num_examples_per_step < config.num_examples_per_epoch)
    steps_per_epoch ++;
  uint64_t epoch    = state.step * 1.0 / steps_per_epoch;
  double sparsity   = config.final_sparsity;
  if (epoch < config.warmup_epochs) {
    sparsity = config.init_sparsity * exp(
      log(config.final_sparsity / config.init_sparsity)
      / (config.warmup_epochs - 1) * epoch);
    //if (epoch * steps_per_epoch == state.step)
    //  printf("Epoch %ld, Step %ld, sparsity = %lf\n",
    //    epoch, state.step, sparsity);
  }
  SizeT  target_num = num_gradients * (1 - sparsity);

  // Communicate all gradients if it's a flushing step
  bool to_flush = false;
  if (config.flush_steps > 0) {
    if ((state.step >= config.flush_steps) &&
        (state.step % config.flush_steps) == 0)
      to_flush = true;
  }

  if (to_flush) {
    printf("%ld\t Flushing %ld elements\n",
      (long)state.step, (long)num_gradients);

    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_begin, stream));
    GUARD_NCCL2("ncclAllReduce",
      ncclAllReduce(elements, output_gradients,
        (size_t)num_gradients, PreDefinedValues<T>::NCCLDataType, ncclSum,
        config.use_hierarchical_allreduce ?
        config.nccl_cross_comm : config.nccl_comm, stream));

    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream2, token -> stream2_begin, 0));
    if (config.use_momentum_correction) {
      for (auto& chunk : chunks) {
        SizeT  chunk_start  = std::get<0>(chunk);
        SizeT  chunk_size   = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);

        T* pervious_verlocity
          = (T*)(state.pervious_verlocity + chunk_offset);
        T* pervious_accumulated_verlocity
          = (T*)(state.pervious_accumulated_verlocity + chunk_offset);

        GUARD_CU(Memset(pervious_verlocity,
          0, chunk_size, Malloc_t::Default, stream2));
        GUARD_CU(Memset(pervious_accumulated_verlocity,
          0, chunk_size, Malloc_t::Default, stream2));
      }
    }
    else {
      for (auto& chunk : chunks) {
        SizeT chunk_start = std::get<0>(chunk);
        SizeT chunk_size  = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);

        T* pervious_accumulated_gradients
          = (T*)(state.pervious_accumulated_gradients + chunk_offset);
        GUARD_CU(Memset(pervious_accumulated_gradients,
          0, chunk_size, Malloc_t::Default, stream2));
      }
    }
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_finish, stream2));

    if (config.learning_rate_decay_factor > 0 &&
        epoch >= config.num_epochs_per_decay) {
      float learning_rate_adjustment = 1;
      auto epoch_ = epoch;
      while (epoch_ >= config.num_epochs_per_decay)
      {
        learning_rate_adjustment *= config.learning_rate_decay_factor;
        epoch_ -= config.num_epochs_per_decay;
      }
      if (learning_rate_adjustment < config.min_learning_rate_factor)
        learning_rate_adjustment = config.min_learning_rate_factor;
      if (config.global_gpu_rank == 0)
        printf("%ld\t learning_rate_adjustment = %f\n",
          (long)state.step, learning_rate_adjustment);

      loop_kernel <<<grid_size, block_size, 0, stream>>>(num_gradients,
        [learning_rate_adjustment, output_gradients] __device__ (const SizeT &i)
        {
          output_gradients[i] *= learning_rate_adjustment;
        });
    }
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream, token -> stream2_finish, 0));

    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> dgc_finish, stream));
    token -> dgc_finished = false;
    state.busy_tokens.push_back(token);

    return retval;
  }

  // Sampling
  auto &samp_starts = state.samp_starts;
  GUARD_CU(GarenteeAllocation(state.samp_starts, state.samp_starts_allocated,
    num_layers + 1));
  GUARD_CU(GarenteeAllocation(token -> h_samp_starts,
    token -> h_samp_starts_allocated, num_layers + 1, Malloc_t::Host));
  uint32_t samp_counter = 0;
  for (int i = 0; i < num_layers; i++)
  {
    auto &layer = layers[i];
    token -> h_samp_starts[i] = samp_counter;

    uint32_t num_samples = 0;
    if (config.sampling_rate < 1 &&
        layer.second > config.min_sampling_num) {

      num_samples = layer.second * config.sampling_rate;
      if (num_samples < config.min_sampling_num)
        num_samples = config.min_sampling_num;
      uint32_t num_selected_samples = config.min_gradients_comm_per_layer
        * config.sampling_rate;
      if (num_selected_samples < config.min_selected_samples_per_layer)
        num_selected_samples = config.min_selected_samples_per_layer;
      if (num_samples < num_selected_samples * 1.0f / (1 - sparsity)) {
        num_samples = num_selected_samples * 1.0f / (1 - sparsity);
      }
      if (num_samples > layer.second)
        num_samples = layer.second;
    }

    else { // no sampling
      num_samples = layer.second;
      //GUARD_CU(GarenteeAllocation(state.samp_data, state.samp_allocated,
      //  num_samples * sizeof(T)));

      //GUARD_CU2("hipMemcpyAsync",
      //  hipMemcpyAsync(state.samp_data, gradients,
      //    sizeof(T) * num_samples, hipMemcpyDeviceToDevice, (to_overlap_mask ? stream3 : stream));
      //T* samp_data = (T*)(state.samp_data);
      //loop_kernel<<<grid_size, block_size, 0, stream>>>(num_samples,
      //  [samp_data, elements] __device__ (const SizeT &i){
      //    samp_data[i] = abs(elements[i]);
      //  });
    }
    //printf("samp %d of %d: [%d, %d)\n", i, num_layers, samp_counter,
    //  samp_counter + num_samples);
    samp_counter += num_samples;
  }
  token -> h_samp_starts[num_layers] = samp_counter;
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(state.samp_starts, token -> h_samp_starts,
      sizeof(uint32_t) * (num_layers + 1), hipMemcpyHostToDevice,
      (to_overlap_mask ? stream3 : stream)));

  auto &rand_states = state.rand_states;
  auto &rand_seed   = config.rand_seed;
  if (rand_states == NULL) {
    GUARD_CU(Malloc(rand_states, grid_size * block_size));

    loop_kernel
      <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(
      (SizeT)grid_size * block_size,
      [rand_states, rand_seed] __device__ (const SizeT &i){
        hiprand_init(rand_seed, i, 0, rand_states + i);
      });
  }

  GUARD_CU(GarenteeAllocation(state.samp_data, state.samp_allocated,
    samp_counter * sizeof(T)));
  T* samp_data = (T*)(state.samp_data);

  //sample_kernel <T, SizeT>
  //  <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(
  //  elements, num_gradients,
  //  samp_data, num_samples,
  //  state.rand_states);
  sample_kernel2
    <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(
    elements, num_gradients,
    state.layer_starts, num_layers,
    state.samp_starts, samp_data, state.rand_states);

  //GUARD_CU2("hipStreamSynchronize after sampling",
  //  hipStreamSynchronize(to_overlap_mask ? stream3 : stream));
  //GUARD_CU2("hipDeviceSynchronize before Sort",
  //  hipDeviceSynchronize());

  // Sort the samples
  //GUARD_CU(Sort(samp_data, num_samples,
  //  (to_overlap_mask ? stream3 : stream), Malloc_t::Default,
  //  &(state.temp_storage), &(state.temp_storage_bytes)));
  GUARD_CU(SegSort(samp_data, samp_counter, state.samp_starts, num_layers,
    (to_overlap_mask ? stream3 : stream), Malloc_t::Default,
    &(state.temp_storage), &(state.temp_storage_bytes)));
  //GUARD_CU2("hipDeviceSynchronize after Sort",
  //  hipDeviceSynchronize());
  //GUARD_CU2("hipStreamSynchronize after Sort",
  //  hipStreamSynchronize(to_overlap_mask ? stream3 : stream));

  //auto &threshold = state.gradient_threshold;
  //if (threshold == NULL) {
  //  GUARD_CU(Malloc(threshold, 1));
  //}

  //loop_kernel<<<1, 1, 0, (to_overlap_mask ? stream3 : stream)>>>((SizeT)1,
  //  [threshold, samp_data, num_samples, sparsity] __device__ (const SizeT &i){
  //    SizeT pos = num_samples * sparsity;
  //    if (pos >= num_samples)
  //      pos = num_samples - 1;
  //    threshold[0] = samp_data[pos];
  //    //printf("selecting samp[%d] from [%d] {%f, %f, ... %f, %f, %f, ... %f, %f}\n",
  //    //  pos, num_samples,
  //    //  num_samples > 0 ? samp_data[0] : -1,
  //    //  num_samples > 1 ? samp_data[1] : -1,
  //    //  num_samples + 1 > pos  && pos > 0 ? samp_data[pos - 1] : -1,
  //    //  num_samples > pos && pos >= 0 ? samp_data[pos] : -1,
  //    //  num_samples > pos + 1 && pos + 1 >= 0 ? samp_data[pos + 1] : -1,
  //    //  num_samples > 1 ? samp_data[num_samples - 2] : -1,
  //    //  num_samples > 0 ? samp_data[num_samples - 1] : -1);
  //  });
  auto &thresholds = state.thresholds;
  auto &min_selected_samples_per_layer = config.min_gradients_comm_per_layer;
  GUARD_CU(GarenteeAllocation(thresholds, state.thresholds_allocated, num_layers));

  loop_kernel
    <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(num_layers,
    [thresholds, samp_data, samp_starts, sparsity, min_selected_samples_per_layer]
    __device__ (const int &layer){
      auto samp_start = samp_starts[layer];
      auto samp_end   = samp_starts[layer + 1];
      auto samp_size  = samp_end - samp_start;
      SizeT pos = samp_size * sparsity;
      if (pos >= samp_size)
        pos = samp_size;
      if (min_selected_samples_per_layer < samp_size &&
        pos > samp_size - min_selected_samples_per_layer)
        pos = samp_size - min_selected_samples_per_layer;
      thresholds[layer] = samp_data[samp_start + pos];
    });

  if (config.use_allReduce) {
    // use allReduce on mask to communicate

    SizeT num_masks = num_gradients / 32;
    if (num_masks * 32 < num_gradients)
      num_masks ++;

    auto mask_allocated_ = state.mask_allocated;
    GUARD_CU(GarenteeAllocation(state.send_masks  , mask_allocated_, num_masks));
    mask_allocated_ = state.mask_allocated;
    GUARD_CU(GarenteeAllocation(state.recv_masks  , mask_allocated_, num_masks));
    if (!config.overlap_mask_allreduce) {
      mask_allocated_ = state.mask_allocated;
      GUARD_CU(GarenteeAllocation(state.h_send_masks, mask_allocated_,
        num_masks, Malloc_t::Host));
      mask_allocated_ = state.mask_allocated;
      GUARD_CU(GarenteeAllocation(state.h_recv_masks, mask_allocated_,
        num_masks, Malloc_t::Host));
    }
    if (state.mask_allocated < num_masks)
      state.mask_allocated = num_masks;

    auto &mask_counters = state.mask_counters;
    auto &mask_offsets  = state.mask_offsets;
    GUARD_CU(GarenteeAllocation(
        mask_counters, state.mask_counters_allocated, (num_masks + 1)));
    GUARD_CU(GarenteeAllocation(
        mask_offsets , state.mask_offsets_allocated , (num_masks + 1)));

    if (state.h_num_gradients_to_communicate == NULL)
        GUARD_CU(Malloc(state.h_num_gradients_to_communicate, 1, Malloc_t::Host));

    //GUARD_CU2("hipStreamSynchronize after allocation",
    //  hipStreamSynchronize(to_overlap_mask ? stream3 : stream));

    auto &send_masks = state.send_masks;
    auto &recv_masks = state.recv_masks;
    loop_kernel
      <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(num_masks,
      [send_masks, num_gradients, thresholds, layer_starts, num_layers,
      elements]
      __device__ (const SizeT &i)
      {
        uint32_t mask = 0;
        SizeT offset = i * 32;
        int end_j = 32, j = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;
        while (j < end_j)
        {
          auto pos = j + offset;
          T element = elements[pos];
          int layer = binarySearch(layer_starts, 0, num_layers, pos);
          if (!isfinite(element * 1.0f))
          {
            j ++;
            continue;
          }

          if (!(abs(element) < thresholds[layer]))
          {
            mask |= (((uint32_t)1) << j);
          }
          j++;
        }
        send_masks[i] = mask;
      });

    if (config.overlap_mask_allreduce) {
      MaskToken *mask_token = NULL;

      // Get token and allocate host space
      GUARD_CU(GetToken(state.free_mask_tokens, state.h2d_mask_queue,
        mask_token, 2));
      mask_allocated_ = mask_token -> mask_allocated;
      GUARD_CU(GarenteeAllocation(mask_token -> h_send_masks, mask_allocated_,
        num_masks, Malloc_t::Host));
      mask_allocated_ = mask_token -> mask_allocated;
      GUARD_CU(GarenteeAllocation(mask_token -> h_recv_masks, mask_allocated_,
        num_masks, Malloc_t::Host));
      if (mask_token -> mask_allocated < num_masks)
        mask_token -> mask_allocated = num_masks;

      // Move the send mask from GPU to CPU
      GUARD_CU2("hipMemcpyAsync",
        hipMemcpyAsync(mask_token -> h_send_masks, send_masks,
          sizeof(uint32_t) * num_masks, hipMemcpyDeviceToHost, stream3));
      GUARD_CU2("hipEventRecord",
        hipEventRecord(mask_token -> d2h_finish, stream3));
      mask_token -> d2h_finished = false;

      // Record the token for every layer
      auto &current_layer_records = state.layer_records[state.step % 2];
      uint32_t layer_start = 0;
      for (auto &layer : layers) {
        auto name = layer.first;
        current_layer_records[name].token = mask_token;
        current_layer_records[name].layer_start = layer_start;
        layer_start += layer.second;
      }
      mask_token -> num_masks = num_masks;
      mask_token -> num_layers = layers.size();
      if (state.step + 1 == config.overlap_skip_steps)
        mask_token -> num_layers_produced = layers.size() * 2;
      else
        mask_token -> num_layers_produced = layers.size();
      mask_token -> num_layers_comsumed = 0;
      state.d2h_mask_queue.push_back(mask_token);

      //printf("%ld\t token = %p, %ld masks d2h started\n",
      //  (long)state.step, mask_token, (long) num_masks);
      if (state.step < config.overlap_skip_steps)
      {
        //GUARD_CU2("hipStreamSynchronize",
        //    hipStreamSynchronize(stream3));
        //printf("%ld\t token = %p, %ld masks d2h finished\n",
        //  (long)state.step, mask_token, (long) num_masks);
        GUARD_CU(TryPushMask(0, config, state));
      }

      // wait for the mask from pervious step
      bool all_layers_ready = false;
      int pervious_step_index = (state.step < config.overlap_skip_steps) ?
        (state.step % 2) : ((state.step + 1) % 2);
      auto &pervious_layer_records = state.layer_records[pervious_step_index];
      //printf("%ld\t Waiting to receive %ld masks from MPI\n",
      //  (long)state.step, (long)num_masks);
      while (!all_layers_ready)
      {
        all_layers_ready = true;
        for (auto &layer : layers) {
          auto name   = layer.first;
          auto record = pervious_layer_records[name];
          bool finished = false;
          //if (record.token == NULL)
          //{
          //  printf("Can't find record for layer %s\n", name.c_str());
          //} else {
            //printf("Checking a token\n");
            GUARD_CU(record.token -> isFinished(finished, 1));
            //printf("Check done, finished = %s\n", finished ? "True" : "False");
          //}
          if (!finished) {
            all_layers_ready = false;
            break;
          }
        }

        if (!all_layers_ready) {
          std::this_thread::sleep_for(std::chrono::microseconds(10));
        }
      }
      //printf("%ld\t All %ld masks received from MPI\n",
      //  (long)state.step, (long)num_masks);

      // Reuse temp_storage to hold masks before bit-swift copy to recv_masks
      size_t request_bytes = sizeof(uint32_t) * (num_masks + layers.size() * 2);
      GUARD_CU(GarenteeAllocation(state.temp_storage2, state.temp_storage2_bytes,
        request_bytes));
      uint32_t* temp_masks_ = (uint32_t*)(state.temp_storage2);

      GUARD_CU(Memset(recv_masks + num_masks - 1, 0, 1, Malloc_t::Default, stream4));
      // move to GPU with bit swift
      SizeT chunk_start = 0;
      SizeT chunk_size = 0;
      SizeT temp_start = 0;
      //SizeT temp_size = 0;
      uint32_t chunk_num_layers = 0;
      SizeT pervious_chunk_start = 0;
      MaskToken *current_token = NULL;
      //for (auto &layer : layers)
      for (auto i = 0; i <= layers.size(); i++)
      {
        if (layers.empty())
          break;
        auto layer = layers[(i == layers.size()) ? i - 1 : i];
        auto name = layer.first;
        auto layer_size = layer.second;
        auto record = pervious_layer_records[name];
        bool new_chunk = false;

        if (i == layers.size())
        {
          //if (i == 1)
            current_token = record.token;
          new_chunk = true;
        } else if (current_token == NULL)
        {
          new_chunk = false;
          current_token = record.token;
          pervious_chunk_start = record.layer_start;
        } else if (current_token != record.token)
          new_chunk = true;
        else if (pervious_chunk_start + chunk_size != record.layer_start)
          new_chunk = true;

        if (new_chunk) {
          if (chunk_size != 0)
          {
            SizeT dest_mask_start  = chunk_start / 32;
            SizeT dest_mask_offset = chunk_start % 32;
            int dest_mask_end    = (chunk_start + chunk_size) / 32;
            if (dest_mask_end * 32 != chunk_start + chunk_size)
              dest_mask_end += 1;
            SizeT dest_mask_size   = dest_mask_end - dest_mask_start;
            uint32_t *dest_masks   = recv_masks + dest_mask_start;

            SizeT src_mask_start   = pervious_chunk_start / 32;
            SizeT src_mask_end     = (pervious_chunk_start + chunk_size) / 32;
            int src_mask_offset  = pervious_chunk_start % 32;
            if (src_mask_end * 32 != pervious_chunk_start + chunk_size)
              src_mask_end += 1;
            SizeT src_mask_size = src_mask_end - src_mask_start;
            uint32_t *temp_masks = temp_masks_ + temp_start;
            int ro = src_mask_offset - dest_mask_offset; // relative offset

            //printf("%ld\t token = %p, Copy with bitswift: src = %ld + %ld, src_size = %ld, "
            //  "dest = %ld + %ld, dest_size = %ld, temp_start = %ld, ro = %d, "
            //  "token -> num_masks = %ld, chunk_size = %ld\n",
            //  (long)state.step, current_token,
            //  (long)src_mask_start, (long)src_mask_offset, (long)src_mask_size,
            //  (long)dest_mask_start, (long)dest_mask_offset, (long)dest_mask_size,
            //  (long)temp_start, ro, (long)current_token -> num_masks, (long)chunk_size);
            GUARD_CU2("hipMemcpyAsync",
              hipMemcpyAsync(temp_masks,
                current_token -> h_recv_masks + src_mask_start,
                sizeof(uint32_t) * src_mask_size, hipMemcpyHostToDevice, stream4));

            loop_kernel<<<grid_size, block_size, 0, stream4>>>(dest_mask_size,
              [temp_masks, dest_masks, dest_mask_size, ro,
              dest_mask_offset, src_mask_offset, chunk_size]
              __device__ (const SizeT &i){
                uint32_t dest_mask = 0, mask0 = 0, mask1 = 0;
                if (i != 0 && i+1 != dest_mask_size) {
                  if (ro > 0) {
                    // move src_mask to the right
                    mask0  = temp_masks[i];
                    mask1  = temp_masks[i+1];
                    // (32-ro) bits from mask0
                    dest_mask = mask0 >> ro;
                    // ro bits from mask1
                    dest_mask |= (mask1 & ((uint32_t(1) << ro) -1)) << (32-ro);
                  } else if (ro < 0) {
                    // move src_mask to the left
                    mask0 = temp_masks[i-1];
                    mask1 = temp_masks[i];
                    // -ro bits from mask0
                    dest_mask = mask0 >> (32 + ro);
                    // (32+ro) bits from mask1
                    dest_mask |= (mask1 & ((uint32_t(1) << (32 + ro))-1)) << (-ro);
                  } else {
                    // direct copy
                    dest_mask = temp_masks[i];
                  }
                }

                else if (i == 0) {
                  // front
                  int num_gradients_in_first_mask = 32 - dest_mask_offset;
                  if (num_gradients_in_first_mask > chunk_size)
                    num_gradients_in_first_mask = chunk_size;
                  SizeT pervious_pos = src_mask_offset;
                  dest_mask = dest_masks[i];
                  for (int k = 0; k < num_gradients_in_first_mask; k++)
                  {
                    mask0 = temp_masks[pervious_pos / 32];
                    mask1 = (mask0 >> (pervious_pos % 32)) & uint32_t(1);
                    mask1 = mask1 << (k + dest_mask_offset);
                    dest_mask |= mask1;
                    pervious_pos ++;
                  }
                }

                else { // i+1 == dest_mask_size
                  // back
                  int num_gradients_in_last_mask
                    = (dest_mask_offset + chunk_size) % 32;
                  SizeT pervious_pos = src_mask_offset + chunk_size
                    - num_gradients_in_last_mask;
                  dest_mask = dest_masks[i];
                  for (int k = 0; k < num_gradients_in_last_mask; k++)
                  {
                    mask0 = temp_masks[pervious_pos / 32];
                    mask1 = (mask0 >> (pervious_pos % 32)) & uint32_t(1);
                    mask1 = mask1 << k;
                    dest_mask |= mask1;
                    pervious_pos ++;
                  }
                }

                dest_masks[i] = dest_mask;
              });

            temp_start = temp_start + src_mask_size;
            current_token -> num_layers_comsumed += chunk_num_layers;
            if (current_token -> num_layers_comsumed
              == current_token -> num_layers_produced)
            {
              GUARD_CU2("hipEventRecord",
                hipEventRecord(current_token -> h2d_finish, stream4));
              current_token -> h2d_finished = false;
            }
          }

          //GUARD_CU2("hipStreamSynchronize after local mask update",
          //  hipStreamSynchronize(stream));

          if (i == layers.size())
            break;
          pervious_chunk_start = record.layer_start;
          current_token = record.token;
          chunk_start += chunk_size;
          chunk_size = 0;
          chunk_num_layers = 0;
        }

        chunk_size += layer_size;
        chunk_num_layers ++;
      }

      //printf("%ld\t Local mask updated\n", (long)state.step);
      //GUARD_CU(TryPushMask(config, state));
      while (!state.mpi_mask_queue.empty())
      {
        auto first_token = state.mpi_mask_queue.front();
        if (first_token -> num_layers_comsumed !=
          first_token -> num_layers_produced)
          break;

        //printf("%ld\t token = %p, cleared\n",
        ///    (long)state.step, first_token);
        state.mpi_mask_queue.pop_front();
        state.h2d_mask_queue.push_back(first_token);
      }
    }

    else {
      GUARD_CU2("hipMemcpyAsync",
        hipMemcpyAsync(state.h_send_masks, send_masks,
          sizeof(uint32_t) * num_masks, hipMemcpyDeviceToHost, stream));
      GUARD_CU2("hipStreamSynchronize after mask",
        hipStreamSynchronize(stream));

      GUARD_MPI2("MPI_Allreduce",
        MPI_Allreduce(state.h_send_masks, state.h_recv_masks,
          (int)num_masks, PreDefinedValues<uint32_t>::getMpiDataType(), MPI_BOR,
          config.use_hierarchical_allreduce ? config.cross_comm : config.mpi_comm));

      GUARD_CU2("hipMemcpyAsync",
        hipMemcpyAsync(recv_masks, state.h_recv_masks,
          sizeof(uint32_t) * num_masks, hipMemcpyHostToDevice, stream));
    }
    //GUARD_CU2("hipStreamSynchronize before popc",
    //  hipStreamSynchronize(stream));
    //GUARD_CU2("hipDeviceSynchronize before popc",
    //  hipDeviceSynchronize());

    loop_kernel<<<grid_size, block_size, 0,
      (to_overlap_mask ? stream4 : stream)>>>(num_masks,
      [recv_masks, mask_counters] __device__ (const SizeT &i)
      {
        mask_counters[i] = __popc(recv_masks[i]);
        //if (mask_counters[i] != 0)
        //  printf("recv_masks[%ld] = %#X\n",
        //    (long)i, recv_masks[i]);
      });

    size_t required_bytes = 0;
    GUARD_CU(hipcub::DeviceScan::InclusiveSum(
      (char*)NULL, required_bytes,
      mask_counters, mask_offsets + 1, num_masks,
      (to_overlap_mask ? stream4 : stream)));
    GUARD_CU(GarenteeAllocation(
      state.temp_storage2, state.temp_storage2_bytes, required_bytes));

    GUARD_CU(Memset(mask_offsets, 0, 1, Malloc_t::Default,
      (to_overlap_mask ? stream4 : stream)));
    GUARD_CU(hipcub::DeviceScan::InclusiveSum(
      state.temp_storage2, required_bytes,
      mask_counters, mask_offsets + 1, num_masks,
      (to_overlap_mask ? stream4 : stream)));

    GUARD_CU2("hipMemcpyAsync",
      hipMemcpyAsync(state.h_num_gradients_to_communicate,
        mask_offsets + num_masks, sizeof(uint32_t),
        hipMemcpyDeviceToHost, to_overlap_mask ? stream4 : stream));
    //printf("%d\t Waiting for stream after InclusiveSum, #masks = %ld\n",
    //  state.step, (long)num_masks);
    GUARD_CU2("hipStreamSynchronize after InclusiveSum",
      hipStreamSynchronize(to_overlap_mask ? stream4 : stream));

    auto num_gradients_comm = state.h_num_gradients_to_communicate[0];
    if (config.global_gpu_rank == 0)
      printf("%d\t #gradients to comm = %ld, #gradients = %ld, rate = %f\n",
        state.step, (long)num_gradients_comm, (long)num_gradients,
        1.0f * num_gradients_comm / num_gradients);

    auto send_allocated_ = state.send_allocated * sizeof(T);
    GUARD_CU(GarenteeAllocation(
      state.send_data, send_allocated_, sizeof(T) * num_gradients_comm));
    if (state.send_allocated < num_gradients_comm)
      state.send_allocated = num_gradients_comm;
    auto recv_allocated_ = state.recv_allocated * sizeof(T);
    GUARD_CU(GarenteeAllocation(
      state.recv_data, recv_allocated_, sizeof(T) * num_gradients_comm));
    if (state.recv_allocated < num_gradients_comm)
      state.recv_allocated = num_gradients_comm;

    T* send_data = (T*)(state.send_data);
    T* recv_data = (T*)(state.recv_data);
    auto global_num_gpus = config.global_num_gpus;
    loop_kernel<<<grid_size, block_size, 0, stream>>>(num_masks,
      [recv_masks, mask_offsets, send_data, global_num_gpus,
      num_gradients, elements]
      __device__ (const SizeT &i)
      {
        uint32_t mask = recv_masks[i];
        if (mask == 0)
          return;

        SizeT offset = i * 32, output_offset = mask_offsets[i];
        int end_j = 32, j = 0, output_count = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;
        while (j < end_j)
        {
          if ((mask & (((uint32_t)1) << j)) == 0)
          {
            j ++;
            continue;
          }
          T element = elements[j + offset];
          if (!isfinite(element * 1.0f))
            element = 0;

          send_data[output_offset + output_count] = element; // / global_num_gpus;
          output_count ++;
          j++;
        }
      });

    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_begin, stream));
    //GUARD_CU2("hipStreamSynchronize after send_data forming",
    //  hipStreamSynchronize(stream));

    GUARD_NCCL2("ncclAllReduce",
      ncclAllReduce(send_data   , (void*)recv_data,
        (size_t)num_gradients_comm,
        PreDefinedValues<T>::NCCLDataType, ncclSum,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));

    GUARD_CU(Memset(output_gradients, 0, num_gradients, Malloc_t::Default, stream));
    loop_kernel<<<grid_size, block_size, 0, stream>>>(num_masks,
      [recv_masks, mask_offsets, recv_data, output_gradients, num_gradients]
      __device__ (const SizeT &i)
      {
        uint32_t mask = recv_masks[i];
        if (mask == 0)
          return;

        SizeT offset = i * 32, output_offset = mask_offsets[i];
        int end_j = 32, j = 0, output_count = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;
        while (j < end_j)
        {
          if ((mask & (((uint32_t)1) << j)) == 0)
          {
            j ++;
            continue;
          }

          output_gradients[j + offset] = recv_data[output_offset + output_count];
          output_count ++;
          j++;
        }
      });

    //GUARD_CU2("hipStreamSynchronize after output_gradient calculation",
    //  hipStreamSynchronize(stream));

    // Updates pervious_verlocity and pervious_accumulated_verlocity
    // Can be overlap with communication
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream2, token -> stream2_begin, 0));
    if (config.use_momentum_correction) {
      for (auto& chunk : chunks) {
        SizeT  chunk_start  = std::get<0>(chunk);
        SizeT  chunk_size   = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);

        T* pervious_verlocity
          = (T*)(state.pervious_verlocity + chunk_offset);
        T* pervious_accumulated_verlocity
          = (T*)(state.pervious_accumulated_verlocity + chunk_offset);

        loop_kernel <<<grid_size, block_size, 0, stream2>>>(chunk_size,
          [recv_masks, chunk_start, chunk_size,
           verlocity, pervious_verlocity,
           accumulated_verlocity, pervious_accumulated_verlocity]
          __device__ (const SizeT &i) {
            //if (i == 0)
            //  printf("gradient [%ld...%ld) \n",
            //    (long)gradient_start_chunk,
            //    (long)(gradient_start_chunk + num_gradients_chunk));
            auto gradient_pos = i + chunk_start;
            auto mask_pos = gradient_pos / 32;
            auto mask = recv_masks[mask_pos];
            auto mask_offset = (gradient_pos & ((uint32_t)31));

            if ((mask & (((uint32_t)1) << mask_offset)) != 0) {
              pervious_verlocity[i] = 0;
              pervious_accumulated_verlocity[i] = 0;
            } else {
              pervious_verlocity[i] = verlocity[gradient_pos];
              pervious_accumulated_verlocity[i]
                = accumulated_verlocity[gradient_pos];
            }
          });
      }
    } else {
      for (auto& chunk : chunks) {
        SizeT  chunk_start  = std::get<0>(chunk);
        SizeT  chunk_size   = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);

        T* pervious_accumulated_gradients
          = (T*)(state.pervious_accumulated_gradients + chunk_offset);
        loop_kernel <<<grid_size, block_size, 0, stream2>>>(chunk_size,
          [recv_masks, chunk_start, chunk_size,
          accumulated_gradients, pervious_accumulated_gradients]
          __device__ (const SizeT &i) {
            auto gradient_pos = i + chunk_start;
            auto mask_pos = gradient_pos / 32;
            auto mask = recv_masks[mask_pos];
            auto mask_offset = (gradient_pos & ((uint32_t)31));

            if ((mask & (((uint32_t)1) << mask_offset)) != 0) {
              pervious_accumulated_gradients[i] = 0;
            } else {
              pervious_accumulated_gradients[i]
                = accumulated_gradients[gradient_pos];
            }
          });
      }
    }
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_finish, stream2));
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream, token -> stream2_finish, 0));
  } // end of if (use_allReduce)

  else {
    // use allGather to communicate
    //auto &samp_counter = state.samp_counter;
    //loop_kernel <<<1, 1, 0, stream>>>((SizeT)1,
    //  [samp_counter] __device__ (const SizeT &i)
    //  {
    //    samp_counter[0] = 0;
    //  });
    //loop_kernel <<<grid_size, block_size, 0, stream>>>(num_samples,
    //  [samp_data, num_samples, samp_counter, threshold] __device__ (const SizeT &i)
    //  {
    //    if (!(samp_data[i] < threshold[0]))
    //    {
    //      atomicAdd(samp_counter, (uint64_t)1);
    //    }
    //  });
    //loop_kernel <<<1, 1, 0, stream>>>((SizeT)1,
    //  [samp_counter] __device__ (const SizeT &i)
    //  {
    //    printf("Recount = %d\n", samp_counter[0]);
    //  });

    // Pick those larger than threshold
    auto &send_counter   = state.send_counter;
    //auto &send_data      = state.send_data;
    auto &send_indices   = state.send_indices;
    auto &send_allocated = state.send_allocated;
    auto send_allocated_ = send_allocated * sizeof(T);
    if (send_counter == NULL) {
      GUARD_CU(Malloc(send_counter, 1));
    }

    GUARD_CU(GarenteeAllocation(
      state.send_data, send_allocated_, target_num * sizeof(T)));
    GUARD_CU(GarenteeAllocation(
      send_indices, send_allocated , target_num));
    if (state.max_gradient == NULL) {
      GUARD_CU(Malloc(state.max_gradient, 1));
    }
    //loop_kernel <<<1, 1, 0, stream>>>((SizeT)1,
    //  [send_counter] __device__ (const SizeT &i)
    //  {
    //    send_counter[0] = 0;
    //  });
    GUARD_CU(Memset(send_counter, 0, 1, Malloc_t::Default, stream));
    GUARD_CU(Memset(state.max_gradient, 0, 1, Malloc_t::Default, stream));

    T* send_data = (T*)(state.send_data);
    // select at most target_num gradients
    //select_kernel
    //  <<<grid_size, block_size, 0, stream>>>
    //  (elements, num_gradients, config.global_num_gpus,
    //  threshold, target_num, send_data, send_indices, send_counter,
    //  state.max_gradient);
    select_kernel3
      <<<grid_size, block_size, 0, stream>>>
      (elements, config.global_num_gpus,
      thresholds, layer_starts, num_layers, target_num,
      send_data, send_indices, send_counter, state.max_gradient);

    // pad if num_slected < target_num
    pad_kernel
      <<<grid_size, block_size, 0, stream>>>
      ((T*)send_data, send_indices, target_num, send_counter, state.max_gradient);

    // Reallocate if not enough
    SizeT recv_count      = target_num * (config.use_hierarchical_allreduce ?
        config.global_num_nodes : config.global_num_gpus);
    auto &recv_allocated  = state.recv_allocated;
    auto  recv_allocated_ = state.recv_allocated * sizeof(T);
    //auto &recv_data       = state.recv_data;
    auto &recv_indices    = state.recv_indices;

    //printf("recv_count = %lld\n", (long long)recv_count);
    GUARD_CU(GarenteeAllocation(
        state.recv_data, recv_allocated_, recv_count * sizeof(T)));
    GUARD_CU(GarenteeAllocation(
        recv_indices, recv_allocated, recv_count));

    //GUARD_CU2("hipStreamSynchronize after send data forming",
    //    hipStreamSynchronize(stream));
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_begin, stream));

    T* recv_data = (T*)(state.recv_data);
    // Collect selected data & indices from all peers
    GUARD_NCCL2("ncclAllGather",
      ncclAllGather(send_data   , (void*)recv_data,
        (size_t)target_num, PreDefinedValues<T       >::NCCLDataType,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));
    GUARD_NCCL2("ncclAllGather",
      ncclAllGather(send_indices, (void*)recv_indices,
        (size_t)target_num, PreDefinedValues<uint32_t>::NCCLDataType,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));
    //GUARD_CU2("hipStreamSynchronize after AllGather",
    //    hipStreamSynchronize(stream));

    //auto &global_gradients_= state.global_gradients;
    //auto &global_allocated = state.global_allocated;
    //GUARD_CU(GarenteeAllocation(
    //  state.global_gradients, global_allocated, num_gradients * sizeof(T)));
    //T* global_gradients = (T*)(state.global_gradients);

    // Post process gradients
    //loop_kernel <<<grid_size, block_size, 0, stream>>>(num_gradients,
    //  [global_gradients] __device__ (const SizeT &i)
    //  {
    //    global_gradients[i] = 0;
    //  });
    GUARD_CU(Memset(output_gradients, 0, num_gradients, Malloc_t::Default, stream));

    // Unpack recv data
    loop_kernel <<<grid_size, block_size, 0, stream>>>(recv_count,
      [recv_data, recv_indices, output_gradients] __device__ (const SizeT &i)
      {
        T     element = recv_data   [i];
        SizeT index   = recv_indices[i];
        if (isValid(index))
          atomicAdd(output_gradients + index, element);
      });

    // Updates pervious_verlocity and pervious_accumulated_verlocity
    // Can be overlap with communication
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream2, token -> stream2_begin, 0));
    if (config.use_momentum_correction) {
      for (auto& chunk : chunks) {
        SizeT  chunk_start  = std::get<0>(chunk);
        SizeT  chunk_size   = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);

        T* pervious_verlocity
          = (T*)(state.pervious_verlocity + chunk_offset);
        T* pervious_accumulated_verlocity
          = (T*)(state.pervious_accumulated_verlocity + chunk_offset);

        loop_kernel <<<grid_size, block_size, 0, stream2>>>(chunk_size,
          [thresholds, chunk_start, chunk_size,
          verlocity, pervious_verlocity,
          accumulated_verlocity, pervious_accumulated_verlocity,
          layer_starts, num_layers]
          __device__ (const SizeT &i) {
            //if (i == 0)
            //  printf("gradient [%ld...%ld) \n",
            //    (long)gradient_start_chunk,
            //    (long)(gradient_start_chunk + num_gradients_chunk));
            auto gradient_pos = i + chunk_start;
            auto v = accumulated_verlocity[gradient_pos];
            int layer = binarySearch(layer_starts, 0, num_layers, gradient_pos);
            if (isfinite(v * 1.0f) && abs(v) > thresholds[layer]) {
              pervious_verlocity[i] = 0;
              pervious_accumulated_verlocity[i] = 0;
            } else {
              pervious_verlocity[i] = verlocity[gradient_pos];
              pervious_accumulated_verlocity[i] = v;
            }
          });
      }
    }
    else {
      for (auto& chunk : chunks) {
        SizeT   chunk_start  = std::get<0>(chunk);
        SizeT   chunk_size   = std::get<1>(chunk);
        size_t  chunk_offset = std::get<2>(chunk);

        T* pervious_accumulated_gradients
          = (T*)(state.pervious_accumulated_gradients + chunk_offset);
        loop_kernel <<<grid_size, block_size, 0, stream2>>>(chunk_size,
          [thresholds, chunk_start, chunk_size,
          accumulated_gradients, pervious_accumulated_gradients,
          layer_starts, num_layers]
          __device__ (const SizeT &i) {
            auto gradient_pos = i + chunk_start;
            auto g = accumulated_gradients[gradient_pos];
            int layer = binarySearch(layer_starts, 0, num_layers, gradient_pos);
            if (isfinite(g * 1.0f) && abs(g) > thresholds[layer]) {
              pervious_accumulated_gradients[i] = 0;
            } else {
              pervious_accumulated_gradients[i] = g;
            }
          });
      }
    }
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_finish, stream2));
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream, token -> stream2_finish, 0));
  }

  if (config.learning_rate_decay_factor > 0 &&
      epoch >= config.num_epochs_per_decay) {
    float learning_rate_adjustment = 1;
    auto epoch_ = epoch;
    while (epoch_ >= config.num_epochs_per_decay)
    {
      learning_rate_adjustment *= config.learning_rate_decay_factor;
      epoch_ -= config.num_epochs_per_decay;
    }
    if (learning_rate_adjustment < config.min_learning_rate_factor)
      learning_rate_adjustment = config.min_learning_rate_factor;
    if (config.global_gpu_rank == 0)
      printf("%ld\t learning_rate_adjustment = %f\n",
        (long)state.step, learning_rate_adjustment);

    loop_kernel <<<grid_size, block_size, 0, stream>>>(num_gradients,
      [learning_rate_adjustment, output_gradients] __device__ (const SizeT &i)
      {
        output_gradients[i] *= learning_rate_adjustment;
      });
  }

  //GUARD_CU2("hipStreamSynchronize after",
  //  hipStreamSynchronize(stream));
  GUARD_CU2("hipEventRecord",
    hipEventRecord(token -> dgc_finish, stream));
  token -> dgc_finished = false;
  state.busy_tokens.push_back(token);

  if (to_overlap_mask)
  {
    //GUARD_CU2("hipStreamSynchronize",
    //    hipStreamSynchronize(stream3));
    GUARD_CU(TryPushMask(2, config, state));
  }
  return retval;
}

// Entry warper function
// hipError_t ClipGradient(
//   ncclDataType_t  gradient_type, // type of gradient
//   void           *gradients,     // GPU pointer to the gradients
//   //uint64_t       *layer_offsets, // gradient layer offsets, on host
//   //int             num_layers,    // The number of layers in the gradients
//   std::vector<std::pair<std::string, uint64_t> > &layers,
//                                 // <name, #elements> of layers
//   DgcConfig      &config,        // DGC configuration
//   DgcState       &state)         // DGC running states
// {
//   typedef uint32_t SizeT;
//   hipError_t retval = hipSuccess;
//
//   switch (gradient_type)
//   {
//   case ncclFloat32:
//     retval = ClipGradient <float> (
//       //(float*)gradients, layer_offsets, num_layers, config, state);
//       (float*)gradients, layers, config, state);
//     break;
//
//   case ncclFloat64:
//     retval = ClipGradient <double> (
//       //(double*)gradients, layer_offsets, num_layers, config, state);
//       (double*)gradients, layers, config, state);
//     break;
//
//   case ncclInt32:
//     retval = ClipGradient <int32_t> (
//       //(int32_t*)gradients, layer_offsets, num_layers, config, state);
//       (int32_t*)gradients, layers, config, state);
//     break;
//
//   case ncclInt64:
//     retval = ClipGradient <int64_t> (
//       //(int64_t*)gradients, layer_offsets, num_layers, config, state);
//       (int64_t*)gradients, layers, config, state);
//     break;
//
//   default:
//     break;
//   }
//   return retval;
// }

hipError_t GradientAllReduce(
  ncclDataType_t  gradient_type, // type of gradient
  void           *input_gradients, // GPU pointer to the input graients
  void           *output_gradients,// GPU pointer to the output gradients
  //uint64_t        num_gradients, // number of gradients
  //std::vector<std::tuple<uint64_t, uint64_t, size_t> >
  //               &offset_map,    // <start, length, offset> mappings for
                                 // continous chunks of gradients
  std::vector<std::pair<std::string, uint64_t> > &layers,
                                 // <name, #elements> of layers
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  typedef uint32_t SizeT;
  hipError_t retval = hipSuccess;

  if (config.use_hierarchical_allreduce &&
      !config.cross_comm_inited) {
    ncclUniqueId nccl_cross_id;
    if (config.global_node_rank == 0) {
      GUARD_NCCL2("ncclGetUniqueId",
        ncclGetUniqueId(&nccl_cross_id));
    }

    GUARD_MPI2("MPI_Bcast",
      MPI_Bcast((void*)&nccl_cross_id, sizeof(nccl_cross_id),
        MPI_BYTE, 0, config.cross_comm));

    ncclComm_t new_nccl_comm;
    GUARD_NCCL2("ncclCommInitRank",
      ncclCommInitRank(&new_nccl_comm, config.global_num_nodes,
        nccl_cross_id, config.global_node_rank));
    config.nccl_cross_comm = new_nccl_comm;

    ncclUniqueId nccl_local_id;
    if (config.local_gpu_rank == 0) {
      GUARD_NCCL2("ncclGetUniqueId",
        ncclGetUniqueId(&nccl_local_id));
    }

    GUARD_MPI2("MPI_Bcast",
      MPI_Bcast((void*)&nccl_local_id, sizeof(nccl_local_id),
        MPI_BYTE, 0, config.local_comm));

    ncclComm_t new_nccl_comm2;
    GUARD_NCCL2("ncclCommInitRank",
      ncclCommInitRank(&new_nccl_comm2, config.local_num_gpus,
        nccl_local_id, config.local_gpu_rank));
    config.nccl_local_comm = new_nccl_comm2;

    GUARD_MPI2("MPI_Barrier",
      MPI_Barrier(config.mpi_comm));
    printf("local = %d of %d, cross = %d of %d, global = %d of %d\n",
        config.local_gpu_rank, config.local_num_gpus,
        config.global_node_rank, config.global_num_nodes,
        config.global_gpu_rank, config.global_num_gpus);
    config.cross_comm_inited = true;
  }

  size_t num_gradients = 0;
  if (config.use_hierarchical_allreduce) {
    for (auto& layer : layers)
      num_gradients += layer.second;

    GUARD_NCCL2("ncclReduce",
      ncclReduce(input_gradients, input_gradients, num_gradients,
        gradient_type, ncclSum, 0, config.nccl_local_comm, config.stream));
  }

  if ((config.use_hierarchical_allreduce && config.local_gpu_rank == 0) ||
      !config.use_hierarchical_allreduce) {
    switch (gradient_type)
    {
    case ncclFloat32:
      retval = GradientAllReduce <float, SizeT> (
        (float*)input_gradients, (float*)output_gradients,
        //(SizeT)num_gradients, offset_map, config, state);
        layers, config, state);
      break;

    case ncclFloat64:
      retval = GradientAllReduce<double, SizeT> (
        (double*)input_gradients, (double*)output_gradients,
        //(SizeT)num_gradients, offset_map, config, state);
        layers, config, state);
      break;

    case ncclInt32:
      retval = GradientAllReduce<int32_t, SizeT> (
        (int32_t*)input_gradients, (int32_t*)output_gradients,
        //(SizeT)num_gradients, offset_map, config, state);
        layers, config, state);
      break;

    case ncclInt64:
      retval = GradientAllReduce<int64_t, SizeT> (
        (int64_t*)input_gradients, (int64_t*)output_gradients,
        //(SizeT)num_gradients, offset_map, config, state);
        layers, config, state);
      break;

    default:
      break;
    }
  }

  if (config.use_hierarchical_allreduce) {
    GUARD_NCCL2("ncclBcast",
      ncclBcast(output_gradients, num_gradients,
        gradient_type, 0, config.nccl_local_comm, config.stream));
  }
  return retval;
}

} // end of namespace dgc
} // end of namespace horovod
