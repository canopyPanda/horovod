#include "hip/hip_runtime.h"
// DGC host function implementations
// by Yuechao Pan
// for NVIDIA

// past compile with following command:
// nvcc -std=c++11 -c -o dgc.cu.o horovod_nvidia/horovod/common/dgc.cu.cc      \
   -x cu -Xcompiler -fPIC -dlink --expt-extended-lambda                        \
   -gencode=arch=compute_70,code=\"sm_70,compute_70\"

//#pragma once

#include <string>
#include <algorithm>
#include <chrono>
#include <thread>
#include <locale>
#include <hiprand/hiprand_kernel.h>
//#include <thrust/sort.h>
//#include <thrust/execution_policy.h>
#include <hipcub/hipcub.hpp>
#include <mpi.h>
#include "dgc.h"
#include "dgc_kernel.cu.cc"

namespace horovod {
namespace dgc {

// Pertector for CUDA calls
#define GUARD_CU2(op_name, op)                                                 \
{                                                                              \
  retval = (op);                                                               \
  if (retval != hipSuccess) {                                                 \
    std::string error_message = std::string(__FILE__) + std::string(":")       \
      + std::to_string(__LINE__) + std::string("(")                            \
      + std::string(op_name) + std::string(") failed: ")                       \
      + hipGetErrorString(retval);                                            \
    fprintf(stderr, "%s\n", error_message.c_str());                            \
    fflush(stderr);                                                            \
    return retval;                                                             \
  }                                                                            \
}

// Pertector for CUDA calls
#define GUARD_CU(op)                                                           \
{                                                                              \
  retval = (op);                                                               \
  if (retval != hipSuccess) {                                                 \
    std::string error_message = std::string(__FILE__) + std::string(":")       \
      + std::to_string(__LINE__) + std::string(" failed: ")                    \
      + hipGetErrorString(retval);                                            \
    fprintf(stderr, "%s\n", error_message.c_str());                            \
    fflush(stderr);                                                            \
    return retval;                                                             \
  }                                                                            \
}

// Pertector for NCCL calls
#define GUARD_NCCL2(op_name, op)                                               \
{                                                                              \
  auto nccl_result = (op);                                                     \
  if (nccl_result != ncclSuccess) {                                            \
    std::string error_message = std::string(__FILE__) + std::string(":")       \
      + std::to_string(__LINE__) + std::string("(")                            \
      + std::string(op_name) + std::string(") failed: ")                       \
      + ncclGetErrorString(nccl_result);                                       \
    fprintf(stderr, "%s\n", error_message.c_str());                            \
    fflush(stderr);                                                            \
    return hipErrorUnknown;                                                   \
  }                                                                            \
}

// Pertector for MPI calls
#define GUARD_MPI2(op_name, op)                                                \
{                                                                              \
  auto mpi_result = (op);                                                      \
  if (mpi_result != MPI_SUCCESS) {                                             \
    char  error_string[MPI_MAX_ERROR_STRING + 1];                              \
    error_string[MPI_MAX_ERROR_STRING] = 0;                                    \
    int   error_length = 0;                                                    \
    MPI_Error_string(mpi_result, error_string, &error_length);                 \
    std::string error_message = std::string(__FILE__) + std::string(":")       \
      + std::to_string(__LINE__) + std::string("(")                            \
      + std::string(op_name) + std::string(") failed: ")                       \
      + std::string(error_string);                                             \
    fprintf(stderr, "%s\n", error_message.c_str());                            \
    fflush(stderr);                                                            \
    return hipErrorUnknown;                                                   \
  }                                                                            \
}

// ****************************
// Memory management
// ****************************

enum Malloc_t
{
  Default,
  Host,
  Managed,
  Raw,
};

// Unified free function
template <typename T>
hipError_t Free(
  T* &ptr,
  Malloc_t malloc_type = Malloc_t::Default)
{
  hipError_t retval = hipSuccess;
  if (ptr == NULL)
    return retval;

  //printf("Freeing @ %p\n", ptr);
  if (malloc_type == Host) {
    GUARD_CU2("hipHostFree",
      hipHostFree(ptr));
  } else if (malloc_type == Default || malloc_type == Managed) {
    GUARD_CU2("hipFree",
      hipFree(ptr));
  } else if (malloc_type == Raw)
    free(ptr);

  //printf("Freed @ %p\n", ptr);
  ptr = NULL;
  return retval;
}

// Unified malloc function
template <typename T>
hipError_t Malloc(
  T* &ptr,
  size_t target,
  Malloc_t malloc_type = Malloc_t::Default,
  unsigned int flags = hipMemAttachGlobal,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;

  size_t size = target * sizeof(T);
  //printf("Allocating %ld x %ld bytes on %s\n", target, sizeof(T),
  //   malloc_type == Default ? "Default" :
  //  (malloc_type == Host    ? "Host" :
  //  (malloc_type == Managed ? "Managed" : "Raw")));

  if (malloc_type == Default) {
    GUARD_CU2("hipMalloc",
      hipMalloc(&ptr, size));
  } else if (malloc_type == Host) {
    GUARD_CU2("hipHostMalloc",
      hipHostMalloc(&ptr, size));
  } else if (malloc_type == Managed) {
    GUARD_CU2("hipMallocManaged",
      hipMallocManaged(&ptr, size, flags));
  } else if (malloc_type == Raw)
    ptr = (T*)malloc(size);

  //printf("Allocated %ld x %ld bytes @ %p\n", target, sizeof(T), ptr);
  return retval;
}

// Unified memcpy function
template <typename T, typename SizeT>
hipError_t Memcpy(
  T* dest,
  T* src,
  SizeT num_elements,
  Malloc_t malloc_type = Malloc_t::Default,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;
  if (num_elements == 0)
    return retval;
  if (dest == NULL || src == NULL)
    return retval;

  if (malloc_type != Raw) {
    if (stream == 0) {
      retval = hipMemcpyAsync(dest, src, sizeof(T) * num_elements,
        hipMemcpyDefault, stream);
    } else {
      retval = hipMemcpy(dest, src, sizeof(T) * num_elements,
        hipMemcpyDefault);
    }
  } else {
    memcpy(dest, src, sizeof(T) * num_elements);
  }
  return retval;
}

// Unified memset function
template <typename T, typename SizeT>
hipError_t Memset(
  T* ptr,
  int value,
  SizeT num_elements,
  Malloc_t malloc_type = Malloc_t::Default,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;
  if (num_elements == 0 || ptr == NULL)
    return retval;

  if (malloc_type != Malloc_t::Raw) {
    if (stream == 0) {
      retval = hipMemset(ptr, value, num_elements * sizeof(T));
    } else {
      retval = hipMemsetAsync(ptr, value, num_elements * sizeof(T), stream);
    }
  } else {
    memset(ptr, value, num_elements * sizeof(T));
  }

  return retval;
}

// Garentee sufficient allocation
template <typename T, typename SizeT>
hipError_t GarenteeAllocation(
  T*      &ptr,
  SizeT   &allocated,
  size_t   target,
  Malloc_t malloc_type = Malloc_t::Default,
  unsigned int flags = hipMemAttachGlobal,
  hipStream_t stream = 0,
  bool     keep_content = false,
  bool     init_to_zero = false)
{
  hipError_t retval = hipSuccess;
  if (allocated >= target)
    return retval;

  if (!keep_content) {
    auto temp_ptr = ptr;
    GUARD_CU(Free<T> (temp_ptr, malloc_type));
    GUARD_CU(Malloc(ptr, target, malloc_type, flags));
    if (init_to_zero) {
      GUARD_CU(Memset(ptr, 0, target, malloc_type, stream));
    }
  }

  else {
    T* temp_ptr = NULL;
    GUARD_CU(Malloc(temp_ptr, target, malloc_type, flags));
    GUARD_CU(Memcpy(temp_ptr, ptr, allocated, malloc_type, stream));
    if (init_to_zero) {
      GUARD_CU(Memset(temp_ptr + allocated, 0, target - allocated,
        malloc_type, stream));
    }
    GUARD_CU(Free(ptr, malloc_type));
    ptr = temp_ptr;
    temp_ptr = NULL;
  }

  allocated = target;
  return retval;
}

template <typename T>
std::string ToString(const T& val)
{
  return std::to_string(val);
}

template <>
std::string ToString(const common::Framework& val)
{
  std::string str = "Unknown";
  if (val == common::Framework::TENSORFLOW)
    str = "TensorFlow";
  else if (val == common::Framework::PYTORCH)
    str = "PyTorch";
  return str;
}

template <typename T>
hipError_t FreePersistent(
  std::string  name,
  T*          &ptr,
  DgcConfig   &config,
  DgcState    &state)
{
  hipError_t retval = hipSuccess;
  //printf("Freeing %s @ %p\n", name.c_str(), ptr);

  std::string key = std::to_string(config.device) + "::"
    + ToString(config.context -> framework()) + "::" + name;
  if (state.memory_table[key].second == 0)
    return retval;
  auto buffer = state.memory_table[key].first;
  ptr = (T*)(buffer -> AccessData(config.context));
  auto status = config.context -> AllocatePersistent(0, &buffer);
  if (!status.ok()) {
    GUARD_CU2("Allocating 0 byte for " + name, hipErrorUnknown);
  }
  state.memory_table[key].first = buffer;
  state.memory_table[key].second = 0;

  //printf("Freed %s @ %p\n", name.c_str(), ptr);
  ptr = NULL;
  return retval;
}

template <typename T>
hipError_t AccessPersistent(
  std::string  name,
  T*          &ptr,
  DgcConfig   &config,
  DgcState    &state)
{
  hipError_t retval = hipSuccess;
  std::string key = std::to_string(config.device) + "::"
    + ToString(config.context -> framework()) + "::" + name;

  if (state.memory_table[key].second != 0) {
    auto& buffer = state.memory_table[key].first;
    ptr = (T*)(buffer -> AccessData(config.context));
  } else {
    ptr = NULL;
  }
  //printf("Accessing %s @ %p\n", name.c_str(), ptr);
  return retval;
}

template <typename T, typename SizeT>
hipError_t MallocPersistent(
  std::string  name,
  T*          &ptr,
  SizeT        num_elements,
  DgcConfig   &config,
  DgcState    &state)
{
  hipError_t retval = hipSuccess;
  std::string str = std::to_string(sizeof(T)) + " * "
    + std::to_string(num_elements) + " bytes for " + name;
  //printf("Allocating %s\n", str.c_str());

  std::string key = std::to_string(config.device) + "::"
    + ToString(config.context -> framework()) + "::" + name;
  auto buffer = state.memory_table[key].first;
  size_t allocated = state.memory_table[key].second;
  size_t request_bytes = sizeof(T) * num_elements;

  if (allocated != 0) {
    ptr = (T*)(buffer -> AccessData(config.context));
    //printf("Warnning: %s has been allocated %ld bytes on GPU %d, ptr = %p. "
    //  "Reallocating %d * %ld = %ld bytes.",
    //  name.c_str(), (long)allocated, config.device, ptr,
    //  sizeof(T), (long)num_elements, (long)request_bytes);

    GUARD_CU(FreePersistent(name, ptr, config, state));
  }

  auto status = config.context -> AllocatePersistent(request_bytes, &buffer);
  if (!status.ok()) {
    GUARD_CU2("Allocating " + str, hipErrorUnknown);
  }

  state.memory_table[key] = std::make_pair(buffer, request_bytes);
  ptr = (T*)(buffer -> AccessData(config.context));

  //printf("Allocated %s, ptr = %p\n", str.c_str(), ptr);
  return retval;
}

template <typename T, typename SizeT>
hipError_t GarenteeAllocationPersistent(
  std::string   name,
  T*           &ptr,
  SizeT         request_num_elements,
  DgcConfig    &config,
  DgcState     &state,
  hipStream_t  stream = 0,
  bool          keep_content = false,
  bool          init_to_zero = false)
{
  hipError_t retval = hipSuccess;

  std::string key = std::to_string(config.device) + "::"
    + ToString(config.context -> framework()) + "::" + name;
  size_t allocated = state.memory_table[key].second;
  size_t request_bytes = sizeof(T) * request_num_elements;
  if (allocated >= request_bytes) {
    ptr = (T*)(state.memory_table[key].first -> AccessData(config.context));
    return retval;
  }

  if (!keep_content || allocated == 0) {
    GUARD_CU(FreePersistent  (name, ptr, config, state));
    GUARD_CU(MallocPersistent(name, ptr, request_num_elements, config, state));
    if (init_to_zero) {
      GUARD_CU(Memset(ptr, 0, request_num_elements, Malloc_t::Default, stream));
    }
  }

  else {
    auto old_buffer = state.memory_table[key].first;
    T* old_ptr = (T*)(old_buffer -> AccessData(config.context));
    SizeT allocated_num_elements = allocated / sizeof(T);
    GUARD_CU(FreePersistent  (name, ptr, config, state));
    GUARD_CU(MallocPersistent(name, ptr, request_num_elements, config, state));

    GUARD_CU(Memcpy(ptr, old_ptr, allocated_num_elements,
      Malloc_t::Default, stream));
    if (init_to_zero) {
      GUARD_CU(Memset(ptr + allocated_num_elements, 0,
        request_num_elements - allocated_num_elements,
        Malloc_t::Default, stream));
    }
    old_ptr = NULL;
    auto status = config.context -> AllocatePersistent(0, &old_buffer);
    if (!status.ok()) {
      GUARD_CU2("Allocating 0 byte for " + name, hipErrorUnknown);
    }
  }
  return retval;
}

// ****************************
// DGC Functions
// ****************************

void str2bool(std::string str, bool &val)
{
  if (str == "True")
    val = true;
  if (str == "False")
    val = false;
}

// Setting config parameters
void DgcConfig::Set(std::string key, std::string value)
{
  //printf("Setting %s to %s\n", key.c_str(), value.c_str());

  if (key == "dgc_sparsity_warmup_epochs")
    warmup_epochs = std::stoi(value);

  else if (key == "dgc_init_sparsity")
    init_sparsity = std::stod(value);

  else if (key == "dgc_final_sparsity")
    final_sparsity = std::stod(value);

  else if (key == "dgc_sampling_rate")
    sampling_rate = std::stod(value);

  else if (key == "dgc_rand_seed")
    rand_seed = std::stoi(value);

  else if (key == "dgc_grid_size")
    grid_size = std::stoi(value);

  else if (key == "dgc_block_size")
    block_size = std::stoi(value);

  else if (key == "dgc_min_sampling_num")
    min_sampling_num = std::stoi(value);

  else if (key == "dgc_local_gradient_clipping")
    str2bool(value, local_gradient_clipping);

  else if (key == "dgc_clipping_threshold")
    clipping_threshold = std::stof(value);

  else if (key == "dgc_use_allreduce")
    str2bool(value, use_allReduce);

  else if (key == "dgc_use_hierarchical_allreduce")
    str2bool(value, use_hierarchical_allreduce);

  else if (key == "dgc_overlap_mask_allreduce")
    str2bool(value, overlap_mask_allreduce);

  else if (key == "dgc_learning_rate_decay_factor")
    learning_rate_decay_factor = std::stof(value);

  else if (key == "dgc_num_epochs_per_decay")
    num_epochs_per_decay = std::stof(value);

  else if (key == "dgc_min_learning_rate_factor")
    min_learning_rate_factor = std::stof(value);

  else if (key == "dgc_flush_steps")
    flush_steps = std::stoi(value);

  else if (key == "dgc_use_momentum_correction")
    str2bool(value, use_momentum_correction);

  else if (key == "dgc_use_gradient_accumulation")
    str2bool(value, use_gradient_accumulation);

  else if (key == "dgc_smooth_sparsity")
    str2bool(value, smooth_sparsity);

  else if (key == "dgc_skip_epochs")
    skip_epochs = std::stof(value);

  else if (key == "momentum")
    momentum = std::stof(value);

  else if (key == "num_examples_per_epoch")
    num_examples_per_epoch = std::stoi(value);

  else if (key == "batch_size")
    batch_size_per_gpu = std::stoi(value);

}

// Get configuration from environmental variables
void DgcConfig::ReadFromENV()
{
  const std::string env_list[] = {
    "dgc_sparsity_warmup_epochs",
    "dgc_init_sparsity",
    "dgc_final_sparsity",
    "dgc_sampling_rate",
    "dgc_rand_seed",
    "dgc_grid_size",
    "dgc_block_size",
    "dgc_min_sampling_num",
    "dgc_local_gradient_clipping",
    "dgc_clipping_threshold",
    "dgc_use_allreduce",
    "dgc_use_hierarchical_allreduce",
    "dgc_overlap_mask_allreduce",
    "dgc_learning_rate_decay_factor",
    "dgc_num_epochs_per_decay",
    "dgc_min_learning_rate_factor",
    "dgc_flush_steps",
    "dgc_use_momentum_correction",
    "dgc_use_gradient_accumulation",
    "dgc_smooth_sparsity",
    "dgc_skip_epochs",
    "momentum",
    "num_examples_per_epoch",
    "batch_size"};
  const int num_parameters = 24;
  auto& f = std::use_facet<std::ctype<char>>(std::locale());

  for (int i = 0; i < num_parameters; i++) {
    std::string env_name = env_list[i];
    std::string env_name_upper = env_name;
    f.toupper(&env_name_upper[0], &env_name_upper[0] + env_name_upper.size());
    char* value = std::getenv(env_name_upper.c_str());
    std::string value_str = "";

    if (value == NULL) {
      value = std::getenv(("NO" + env_name_upper).c_str());
      if (value != NULL) {
        value_str = std::string(value);
        if (value_str == "True")
          value_str = "False";
        else if (value_str == "False")
          value_str = "True";
      }
    } else
      value_str = std::string(value);

    if (value != NULL) {
      Set(env_name, value_str);
    }
  }
}

// Unified sort function
template <typename T, typename SizeT, typename Compare>
hipError_t Sort(
  T           *elements,
  SizeT        num_elements,
  Compare      compare,
  DgcConfig   &config,
  DgcState    &state,
  std::string  temp_storage_name = "temp_storage",
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default)
{
  hipError_t retval = hipSuccess;

  if (malloc_type == Raw) {
    std::sort(elements, elements + num_elements, compare);
    return retval;
  }

  // Not using thrust for now;
  // if sort becomes performance bottleneck, change to cub
  // Note: thrust::sort hit a bug that produced illegal memory access
  //thrust::sort(thrust::cuda::par.on(stream),
  //  elements, elements + num_elements, compare);

  // Cub sorting
  size_t required_bytes = 0;
  GUARD_CU(hipcub::DeviceRadixSort::SortKeys(
    (char*)NULL, required_bytes,
    elements, elements,
    num_elements, 0, sizeof(T) * 8, stream));

  char* temp_storage = NULL;
  GUARD_CU(GarenteeAllocationPersistent(temp_storage_name,
    temp_storage, required_bytes, config, state));

  GUARD_CU(hipcub::DeviceRadixSort::SortKeys(
    temp_storage, required_bytes,
    elements, elements,
    num_elements, 0, sizeof(T) * 8, stream));

  return retval;
}

// Sort with default less than comparator
template <typename T, typename SizeT>
hipError_t Sort(
  T      *elements,
  SizeT   num_elements,
  DgcConfig &config,
  DgcState  &state,
  std::string temp_storage_name = "temp_storage",
  hipStream_t stream = 0,
  Malloc_t malloc_type = Malloc_t::Default)
{
  return Sort(elements, num_elements,
    [] __host__ __device__ (T a, T b){return a < b;},
    config, state, temp_storage_name, stream, malloc_type);
}

// Sort template switch
template <typename SizeT>
hipError_t Sort(
  ncclDataType_t nccl_type,
  void        *elements,
  SizeT        num_elements,
  DgcConfig   &config,
  DgcState    &state,
  std::string  temp_storage_name = "temp_storage",
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default)
{
  hipError_t retval = hipSuccess;

  switch (nccl_type) {
  case ncclFloat32:
    retval = Sort<float> ((float*)elements, num_elements,
      config, state, temp_storage_name, stream, malloc_type);
    break;

  case ncclFloat64:
    retval = Sort<double> ((double*)elements, num_elements,
      config, state, temp_storage_name, stream, malloc_type);
    break;

  case ncclInt32:
    retval = Sort<int32_t> ((int32_t*)elements, num_elements,
      config, state, temp_storage_name, stream, malloc_type);
    break;

  case ncclInt64:
    retval = Sort<int64_t> ((int64_t*)elements, num_elements,
      config, state, temp_storage_name, stream, malloc_type);
    break;

  default:
    break;
  }
  return retval;
}

// Segmeted Sort
template <typename T, typename SizeT, typename Compare>
hipError_t SegSort(
  T           *elements,
  SizeT        num_elements,
  SizeT       *seg_starts,
  int          num_segments,
  Compare      compare,
  DgcConfig   &config,
  DgcState    &state,
  std::string  temp_storage_name = "temp_storage",
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default)
{
  hipError_t retval = hipSuccess;

  if (malloc_type == Raw) {
    for (int i = 0; i < num_segments; i++)
      std::sort(elements + seg_starts[i], elements + seg_starts[i+1], compare);
    return retval;
  }

  // Cub segmented sort
  size_t required_bytes = 0;
  GUARD_CU(hipcub::DeviceSegmentedRadixSort::SortKeys(
    (char*)NULL, required_bytes,
    elements, elements, num_elements,
    num_segments, seg_starts, seg_starts + 1,
    0, sizeof(T) * 8, stream));

  char* temp_storage = NULL;
  GUARD_CU(GarenteeAllocationPersistent(temp_storage_name,
    temp_storage, required_bytes, config, state));

  GUARD_CU(hipcub::DeviceSegmentedRadixSort::SortKeys(
    temp_storage, required_bytes,
    elements, elements, num_elements,
    num_segments, seg_starts, seg_starts + 1,
    0, sizeof(T) * 8, stream));

  return retval;
}

// Segmented sort with default less than comparator
template <typename T, typename SizeT>
hipError_t SegSort(
  T      *elements,
  SizeT   num_elements,
  SizeT  *seg_starts,
  int     num_segments,
  DgcConfig &config,
  DgcState  &state,
  std::string temp_storage_name = "temp_storage",
  hipStream_t stream = 0,
  Malloc_t malloc_type = Malloc_t::Default)
{
  return SegSort(elements, num_elements, seg_starts, num_segments,
    [] __host__ __device__ (T a, T b){ return a < b;},
    config, state, temp_storage_name, stream, malloc_type);
}

// Local gradient clipping
template <typename T>
hipError_t ClipGradient(
  T          *gradients,
  std::vector<std::pair<std::string, uint64_t> > &layers,
             // <name, #elements> of layers
  DgcConfig  &config,
  DgcState   &state,
  DgcToken   *token)
{
  hipError_t retval = hipSuccess;

  // skip first step, because total number of layers are unknown
  if (state.step == 0)
    return retval;

  int num_layers = layers.size();
  char* temp_storage = NULL;
  GUARD_CU(GarenteeAllocationPersistent("temp_storage",
    temp_storage, sizeof(T) * 2 * num_layers
      + sizeof(uint32_t) * (num_layers + 1), config, state));
  GUARD_CU(GarenteeAllocation(token -> h_layer_starts,
    token -> h_layer_starts_allocated, num_layers + 1, Malloc_t::Host));

  uint32_t start_counter = 0;
  for (int i = 0; i < num_layers; i++) {
    token -> h_layer_starts[i] = start_counter;
    start_counter += layers[i].second;
  }
  token -> h_layer_starts[num_layers] = start_counter;

  T* sums         = (T*)(temp_storage);
  T* coefficients = (T*)(temp_storage + sizeof(T) * num_layers);
  uint32_t* layer_starts
    = (uint32_t*)(temp_storage + sizeof(T) * 2 * num_layers);
  auto stream     = config.stream;
  int  grid_size  = config.grid_size;
  int  block_size = config.block_size;
  auto clipping_threshold = config.clipping_threshold;

  GUARD_CU(Memset(sums, 0, num_layers, Malloc_t::Default, stream));
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(layer_starts, token -> h_layer_starts,
      sizeof(uint32_t) * (num_layers + 1), hipMemcpyHostToDevice, stream));

  // Get per -layer L2 norms
  L2norm_kernel<<<grid_size, block_size, 0, stream>>>(
    gradients, layer_starts, num_layers, sums);

  int total_num_layers = state.layer_offset_bytes.size();
  uint64_t total_num_gradients = state.offset_byte_counter / sizeof(T);

  // Get per-layer coefficients
  loop_kernel<<<grid_size, block_size, 0, stream>>>(num_layers,
    [sums, coefficients, total_num_layers, total_num_gradients,
    clipping_threshold, layer_starts]
    __device__ (const int &layer) {
      coefficients[layer] = clipping_threshold /
        (sqrt(sums[layer]) * total_num_layers + 1e-6);
    });

  // Update gradients
  loop_kernel<<<grid_size, block_size, 0, stream>>>(start_counter,
    [layer_starts, gradients, coefficients, num_layers]
    __device__ (const uint32_t &i) {
      int layer = binarySearch(layer_starts, 0, num_layers, i);
      auto coefficient = coefficients[layer];
      if (coefficient < 1)
        gradients[i] *= coefficient;
    });

  return retval;
}

hipError_t DgcToken::Init()
{
  hipError_t retval = hipSuccess;
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> dgc_finish    ), hipEventDisableTiming));
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> stream2_begin ), hipEventDisableTiming));
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> stream2_finish), hipEventDisableTiming));
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> stream3_begin ), hipEventDisableTiming));
  return retval;
}

hipError_t DgcToken::isFinished(bool &finished, int check)
{
  hipError_t retval = hipSuccess;
  if (this -> dgc_finished) {
    finished = true;
    return retval;
  }

  retval = hipEventQuery(this -> dgc_finish);
  if (retval == hipSuccess) {
    finished = true;
    this -> dgc_finished = true;
  } else if (retval == hipErrorNotReady) {
    finished = false;
    retval = hipSuccess;
  }
  return retval;
}

hipError_t MaskToken::Init()
{
  hipError_t retval = hipSuccess;
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> d2h_finish), hipEventDisableTiming));
  GUARD_CU2("hipEventCreateWithFlags",
    hipEventCreateWithFlags(&(this -> h2d_finish), hipEventDisableTiming));
  return retval;
}

hipError_t MaskToken::isFinished(bool &finished, int check)
{
  hipError_t retval = hipSuccess;
  if (check == 0) {
    if (this -> d2h_finished) {
      finished = true;
      return retval;
    }
    retval = hipEventQuery(this -> d2h_finish);
    if (retval == hipSuccess) {
      finished = true;
      this -> d2h_finished = true;
    } else if (retval == hipErrorNotReady) {
      finished = false;
      retval = hipSuccess;
    }
  }

  else if (check == 1)
  {
    if (this -> mpi_finished) {
      finished = true;
      return retval;
    }
    if (!this -> mpi_started) {
      finished = false;
      return retval;
    }

    int flag = 0;
    GUARD_MPI2("MPI_Test",
      MPI_Test(&(this -> mpi_request), &flag, MPI_STATUS_IGNORE));
    if (flag) {
      //printf("\t token = %p, received %ld masks from MPI, "
      //       "first 3: %#X, %#X, %#X\n",
      //  this, (long)this -> num_masks,
      //  this -> h_recv_masks[0], this -> h_recv_masks[1],
      //  this -> h_recv_masks[2]);
      finished = true;
      this -> mpi_finished = true;
      this -> mpi_started = false;
    } else {
      finished = false;
    }
  }

  else if (check == 2) {
    if (this -> h2d_finished) {
      finished = true;
      return retval;
    }
    retval = hipEventQuery(this -> h2d_finish);
    if (retval == hipSuccess) {
      finished = true;
      this -> h2d_finished = true;
    } else if (retval == hipErrorNotReady) {
      finished = false;
      retval = hipSuccess;
    }
  }

  return retval;
}

template <typename TokenT>
hipError_t GetToken(
  std::list<TokenT*> &free_queue,
  std::list<TokenT*> &busy_queue,
  TokenT* &token,
  int check = 0)
{
  hipError_t retval = hipSuccess;
  if (free_queue.size() != 0) {
    token = free_queue.front();
    free_queue.pop_front();
    return retval;
  }

  if (busy_queue.size() != 0 && check != -1) {
    auto first_token = busy_queue.front();
    bool finished = false;
    GUARD_CU(first_token -> isFinished(finished, check));
    if (finished) {
      token = first_token;
      busy_queue.pop_front();
      return retval;
    }
  }

  token = new TokenT;
  GUARD_CU(token -> Init());
  return retval;
}

// Wait for mask to be ready on host and push MPIAllReduce
hipError_t TryPushMask(
  int             max_requests_allowed_waiting,
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  hipError_t retval = hipSuccess;
  if (max_requests_allowed_waiting != 0) {
    int total_num_layers = 0;
    for (auto &token : state.d2h_mask_queue) {
      total_num_layers += token -> num_layers;
    }
    // if the whole model is waiting, push everything out
    if (total_num_layers >= state.layer_offset_bytes.size())
      max_requests_allowed_waiting = 0;
  }

  while (state.d2h_mask_queue.size() > max_requests_allowed_waiting) {
    auto token = state.d2h_mask_queue.front();
    GUARD_CU2("hipEventSynchronize",
      hipEventSynchronize(token -> d2h_finish));
    token -> d2h_finished = true;

    state.d2h_mask_queue.pop_front();
    GUARD_MPI2("MPI_Iallreduce",
      MPI_Iallreduce(
        token -> h_send_masks, token -> h_recv_masks, (int)token -> num_masks,
        PreDefinedValues<uint32_t>::getMpiDataType(), MPI_BOR,
        config.use_hierarchical_allreduce ? config.cross_comm : config.mpi_comm,
        &(token -> mpi_request)));
    token -> mpi_started  = true;
    token -> mpi_finished = false;
    //printf("%ld\t token = %p, %ld masks pushed to MPI, "
    //       "first 3: %#X, %#X, %#X\n",
    //  (long)state.step, token, (long)token -> num_masks,
    //  token -> h_send_masks[0], token -> h_send_masks[1],
    //  token -> h_send_masks[2]);
    state.mpi_mask_queue.push_back(token);
  }
  return retval;
}

// Learning rate adjustment via gradient, only use if really necessary
template <typename T, typename SizeT>
hipError_t LearningRateAdjustment(
  T              *gradients,
  SizeT           num_gradients,
  uint64_t        epoch,
  DgcConfig      &config,
  DgcState       &state)
{
  hipError_t retval = hipSuccess;
  float learning_rate_adjustment = 1;
  auto epoch_ = epoch;
  while (epoch_ >= config.num_epochs_per_decay)
  {
    learning_rate_adjustment *= config.learning_rate_decay_factor;
    epoch_ -= config.num_epochs_per_decay;
  }
  if (learning_rate_adjustment < config.min_learning_rate_factor)
    learning_rate_adjustment = config.min_learning_rate_factor;
  //if (config.global_gpu_rank == 0)
  //  printf("%ld\t learning_rate_adjustment = %f\n",
  //    (long)state.step, learning_rate_adjustment);

  loop_kernel <<<config.grid_size, config.block_size, 0, config.stream>>>(
    num_gradients,
    [learning_rate_adjustment, gradients] __device__ (const SizeT &i)
    {
      gradients[i] *= learning_rate_adjustment;
    });

  return retval;
}

// Main DGC routine
template <typename T, typename SizeT>
hipError_t GradientAllReduce(
  T              *input_gradients,     // GPU pointer to the input_gradients
  T              *output_gradients,     // GPU pointer to the output_gradients
  std::vector<std::pair<std::string, uint64_t> > &layers,
                                  // <name, #elements> of layers
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  hipError_t retval = hipSuccess;
  auto  block_size   = config.block_size;
  auto  grid_size    = config.grid_size;
  auto  stream       = config.stream;
  int   num_layers   = layers.size();
  SizeT num_gradients = 0;

  // find the step number
  for (auto& layer : layers) {
    auto name = layer.first;
    auto counter_it = state.step_counters.find(name);
    if (counter_it == state.step_counters.end())
      state.step_counters[name] = 0;
    else {
      auto step = counter_it -> second;
      counter_it -> second ++;
      if (state.step < step)
        state.step = step;
    }

    num_gradients += layer.second;
  }

  // Determine the epoch number
  uint64_t num_examples_per_step
    = config.batch_size_per_gpu * config.global_num_gpus;
  uint64_t steps_per_epoch
    = config.num_examples_per_epoch / num_examples_per_step;
  if (steps_per_epoch * num_examples_per_step < config.num_examples_per_epoch)
    steps_per_epoch ++;
  float    epoch_f  = state.step * 1.0 / steps_per_epoch;
  uint64_t epoch    = (uint64_t)epoch_f;

  // if bypass both momentum correction and gradient accumulation
  if (!config.use_momentum_correction && !config.use_gradient_accumulation) {
    GUARD_NCCL2("ncclAllReduce",
      ncclAllReduce(input_gradients, output_gradients, num_gradients,
      PreDefinedValues<T>::NCCLDataType, ncclSum,
      config.use_hierarchical_allreduce ?
      config.nccl_cross_comm : config.nccl_comm, stream));

    GUARD_CU(LearningRateAdjustment(output_gradients,
      num_gradients, epoch, config, state));
    return retval;
  }

  // Calcuate sparsity based on epoch number
  double sparsity   = config.final_sparsity;
  if ((config.skip_epochs > 0 &&
       epoch_f < config.skip_epochs + config.warmup_epochs) ||
    (!(config.skip_epochs > 0) && epoch_f < config.warmup_epochs)) {
    auto init_comm_rate = 1 - config.init_sparsity;
    auto final_comm_rate = 1 - config.final_sparsity;
    if (config.skip_epochs > 0 && epoch_f < config.skip_epochs) {
      sparsity = config.init_sparsity;
    } else if (config.smooth_sparsity) {
      auto comm_rate = init_comm_rate * exp(
        log(final_comm_rate / init_comm_rate)
        / config.warmup_epochs
        * (config.skip_epochs > 0 ? (epoch_f - config.skip_epochs) : epoch_f));
      sparsity = 1 - comm_rate;
    } else {
      auto comm_rate = init_comm_rate * exp(
        log(final_comm_rate / init_comm_rate)
        / config.warmup_epochs
        * (config.skip_epochs > 0 ? (epoch - config.skip_epochs) : epoch));
      sparsity = 1 - comm_rate;
    }

    //if (epoch * steps_per_epoch == state.step && config.global_gpu_rank == 0)
    //if (config.global_gpu_rank == 0)
    //  printf("Epoch %ld, Step %ld, sparsity = %lf\n",
    //    epoch, state.step, sparsity);
  }
  SizeT  target_num = num_gradients * (1 - sparsity);

  // Prepare token and streams
  DgcToken *token = NULL;
  GUARD_CU(GetToken(state.free_tokens, state.busy_tokens, token));
  if (config.stream2 == 0) {
    int greatest_priority;
    GUARD_CU2("hipDeviceGetStreamPriorityRange",
      hipDeviceGetStreamPriorityRange(NULL, &greatest_priority));
    GUARD_CU2("hipStreamCreateWithPriority",
      hipStreamCreateWithPriority(&(config.stream2), hipStreamNonBlocking,
        greatest_priority));
    GUARD_CU2("hipStreamCreateWithPriority",
      hipStreamCreateWithPriority(&(config.stream3), hipStreamNonBlocking,
        greatest_priority));
    GUARD_CU2("hipStreamCreateWithPriority",
      hipStreamCreateWithPriority(&(config.stream4), hipStreamNonBlocking,
        greatest_priority));
  }
  auto stream2 = config.stream2;
  auto stream3 = config.stream3;
  auto stream4 = config.stream4;

  if (config.local_gradient_clipping)
    GUARD_CU(ClipGradient(input_gradients, layers, config, state, token));

  // find which step is currently in and look for unallocated layers
  std::vector<std::pair<std::string, uint64_t> > layers_to_allocate;
  SizeT num_gradients_to_allocate = 0;
  for (auto &layer : layers)
  {
    auto name = layer.first;
    auto offset_it = state.layer_offset_bytes.find(name);
    if (offset_it == state.layer_offset_bytes.end()) {
      layers_to_allocate.push_back(std::make_pair(layer.first, layer.second));
      num_gradients_to_allocate += layer.second;
    }
  }

  // allocate new layers
  char* state_pervious_verlocity = NULL;
  char* state_pervious_accumulated_verlocity = NULL;
  char* state_pervious_accumulated_gradients = NULL;
  char* state_pervious_comm_steps = NULL;
  if (num_gradients_to_allocate > 0) {
    if (config.use_momentum_correction) {
      GUARD_CU(GarenteeAllocationPersistent("pervious_verlocity",
        state_pervious_verlocity,
        state.offset_byte_counter + sizeof(T) * num_gradients_to_allocate,
        config, state, stream, true, true));
      GUARD_CU(GarenteeAllocationPersistent("pervious_accumulated_verlocity",
        state_pervious_accumulated_verlocity,
        state.offset_byte_counter + sizeof(T) * num_gradients_to_allocate,
        config, state, stream, true, true));
    } else {
      GUARD_CU(GarenteeAllocationPersistent("pervious_accumulated_gradients",
        state_pervious_accumulated_gradients,
        state.offset_byte_counter + sizeof(T) * num_gradients_to_allocate,
        config, state, stream, true, true));
      GUARD_CU(GarenteeAllocationPersistent("pervious_comm_steps",
        state_pervious_comm_steps,
        state.offset_byte_counter / sizeof(T) * sizeof(uint32_t)
          + sizeof(uint32_t) * num_gradients_to_allocate,
        config, state, stream, true, true));
    }
    for (auto& layer : layers_to_allocate) {
      state.layer_offset_bytes[layer.first] = state.offset_byte_counter;
      state.offset_byte_counter += layer.second * sizeof(T);
    }
  }
  GUARD_CU(AccessPersistent("pervious_verlocity",
    state_pervious_verlocity, config, state));
  GUARD_CU(AccessPersistent("pervious_accumulated_verlocity",
    state_pervious_accumulated_verlocity, config, state));
  GUARD_CU(AccessPersistent("pervious_accumulated_gradients",
    state_pervious_accumulated_gradients, config, state));
  GUARD_CU(AccessPersistent("pervious_comm_steps",
    state_pervious_comm_steps, config, state));
  GUARD_CU(GarenteeAllocation(token -> h_layer_starts,
    token -> h_layer_starts_allocated, num_layers + 1, Malloc_t::Host));

  // find continous layers as
  // <start, size, offset> of chunks
  std::vector<std::tuple<SizeT, SizeT, size_t> > chunks;
  size_t chunk_offset_bytes = state.layer_offset_bytes[layers.begin() -> first];
  SizeT  layer_start = 0;
  SizeT  chunk_start = 0;
  SizeT  chunk_size  = 0;
  for (int i = 0; i < num_layers; i++) {
    auto &layer = layers[i];
    token -> h_layer_starts[i] = layer_start;
    if (chunk_offset_bytes + chunk_size * sizeof(T) !=
      state.layer_offset_bytes[layer.first]) {
      // mismatch, means new layer starts
      chunks.push_back(std::make_tuple(
        chunk_start, chunk_size, chunk_offset_bytes));
      chunk_size  = 0;
      chunk_start = layer_start;
      chunk_offset_bytes = state.layer_offset_bytes[layer.first];
    }

    chunk_size  += layer.second;
    layer_start += layer.second;
  } // end of for layers
  token -> h_layer_starts[num_layers] = layer_start;
  if (chunk_size != 0)
    chunks.push_back(std::make_tuple(
      chunk_start, chunk_size, chunk_offset_bytes));

  uint32_t *layer_starts = NULL;
  GUARD_CU(GarenteeAllocationPersistent("layer_starts",
    layer_starts, num_layers + 1, config, state));
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(layer_starts, token -> h_layer_starts,
      sizeof(uint32_t) * (num_layers + 1), hipMemcpyHostToDevice, stream));

  // Memory allocation and type conversion
  T* verlocity = NULL;
  T* accumulated_verlocity = NULL;
  T* accumulated_gradients = NULL;
  if (config.use_momentum_correction) {
    GUARD_CU(GarenteeAllocationPersistent("verlocity",
      verlocity, num_gradients, config, state));
    GUARD_CU(GarenteeAllocationPersistent("accumulated_verlocity",
      accumulated_verlocity, num_gradients, config, state));
  } else {
    GUARD_CU(GarenteeAllocationPersistent("accumulated_gradients",
      accumulated_gradients, num_gradients, config, state));
  }

  T* elements = NULL;
  if (config.use_momentum_correction) {
    // momentum correction by chunks
    for (auto& chunk : chunks) {
      SizeT chunk_start = std::get<0>(chunk);
      SizeT chunk_size  = std::get<1>(chunk);
      size_t chunk_offset = std::get<2>(chunk);

      T* pervious_verlocity
        = (T*)(state_pervious_verlocity + chunk_offset);
      T* pervious_accumulated_verlocity
        = (T*)(state_pervious_accumulated_verlocity + chunk_offset);
      auto &momentum = config.momentum;

      loop_kernel<<<grid_size, block_size, 0, stream>>>(chunk_size,
        [momentum, input_gradients, chunk_start,
        pervious_verlocity, verlocity,
        accumulated_verlocity, pervious_accumulated_verlocity]
        __device__ (const SizeT &i) {
          auto pos = i + chunk_start;
          auto u = pervious_verlocity[i] * momentum + input_gradients[pos];
          accumulated_verlocity[pos] = pervious_accumulated_verlocity[i] + u;
          verlocity[pos] = u;
        });
    }
    elements = accumulated_verlocity;
  }

  else {
    // accumulate gradients
    for (auto& chunk : chunks) {
      SizeT chunk_start = std::get<0>(chunk);
      SizeT chunk_size  = std::get<1>(chunk);
      size_t chunk_offset = std::get<2>(chunk);

      T* pervious_accumulated_gradients
        = (T*)(state_pervious_accumulated_gradients + chunk_offset);

      loop_kernel<<<grid_size, block_size, 0, stream>>>(chunk_size,
        [input_gradients, chunk_start,
        accumulated_gradients, pervious_accumulated_gradients]
        __device__ (const SizeT &i) {
          auto pos = i + chunk_start;
          auto g = pervious_accumulated_gradients[i] + input_gradients[pos];
          accumulated_gradients[pos] = g;
        });
    }
    elements = accumulated_gradients;
  }

  // Prepare for mask communication overlapping
  bool to_overlap_mask = config.use_allReduce && config.overlap_mask_allreduce;
  if (to_overlap_mask) {
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream3_begin, stream));
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream3, token -> stream3_begin, 0));
  }

  // Communicate all gradients if it's a flushing step
  bool to_flush = false;
  if (config.skip_epochs > 0 && epoch_f < config.skip_epochs)
    to_flush = true;
  else if (config.flush_steps > 0) {
    if ((state.step >= config.flush_steps) &&
        (state.step % config.flush_steps) == 0)
      to_flush = true;
  }
  if (to_flush) {
    //printf("%ld\t Flushing %ld elements\n",
    //  (long)state.step, (long)num_gradients);

    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_begin, stream));
    GUARD_NCCL2("ncclAllReduce",
      ncclAllReduce(elements, output_gradients,
        (size_t)num_gradients, PreDefinedValues<T>::NCCLDataType, ncclSum,
        config.use_hierarchical_allreduce ?
        config.nccl_cross_comm : config.nccl_comm, stream));

    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream2, token -> stream2_begin, 0));
    if (config.use_momentum_correction) {
      //if (!(config.skip_epochs > 0) || (epoch_f > config.skip_epochs)) {
      {
        for (auto& chunk : chunks) {
          SizeT  chunk_start  = std::get<0>(chunk);
          SizeT  chunk_size   = std::get<1>(chunk);
          size_t chunk_offset = std::get<2>(chunk);

          T* pervious_verlocity
            = (T*)(state_pervious_verlocity + chunk_offset);
          T* pervious_accumulated_verlocity
            = (T*)(state_pervious_accumulated_verlocity + chunk_offset);

          GUARD_CU(Memset(pervious_verlocity,
            0, chunk_size, Malloc_t::Default, stream2));
          GUARD_CU(Memset(pervious_accumulated_verlocity,
            0, chunk_size, Malloc_t::Default, stream2));
        }
      }
    }

    else {
      for (auto& chunk : chunks) {
        SizeT chunk_start = std::get<0>(chunk);
        SizeT chunk_size  = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);

        T* pervious_accumulated_gradients
          = (T*)(state_pervious_accumulated_gradients + chunk_offset);
        GUARD_CU(Memset(pervious_accumulated_gradients,
          0, chunk_size, Malloc_t::Default, stream2));

        uint32_t* pervious_comm_steps
          = (uint32_t*)(state_pervious_comm_steps
            + (chunk_offset / sizeof(T) * sizeof(uint32_t)));
        auto step = state.step;
        loop_kernel<<<grid_size, block_size, 0, stream2>>>(chunk_size,
          [step, pervious_comm_steps] __device__ (const SizeT &i) {
            pervious_comm_steps[i] = step;
          });
      }
    }
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_finish, stream2));

    if (config.learning_rate_decay_factor > 0 &&
        epoch >= config.num_epochs_per_decay) {
      GUARD_CU(LearningRateAdjustment(output_gradients,
        num_gradients, epoch, config, state));
    }

    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream, token -> stream2_finish, 0));
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> dgc_finish, stream));
    token -> dgc_finished = false;
    state.busy_tokens.push_back(token);

    return retval;
  }

  // Sampling
  uint32_t *samp_starts = NULL;
  GUARD_CU(GarenteeAllocationPersistent("samp_starts",
    samp_starts, num_layers + 1, config, state));
  GUARD_CU(GarenteeAllocation(token -> h_samp_starts,
    token -> h_samp_starts_allocated, num_layers + 1, Malloc_t::Host));
  uint32_t samp_counter = 0;
  // Find number of samples for each layer
  for (int i = 0; i < num_layers; i++) {
    auto &layer = layers[i];
    token -> h_samp_starts[i] = samp_counter;

    uint32_t num_samples = 0;
    if (config.sampling_rate < 1 &&
        layer.second > config.min_sampling_num) {

      num_samples = layer.second * config.sampling_rate;
      if (num_samples < config.min_sampling_num)
        num_samples = config.min_sampling_num;
      uint32_t num_selected_samples = config.min_gradients_comm_per_layer
        * config.sampling_rate;
      if (num_selected_samples < config.min_selected_samples_per_layer)
        num_selected_samples = config.min_selected_samples_per_layer;
      if (num_samples < num_selected_samples * 1.0f / (1 - sparsity)) {
        num_samples = num_selected_samples * 1.0f / (1 - sparsity);
      }
      if (num_samples > layer.second)
        num_samples = layer.second;
    }

    else { // no sampling
      num_samples = layer.second;
    }
    samp_counter += num_samples;
  }
  token -> h_samp_starts[num_layers] = samp_counter;
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(samp_starts, token -> h_samp_starts,
      sizeof(uint32_t) * (num_layers + 1), hipMemcpyHostToDevice,
      (to_overlap_mask ? stream3 : stream)));

  // Prepare rand states
  hiprandState *rand_states = NULL;
  GUARD_CU(AccessPersistent("rand_states", rand_states, config, state));
  auto &rand_seed   = config.rand_seed;
  if (rand_states == NULL) {
    GUARD_CU(MallocPersistent("rand_states", rand_states,
      grid_size * block_size, config, state));

    loop_kernel
      <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(
      (SizeT)grid_size * block_size,
      [rand_states, rand_seed] __device__ (const SizeT &i){
        hiprand_init(rand_seed, i, 0, rand_states + i);
      });
  }

  T* samp_data = NULL;
  GUARD_CU(GarenteeAllocationPersistent("samp_data",
    samp_data, samp_counter, config, state));
  sample_kernel2
    <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(
    elements, num_gradients,
    layer_starts, num_layers,
    samp_starts, samp_data, rand_states);

  // Sort the samples
  GUARD_CU(SegSort(samp_data, samp_counter, samp_starts, num_layers,
    config, state, "temp_storage",
    (to_overlap_mask ? stream3 : stream), Malloc_t::Default));

  //loop_kernel<<<1, 1, 0, (to_overlap_mask ? stream3 : stream)>>>((SizeT)1,
  //  [threshold, samp_data, num_samples, sparsity] __device__ (const SizeT &i){
  //    SizeT pos = num_samples * sparsity;
  //    if (pos >= num_samples)
  //      pos = num_samples - 1;
  //    threshold[0] = samp_data[pos];
  //    //printf("selecting samp[%d] from [%d] {%f, %f, ... %f, %f, %f, ... %f, %f}\n",
  //    //  pos, num_samples,
  //    //  num_samples > 0 ? samp_data[0] : -1,
  //    //  num_samples > 1 ? samp_data[1] : -1,
  //    //  num_samples + 1 > pos  && pos > 0 ? samp_data[pos - 1] : -1,
  //    //  num_samples > pos && pos >= 0 ? samp_data[pos] : -1,
  //    //  num_samples > pos + 1 && pos + 1 >= 0 ? samp_data[pos + 1] : -1,
  //    //  num_samples > 1 ? samp_data[num_samples - 2] : -1,
  //    //  num_samples > 0 ? samp_data[num_samples - 1] : -1);
  //  });

  auto &min_selected_samples_per_layer = config.min_gradients_comm_per_layer;
  T *thresholds = NULL;
  GUARD_CU(GarenteeAllocationPersistent("thresholds",
    thresholds, num_layers, config, state));

  // Get the per-layer thresholds
  loop_kernel
    <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(
      num_layers, [thresholds, samp_data, samp_starts, sparsity,
      min_selected_samples_per_layer]
    __device__ (const int &layer) {
      auto samp_start = samp_starts[layer];
      auto samp_end   = samp_starts[layer + 1];
      auto samp_size  = samp_end - samp_start;
      SizeT pos = samp_size * sparsity;
      if (pos >= samp_size)
        pos = samp_size;
      if (min_selected_samples_per_layer < samp_size &&
        pos > samp_size - min_selected_samples_per_layer)
        pos = samp_size - min_selected_samples_per_layer;
      thresholds[layer] = samp_data[samp_start + pos];
    });

  if (config.use_allReduce) {
    // use allReduce on mask to communicate
    SizeT num_masks = num_gradients / MASK_BITS;
    if (num_masks * MASK_BITS < num_gradients)
      num_masks ++;

    // Garentee sufficient memory allocation
    MaskT   * send_masks   = NULL;
    MaskT   * recv_masks   = NULL;
    GUARD_CU(GarenteeAllocationPersistent("send_masks",
      send_masks, num_masks, config, state));
    GUARD_CU(GarenteeAllocationPersistent("recv_masks",
      recv_masks, num_masks, config, state));

    if (!config.overlap_mask_allreduce) {
      auto mask_allocated_ = state.mask_allocated;
      GUARD_CU(GarenteeAllocation(state.h_send_masks, mask_allocated_,
        num_masks, Malloc_t::Host));
      mask_allocated_ = state.mask_allocated;
      GUARD_CU(GarenteeAllocation(state.h_recv_masks, mask_allocated_,
        num_masks, Malloc_t::Host));
      if (state.mask_allocated < num_masks)
        state.mask_allocated = num_masks;
    }

    uint32_t *mask_counters = NULL;
    uint32_t *mask_offsets  = NULL;
    GUARD_CU(GarenteeAllocationPersistent("mask_counters",
      mask_counters, num_masks + 1, config, state));
    GUARD_CU(GarenteeAllocationPersistent("mask_offsets",
      mask_offsets, num_masks + 1, config, state));

    if (state.h_num_gradients_to_communicate == NULL)
        GUARD_CU(Malloc(state.h_num_gradients_to_communicate, 1, Malloc_t::Host));

    // Prepare the mask
    loop_kernel
      <<<grid_size, block_size, 0, (to_overlap_mask ? stream3 : stream)>>>(
        num_masks, [send_masks, num_gradients, thresholds,
        layer_starts, num_layers, elements]
      __device__ (const SizeT &i) {
        MaskT mask = 0;
        SizeT offset = i << LOG_MASK_BITS;
        int end_j = MASK_BITS, j = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;

        while (j < end_j) {
          auto pos = j + offset;
          T element = elements[pos];
          int layer = binarySearch(layer_starts, 0, num_layers, pos);
          if (!isfinite(element * 1.0f)) {
            j ++;
            continue;
          }

          if (!(abs(element) < thresholds[layer])) {
            mask |= (((MaskT)1) << j);
          }
          j++;
        }
        send_masks[i] = mask;
      });

    if (config.overlap_mask_allreduce) {
      MaskToken *mask_token = NULL;

      // Get token and allocate host space
      GUARD_CU(GetToken(state.free_mask_tokens, state.h2d_mask_queue,
        mask_token, 2));
      auto mask_allocated_ = mask_token -> mask_allocated;
      GUARD_CU(GarenteeAllocation(mask_token -> h_send_masks, mask_allocated_,
        num_masks, Malloc_t::Host));
      mask_allocated_ = mask_token -> mask_allocated;
      GUARD_CU(GarenteeAllocation(mask_token -> h_recv_masks, mask_allocated_,
        num_masks, Malloc_t::Host));
      if (mask_token -> mask_allocated < num_masks)
        mask_token -> mask_allocated = num_masks;

      // Move the send mask from GPU to CPU
      GUARD_CU2("hipMemcpyAsync",
        hipMemcpyAsync(mask_token -> h_send_masks, send_masks,
          sizeof(MaskT) * num_masks, hipMemcpyDeviceToHost, stream3));
      GUARD_CU2("hipEventRecord",
        hipEventRecord(mask_token -> d2h_finish, stream3));
      mask_token -> d2h_finished = false;

      // Record the token for every layer
      auto &current_layer_records = state.layer_records[state.step % 2];
      uint32_t layer_start = 0;
      for (auto &layer : layers) {
        auto name = layer.first;
        current_layer_records[name].token = mask_token;
        current_layer_records[name].layer_start = layer_start;
        layer_start += layer.second;
      }
      mask_token -> num_masks = num_masks;
      mask_token -> num_layers = layers.size();
      if ((!(config.skip_epochs > 0) &&
           state.step + 1 == config.overlap_skip_steps) ||
          (config.skip_epochs > 0) &&
           state.step + 1 == int(config.skip_epochs * steps_per_epoch)
             + config.overlap_skip_steps)
        mask_token -> num_layers_produced = layers.size() * 2;
      else
        mask_token -> num_layers_produced = layers.size();
      mask_token -> num_layers_comsumed = 0;
      state.d2h_mask_queue.push_back(mask_token);

      bool to_skip = false;
      if (!(config.skip_epochs > 0) && state.step < config.overlap_skip_steps)
        to_skip = true;
      if (config.skip_epochs > 0 && state.step < config.overlap_skip_steps
        + int(config.skip_epochs * steps_per_epoch))
        to_skip = true;
      if (to_skip)
      {
        // Force sync mask communication for the first few steps
        GUARD_CU(TryPushMask(0, config, state));
      }

      // wait for the mask from pervious step
      bool all_layers_ready = false;
      int pervious_step_index = (to_skip) ?
        (state.step % 2) : ((state.step + 1) % 2);
      auto &pervious_layer_records = state.layer_records[pervious_step_index];

      while (!all_layers_ready) {
        all_layers_ready = true;
        for (auto &layer : layers) {
          auto name   = layer.first;
          auto record = pervious_layer_records[name];
          bool finished = false;
          GUARD_CU(record.token -> isFinished(finished, 1));
          if (!finished) {
            all_layers_ready = false;
            break;
          }
        }

        if (!all_layers_ready) {
          std::this_thread::sleep_for(std::chrono::microseconds(10));
        }
      }

      // Reuse temp_storage to hold masks before bit-swift copy to recv_masks
      size_t request_bytes = sizeof(MaskT) * (num_masks + layers.size() * 2);
      char* temp_storage2 = NULL;
      GUARD_CU(GarenteeAllocationPersistent("temp_storage2",
        temp_storage2, request_bytes, config, state));
      MaskT* temp_masks_ = (MaskT*)temp_storage2;

      GUARD_CU(
        Memset(recv_masks + num_masks - 1, 0, 1, Malloc_t::Default, stream4));
      // move to GPU with bit swift
      SizeT chunk_start = 0;
      SizeT chunk_size = 0;
      SizeT temp_start = 0;
      uint32_t chunk_num_layers = 0;
      SizeT pervious_chunk_start = 0;
      MaskToken *current_token = NULL;
      for (auto i = 0; i <= layers.size(); i++) {
        if (layers.empty())
          break;
        auto layer = layers[(i == layers.size()) ? i - 1 : i];
        auto name = layer.first;
        auto layer_size = layer.second;
        auto record = pervious_layer_records[name];
        bool new_chunk = false;

        if (i == layers.size()) {
        current_token = record.token;
          new_chunk = true;
        } else if (current_token == NULL) {
          new_chunk = false;
          current_token = record.token;
          pervious_chunk_start = record.layer_start;
        } else if (current_token != record.token)
          new_chunk = true;
        else if (pervious_chunk_start + chunk_size != record.layer_start)
          new_chunk = true;

        if (new_chunk) {
          if (chunk_size != 0) {
            SizeT dest_mask_start  = chunk_start / MASK_BITS;
            SizeT dest_mask_offset = chunk_start % MASK_BITS;
            int dest_mask_end    = (chunk_start + chunk_size) / MASK_BITS;
            if (dest_mask_end * MASK_BITS != chunk_start + chunk_size)
              dest_mask_end += 1;
            SizeT dest_mask_size   = dest_mask_end - dest_mask_start;
            MaskT *dest_masks   = recv_masks + dest_mask_start;

            SizeT src_mask_start   = pervious_chunk_start / MASK_BITS;
            SizeT src_mask_end     = (pervious_chunk_start + chunk_size) / MASK_BITS;
            int src_mask_offset  = pervious_chunk_start % MASK_BITS;
            if (src_mask_end * MASK_BITS != pervious_chunk_start + chunk_size)
              src_mask_end += 1;
            SizeT src_mask_size = src_mask_end - src_mask_start;
            MaskT *temp_masks = temp_masks_ + temp_start;
            int ro = src_mask_offset - dest_mask_offset; // relative offset

            //printf("%ld\t token = %p, Copy with bitswift: src = %ld + %ld, src_size = %ld, "
            //  "dest = %ld + %ld, dest_size = %ld, temp_start = %ld, ro = %d, "
            //  "token -> num_masks = %ld, chunk_size = %ld\n",
            //  (long)state.step, current_token,
            //  (long)src_mask_start, (long)src_mask_offset, (long)src_mask_size,
            //  (long)dest_mask_start, (long)dest_mask_offset, (long)dest_mask_size,
            //  (long)temp_start, ro, (long)current_token -> num_masks, (long)chunk_size);
            GUARD_CU2("hipMemcpyAsync",
              hipMemcpyAsync(temp_masks,
                current_token -> h_recv_masks + src_mask_start,
                sizeof(MaskT) * src_mask_size,
                hipMemcpyHostToDevice, stream4));

            loop_kernel<<<grid_size, block_size, 0, stream4>>>(dest_mask_size,
              [temp_masks, dest_masks, dest_mask_size, ro,
              dest_mask_offset, src_mask_offset, chunk_size]
              __device__ (const SizeT &i){
                MaskT dest_mask = 0, mask0 = 0, mask1 = 0;
                if (i != 0 && i+1 != dest_mask_size) {
                  if (ro > 0) {
                    // move src_mask to the right
                    mask0  = temp_masks[i];
                    mask1  = temp_masks[i+1];
                    // (32-ro) bits from mask0
                    dest_mask = mask0 >> ro;
                    // ro bits from mask1
                    dest_mask |= (mask1 & ((MaskT(1) << ro) -1)) << (MASK_BITS-ro);
                  } else if (ro < 0) {
                    // move src_mask to the left
                    mask0 = temp_masks[i-1];
                    mask1 = temp_masks[i];
                    // -ro bits from mask0
                    dest_mask = mask0 >> (MASK_BITS + ro);
                    // (32+ro) bits from mask1
                    dest_mask |= (mask1 & ((MaskT(1) << (MASK_BITS + ro))-1)) << (-ro);
                  } else {
                    // direct copy
                    dest_mask = temp_masks[i];
                  }
                }

                else if (i == 0) {
                  // front
                  int num_gradients_in_first_mask = MASK_BITS - dest_mask_offset;
                  if (num_gradients_in_first_mask > chunk_size)
                    num_gradients_in_first_mask = chunk_size;
                  SizeT pervious_pos = src_mask_offset;
                  dest_mask = dest_masks[i];
                  for (int k = 0; k < num_gradients_in_first_mask; k++) {
                    mask0 = temp_masks[pervious_pos >> LOG_MASK_BITS];
                    mask1 = (mask0 >> (pervious_pos & MASK_BITS_MASK)) & MaskT(1);
                    mask1 = mask1 << (k + dest_mask_offset);
                    dest_mask |= mask1;
                    pervious_pos ++;
                  }
                }

                else { // i+1 == dest_mask_size
                  // back
                  int num_gradients_in_last_mask
                    = (dest_mask_offset + chunk_size) & MASK_BITS_MASK;
                  SizeT pervious_pos = src_mask_offset + chunk_size
                    - num_gradients_in_last_mask;
                  dest_mask = dest_masks[i];
                  for (int k = 0; k < num_gradients_in_last_mask; k++)
                  {
                    mask0 = temp_masks[pervious_pos >> LOG_MASK_BITS];
                    mask1 = (mask0 >> (pervious_pos & MASK_BITS_MASK)) & MaskT(1);
                    mask1 = mask1 << k;
                    dest_mask |= mask1;
                    pervious_pos ++;
                  }
                }

                dest_masks[i] = dest_mask;
              });

            temp_start = temp_start + src_mask_size;
            current_token -> num_layers_comsumed += chunk_num_layers;
            if (current_token -> num_layers_comsumed
              == current_token -> num_layers_produced) {
              GUARD_CU2("hipEventRecord",
                hipEventRecord(current_token -> h2d_finish, stream4));
              current_token -> h2d_finished = false;
            }
          }

          if (i == layers.size())
            break;
          pervious_chunk_start = record.layer_start;
          current_token = record.token;
          chunk_start += chunk_size;
          chunk_size = 0;
          chunk_num_layers = 0;
        }

        chunk_size += layer_size;
        chunk_num_layers ++;
      }

      while (!state.mpi_mask_queue.empty()) {
        auto first_token = state.mpi_mask_queue.front();
        if (first_token -> num_layers_comsumed !=
          first_token -> num_layers_produced)
          break;

        state.mpi_mask_queue.pop_front();
        state.h2d_mask_queue.push_back(first_token);
      }
    } // enf of if (config.overlap_mask_allreduce)

    else {
      // not overlapping mask allreduce
      GUARD_CU2("hipMemcpyAsync",
        hipMemcpyAsync(state.h_send_masks, send_masks,
          sizeof(MaskT) * num_masks, hipMemcpyDeviceToHost, stream));
      GUARD_CU2("hipStreamSynchronize after mask",
        hipStreamSynchronize(stream));

      GUARD_MPI2("MPI_Allreduce",
        MPI_Allreduce(state.h_send_masks, state.h_recv_masks,
          (int)num_masks, PreDefinedValues<MaskT>::getMpiDataType(), MPI_BOR,
          config.use_hierarchical_allreduce ? config.cross_comm : config.mpi_comm));

      GUARD_CU2("hipMemcpyAsync",
        hipMemcpyAsync(recv_masks, state.h_recv_masks,
          sizeof(MaskT) * num_masks, hipMemcpyHostToDevice, stream));
    }

    // Count received mask
    loop_kernel<<<grid_size, block_size, 0,
      (to_overlap_mask ? stream4 : stream)>>>(num_masks,
      [recv_masks, mask_counters] __device__ (const SizeT &i) {
        mask_counters[i] = __popc(recv_masks[i]);
      });

    // Use inclusive sum to calculate the offsets for gradient compaction
    size_t required_bytes = 0;
    GUARD_CU(hipcub::DeviceScan::InclusiveSum(
      (char*)NULL, required_bytes,
      mask_counters, mask_offsets + 1, num_masks,
      (to_overlap_mask ? stream4 : stream)));
    char *temp_storage2 = NULL;
    GUARD_CU(GarenteeAllocationPersistent("temp_storage2",
      temp_storage2, required_bytes, config, state));

    GUARD_CU(Memset(mask_offsets, 0, 1, Malloc_t::Default,
      (to_overlap_mask ? stream4 : stream)));
    GUARD_CU(hipcub::DeviceScan::InclusiveSum(
      temp_storage2, required_bytes,
      mask_counters, mask_offsets + 1, num_masks,
      (to_overlap_mask ? stream4 : stream)));

    // Get the total number of gradients selected
    GUARD_CU2("hipMemcpyAsync",
      hipMemcpyAsync(state.h_num_gradients_to_communicate,
        mask_offsets + num_masks, sizeof(MaskT),
        hipMemcpyDeviceToHost, to_overlap_mask ? stream4 : stream));
    GUARD_CU2("hipStreamSynchronize after InclusiveSum",
      hipStreamSynchronize(to_overlap_mask ? stream4 : stream));

    auto num_gradients_comm = state.h_num_gradients_to_communicate[0];
    if (config.global_gpu_rank == 0)
      printf("%d\t #gradients to comm = %ld, #gradients = %ld, rate = %f\n",
        state.step, (long)num_gradients_comm, (long)num_gradients,
        1.0f * num_gradients_comm / num_gradients);

    T* send_data = NULL;
    T* recv_data = NULL;
    GUARD_CU(GarenteeAllocationPersistent("send_data",
      send_data, num_gradients_comm, config, state));
    GUARD_CU(GarenteeAllocationPersistent("recv_data",
      recv_data, num_gradients_comm, config, state));

    // Compact gradients
    auto global_num_gpus = config.global_num_gpus;
    loop_kernel<<<grid_size, block_size, 0, stream>>>(num_masks,
      [recv_masks, mask_offsets, send_data, global_num_gpus,
      num_gradients, elements]
      __device__ (const SizeT &i) {

        uint32_t mask = recv_masks[i];
        if (mask == 0)
          return;
        SizeT offset = i << LOG_MASK_BITS, output_offset = mask_offsets[i];
        int end_j = MASK_BITS, j = 0, output_count = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;

        while (j < end_j) {
          if ((mask & (((MaskT)1) << j)) == 0) {
            j ++;
            continue;
          }
          T element = elements[j + offset];
          if (!isfinite(element * 1.0f))
            element = 0;

          send_data[output_offset + output_count] = element; // / global_num_gpus;
          output_count ++;
          j++;
        }
      });

    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_begin, stream));

    GUARD_NCCL2("ncclAllReduce",
      ncclAllReduce(send_data   , (void*)recv_data,
        (size_t)num_gradients_comm,
        PreDefinedValues<T>::NCCLDataType, ncclSum,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));

    // Unpack received gradients and indices
    GUARD_CU(
      Memset(output_gradients, 0, num_gradients, Malloc_t::Default, stream));
    loop_kernel<<<grid_size, block_size, 0, stream>>>(num_masks,
      [recv_masks, mask_offsets, recv_data, output_gradients, num_gradients]
      __device__ (const SizeT &i) {
        uint32_t mask = recv_masks[i];
        if (mask == 0)
          return;

        SizeT offset = i << LOG_MASK_BITS, output_offset = mask_offsets[i];
        int end_j = MASK_BITS, j = 0, output_count = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;
        while (j < end_j) {
          if ((mask & (((MaskT)1) << j)) == 0) {
            j ++;
            continue;
          }

          output_gradients[j + offset]
            = recv_data[output_offset + output_count];
          output_count ++;
          j++;
        }
      });

    // Updates pervious_verlocity and pervious_accumulated_verlocity
    // Can be overlap with communication
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream2, token -> stream2_begin, 0));
    if (config.use_momentum_correction) {
      for (auto& chunk : chunks) {
        SizeT  chunk_start  = std::get<0>(chunk);
        SizeT  chunk_size   = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);

        T* pervious_verlocity
          = (T*)(state_pervious_verlocity + chunk_offset);
        T* pervious_accumulated_verlocity
          = (T*)(state_pervious_accumulated_verlocity + chunk_offset);

        loop_kernel <<<grid_size, block_size, 0, stream2>>>(chunk_size,
          [recv_masks, chunk_start, chunk_size,
           verlocity, pervious_verlocity,
           accumulated_verlocity, pervious_accumulated_verlocity]
          __device__ (const SizeT &i) {
            auto gradient_pos = i + chunk_start;
            auto mask_pos = gradient_pos >> LOG_MASK_BITS;
            auto mask = recv_masks[mask_pos];
            auto mask_offset = (gradient_pos & MASK_BITS_MASK);

            if ((mask & (((MaskT)1) << mask_offset)) != 0) {
              pervious_verlocity[i] = 0;
              pervious_accumulated_verlocity[i] = 0;
            } else {
              pervious_verlocity[i] = verlocity[gradient_pos];
              pervious_accumulated_verlocity[i]
                = accumulated_verlocity[gradient_pos];
            }
          });
      }
    }

    else { // gradient accumulation
      //if (config.global_gpu_rank == 0)
      //  printf("Dividing output gradients\n");
      for (auto& chunk : chunks) {
        SizeT  chunk_start  = std::get<0>(chunk);
        SizeT  chunk_size   = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);
        auto   step = state.step;

        T* pervious_accumulated_gradients
          = (T*)(state_pervious_accumulated_gradients + chunk_offset);
        uint32_t* pervious_comm_steps
          = (uint32_t*)(state_pervious_comm_steps
            + (chunk_offset / sizeof(T) * sizeof(uint32_t)));

        loop_kernel <<<grid_size, block_size, 0, stream>>>(chunk_size,
          [recv_masks, chunk_start, chunk_size,
          accumulated_gradients, pervious_accumulated_gradients,
          step, pervious_comm_steps, output_gradients]
          __device__ (const SizeT &i) {
            auto gradient_pos = i + chunk_start;
            auto mask_pos = gradient_pos >> LOG_MASK_BITS;
            auto mask = recv_masks[mask_pos];
            auto mask_offset = (gradient_pos & MASK_BITS_MASK);

            if ((mask & (((MaskT)1) << mask_offset)) != 0) {
              pervious_accumulated_gradients[i] = 0;
              auto step_gap = step - pervious_comm_steps[i];
              if (step_gap != 0)
                output_gradients[gradient_pos] /= step_gap;
              pervious_comm_steps[i] = step;
            } else {
              pervious_accumulated_gradients[i]
                = accumulated_gradients[gradient_pos];
            }
          });
      }
    }
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_finish, stream2));
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream, token -> stream2_finish, 0));
  } // end of if (use_allReduce)

  else {
    // use allGather to communicate
    // Pick those larger than threshold

    // Prepare send buffer
    uint64_t* send_counter = NULL;
    uint32_t* send_indices = NULL;
    T*        send_data    = NULL;
    T*        max_gradient = NULL;
    GUARD_CU(GarenteeAllocationPersistent("send_counter",
      send_counter, 1, config, state));
    GUARD_CU(GarenteeAllocationPersistent("send_data",
      send_data, target_num, config, state));
    GUARD_CU(GarenteeAllocationPersistent("send_indices",
      send_indices, target_num, config, state));
    GUARD_CU(GarenteeAllocationPersistent("max_gradient",
      max_gradient, 1, config, state));
    GUARD_CU(Memset(send_counter, 0, 1, Malloc_t::Default, stream));
    GUARD_CU(Memset(max_gradient, 0, 1, Malloc_t::Default, stream));

    // Compact gradients
    select_kernel3
      <<<grid_size, block_size, 0, stream>>>
      (elements, config.global_num_gpus,
      thresholds, layer_starts, num_layers, target_num,
      send_data, send_indices, send_counter, max_gradient);

    // pad if num_slected < target_num
    pad_kernel
      <<<grid_size, block_size, 0, stream>>>
      ((T*)send_data, send_indices, target_num, send_counter, max_gradient);

    // Reallocate if not enough
    SizeT recv_count      = target_num * (config.use_hierarchical_allreduce ?
        config.global_num_nodes : config.global_num_gpus);
    T*        recv_data = NULL;
    uint32_t* recv_indices = NULL;
    GUARD_CU(GarenteeAllocationPersistent("recv_data",
      recv_data, recv_count, config, state));
    GUARD_CU(GarenteeAllocationPersistent("recv_indices",
      recv_indices, recv_count, config, state));
    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_begin, stream));

    // Collect selected data & indices from all peers
    GUARD_NCCL2("ncclAllGather",
      ncclAllGather(send_data   , (void*)recv_data,
        (size_t)target_num, PreDefinedValues<T       >::NCCLDataType,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));
    GUARD_NCCL2("ncclAllGather",
      ncclAllGather(send_indices, (void*)recv_indices,
        (size_t)target_num, PreDefinedValues<uint32_t>::NCCLDataType,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));

    // Unpack received gradients and indices
    GUARD_CU(
      Memset(output_gradients, 0, num_gradients, Malloc_t::Default, stream));
    loop_kernel <<<grid_size, block_size, 0, stream>>>(recv_count,
      [recv_data, recv_indices, output_gradients] __device__ (const SizeT &i) {
        T     element = recv_data   [i];
        SizeT index   = recv_indices[i];
        if (isValid(index))
          atomicAdd(output_gradients + index, element);
      });

    // Updates pervious_verlocity and pervious_accumulated_verlocity
    // Can be overlap with communication
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream2, token -> stream2_begin, 0));
    if (config.use_momentum_correction) {
      for (auto& chunk : chunks) {
        SizeT  chunk_start  = std::get<0>(chunk);
        SizeT  chunk_size   = std::get<1>(chunk);
        size_t chunk_offset = std::get<2>(chunk);

        T* pervious_verlocity
          = (T*)(state_pervious_verlocity + chunk_offset);
        T* pervious_accumulated_verlocity
          = (T*)(state_pervious_accumulated_verlocity + chunk_offset);

        loop_kernel <<<grid_size, block_size, 0, stream2>>>(chunk_size,
          [thresholds, chunk_start, chunk_size,
          verlocity, pervious_verlocity,
          accumulated_verlocity, pervious_accumulated_verlocity,
          layer_starts, num_layers]
          __device__ (const SizeT &i) {
            auto gradient_pos = i + chunk_start;
            auto v = accumulated_verlocity[gradient_pos];
            int layer = binarySearch(layer_starts, 0, num_layers, gradient_pos);
            if (isfinite(v * 1.0f) && abs(v) > thresholds[layer]) {
              pervious_verlocity[i] = 0;
              pervious_accumulated_verlocity[i] = 0;
            } else {
              pervious_verlocity[i] = verlocity[gradient_pos];
              pervious_accumulated_verlocity[i] = v;
            }
          });
      }
    }
    else {
      for (auto& chunk : chunks) {
        SizeT   chunk_start  = std::get<0>(chunk);
        SizeT   chunk_size   = std::get<1>(chunk);
        size_t  chunk_offset = std::get<2>(chunk);

        T* pervious_accumulated_gradients
          = (T*)(state_pervious_accumulated_gradients + chunk_offset);
        loop_kernel <<<grid_size, block_size, 0, stream2>>>(chunk_size,
          [thresholds, chunk_start, chunk_size,
          accumulated_gradients, pervious_accumulated_gradients,
          layer_starts, num_layers]
          __device__ (const SizeT &i) {
            auto gradient_pos = i + chunk_start;
            auto g = accumulated_gradients[gradient_pos];
            int layer = binarySearch(layer_starts, 0, num_layers, gradient_pos);
            if (isfinite(g * 1.0f) && abs(g) > thresholds[layer]) {
              pervious_accumulated_gradients[i] = 0;
            } else {
              pervious_accumulated_gradients[i] = g;
            }
          });
      }
    }

    GUARD_CU2("hipEventRecord",
      hipEventRecord(token -> stream2_finish, stream2));
    GUARD_CU2("hipStreamWaitEvent",
      hipStreamWaitEvent(stream, token -> stream2_finish, 0));
  }

  if (config.learning_rate_decay_factor > 0 &&
      epoch >= config.num_epochs_per_decay) {
    GUARD_CU(LearningRateAdjustment(output_gradients,
      num_gradients, epoch, config, state));
  }

  GUARD_CU2("hipEventRecord",
    hipEventRecord(token -> dgc_finish, stream));
  token -> dgc_finished = false;
  state.busy_tokens.push_back(token);

  if (to_overlap_mask) {
    GUARD_CU(TryPushMask(2, config, state));
  }
  return retval;
}

// Entry warper function
hipError_t GradientAllReduce(
  ncclDataType_t  gradient_type, // type of gradient
  void           *input_gradients, // GPU pointer to the input graients
  void           *output_gradients,// GPU pointer to the output gradients
  std::vector<std::pair<std::string, uint64_t> > &layers,
                                 // <name, #elements> of layers
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  typedef uint32_t SizeT;
  hipError_t retval = hipSuccess;

  if (config.use_hierarchical_allreduce &&
      !config.cross_comm_inited) {
    ncclUniqueId nccl_cross_id;
    if (config.global_node_rank == 0) {
      GUARD_NCCL2("ncclGetUniqueId",
        ncclGetUniqueId(&nccl_cross_id));
    }

    GUARD_MPI2("MPI_Bcast",
      MPI_Bcast((void*)&nccl_cross_id, sizeof(nccl_cross_id),
        MPI_BYTE, 0, config.cross_comm));

    ncclComm_t new_nccl_comm;
    GUARD_NCCL2("ncclCommInitRank",
      ncclCommInitRank(&new_nccl_comm, config.global_num_nodes,
        nccl_cross_id, config.global_node_rank));
    config.nccl_cross_comm = new_nccl_comm;

    ncclUniqueId nccl_local_id;
    if (config.local_gpu_rank == 0) {
      GUARD_NCCL2("ncclGetUniqueId",
        ncclGetUniqueId(&nccl_local_id));
    }

    GUARD_MPI2("MPI_Bcast",
      MPI_Bcast((void*)&nccl_local_id, sizeof(nccl_local_id),
        MPI_BYTE, 0, config.local_comm));

    ncclComm_t new_nccl_comm2;
    GUARD_NCCL2("ncclCommInitRank",
      ncclCommInitRank(&new_nccl_comm2, config.local_num_gpus,
        nccl_local_id, config.local_gpu_rank));
    config.nccl_local_comm = new_nccl_comm2;

    GUARD_MPI2("MPI_Barrier",
      MPI_Barrier(config.mpi_comm));
    //printf("local = %d of %d, cross = %d of %d, global = %d of %d\n",
    //    config.local_gpu_rank, config.local_num_gpus,
    //    config.global_node_rank, config.global_num_nodes,
    //    config.global_gpu_rank, config.global_num_gpus);
    config.cross_comm_inited = true;
  }

  size_t num_gradients = 0;
  if (config.use_hierarchical_allreduce) {
    for (auto& layer : layers)
      num_gradients += layer.second;

    GUARD_NCCL2("ncclReduce",
      ncclReduce(input_gradients, input_gradients, num_gradients,
        gradient_type, ncclSum, 0, config.nccl_local_comm, config.stream));
  }

  if ((config.use_hierarchical_allreduce && config.local_gpu_rank == 0) ||
      !config.use_hierarchical_allreduce) {
    switch (gradient_type) {
    case ncclFloat32:
      retval = GradientAllReduce <float, SizeT> (
        (float*)input_gradients, (float*)output_gradients,
        layers, config, state);
      break;

    case ncclFloat64:
      retval = GradientAllReduce<double, SizeT> (
        (double*)input_gradients, (double*)output_gradients,
        layers, config, state);
      break;

    case ncclInt32:
      retval = GradientAllReduce<int32_t, SizeT> (
        (int32_t*)input_gradients, (int32_t*)output_gradients,
        layers, config, state);
      break;

    case ncclInt64:
      retval = GradientAllReduce<int64_t, SizeT> (
        (int64_t*)input_gradients, (int64_t*)output_gradients,
        layers, config, state);
      break;

    default:
      break;
    }

    if (retval)
      return retval;
  }

  if (config.use_hierarchical_allreduce) {
    GUARD_NCCL2("ncclBcast",
      ncclBcast(output_gradients, num_gradients,
        gradient_type, 0, config.nccl_local_comm, config.stream));
  }
  return retval;
}

} // end of namespace dgc
} // end of namespace horovod
