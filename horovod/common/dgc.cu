#include "hip/hip_runtime.h"
// DGC host function implementations
// by Yuechao Pan
// for NVIDIA

// past compile with following command:
// nvcc -std=c++11 -c -o dgc.cu.o horovod_nvidia/horovod/common/dgc.cu.cc      \
   -x cu -Xcompiler -fPIC -dlink --expt-extended-lambda -gencode=arch=compute_70,code=\"sm_70,compute_70\"

//#pragma once

#include <string>
#include <algorithm>
#include <hiprand/hiprand_kernel.h>
//#include <thrust/sort.h>
//#include <thrust/execution_policy.h>
#include <hipcub/hipcub.hpp>
#include <mpi.h>
#include "dgc.h"
#include "dgc_kernel.cu.cc"

namespace horovod {
namespace dgc {

#define GUARD_CU2(op_name, op)                                                 \
{                                                                              \
  do {                                                                         \
    retval = (op);                                                             \
    if (retval != hipSuccess) {                                               \
      std::string error_message = std::string(__FILE__) + std::string(":")     \
        + std::to_string(__LINE__) + std::string("(")                          \
        + std::string(op_name) + std::string(") failed: ")                     \
        + hipGetErrorString(retval);                                          \
      fprintf(stderr, "%s\n", error_message.c_str());                          \
      fflush(stderr);                                                          \
      return retval;                                                           \
    }                                                                          \
  } while (false);                                                             \
}

#define GUARD_CU(op)                                                           \
{                                                                              \
  do {                                                                         \
    retval = (op);                                                             \
    if (retval != hipSuccess) {                                               \
      std::string error_message = std::string(__FILE__) + std::string(":")     \
        + std::to_string(__LINE__) + std::string(" failed: ")                  \
        + hipGetErrorString(retval);                                          \
      fprintf(stderr, "%s\n", error_message.c_str());                          \
      fflush(stderr);                                                          \
      return retval;                                                           \
    }                                                                          \
  } while (false);                                                             \
}

#define GUARD_NCCL2(op_name, op)                                               \
{                                                                              \
  do {                                                                         \
    auto nccl_result = (op);                                                   \
    if (nccl_result != ncclSuccess) {                                          \
      std::string error_message = std::string(__FILE__) + std::string(":")     \
        + std::to_string(__LINE__) + std::string("(")                          \
        + std::string(op_name) + std::string(") failed: ")                     \
        + ncclGetErrorString(nccl_result);                                     \
      fprintf(stderr, "%s\n", error_message.c_str());                          \
      fflush(stderr);                                                          \
      return hipErrorUnknown;                                                 \
    }                                                                          \
  } while (false);                                                             \
}

// ****************************
// Memory management
// ****************************

enum Malloc_t
{
  Default,
  Host,
  Managed,
  Raw,
};

template <typename T>
hipError_t Free(
  T* &ptr,
  Malloc_t malloc_type = Malloc_t::Default)
{
  hipError_t retval = hipSuccess;
  if (ptr == NULL)
    return retval;

  printf("Freeing @ %p\n", ptr);
  if (malloc_type == Host) {
    GUARD_CU2("hipHostFree",
      hipHostFree(ptr));
  } else if (malloc_type == Default || malloc_type == Managed) {
    GUARD_CU2("hipFree",
      hipFree(ptr));
  } else if (malloc_type == Raw)
    free(ptr);

  printf("Freed @ %p\n", ptr);
  ptr = NULL;
  return retval;
}

template <typename T>
hipError_t Malloc(
  T* &ptr,
  size_t target,
  Malloc_t malloc_type = Malloc_t::Default,
  unsigned int flags = hipMemAttachGlobal,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;

  size_t size = target * sizeof(T);
  printf("Allocating %ld x %ld bytes on %s\n", target, sizeof(T),
     malloc_type == Default ? "Default" :
    (malloc_type == Host    ? "Host" :
    (malloc_type == Managed ? "Managed" : "Raw")));

  if (malloc_type == Default) {
    GUARD_CU2("hipMalloc",
      hipMalloc(&ptr, size));
  } else if (malloc_type == Host) {
    GUARD_CU2("hipHostMalloc",
      hipHostMalloc(&ptr, size));
  } else if (malloc_type == Managed) {
    GUARD_CU2("hipMallocManaged",
      hipMallocManaged(&ptr, size, flags));
  } else if (malloc_type == Raw)
    ptr = (T*)malloc(size);

  printf("Allocated %ld x %ld bytes @ %p\n", target, sizeof(T), ptr);
  return retval;
}

template <typename T, typename SizeT>
hipError_t Memcpy(
  T* dest,
  T* src,
  SizeT num_elements,
  Malloc_t malloc_type = Malloc_t::Default,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;
  if (num_elements == 0)
    return retval;
  if (dest == NULL || src == NULL)
    return retval;

  if (malloc_type != Raw)
  {
    if (stream == 0)
    {
      retval = hipMemcpyAsync(dest, src, sizeof(T) * num_elements,
        hipMemcpyDefault, stream);
    } else {
      retval = hipMemcpy(dest, src, sizeof(T) * num_elements,
        hipMemcpyDefault);
    }
  } else {
    memcpy(dest, src, sizeof(T) * num_elements);
  }
  return retval;
}

template <typename T, typename SizeT>
hipError_t Memset(
  T* ptr,
  int value,
  SizeT num_elements,
  Malloc_t malloc_type = Malloc_t::Default,
  hipStream_t stream = 0)
{
  hipError_t retval = hipSuccess;
  if (num_elements == 0 || ptr == NULL)
    return retval;

  if (malloc_type != Malloc_t::Raw)
  {
    if (stream == 0)
    {
      retval = hipMemset(ptr, value, num_elements * sizeof(T));
    } else {
      retval = hipMemsetAsync(ptr, value, num_elements * sizeof(T), stream);
    }
  } else {
    memset(ptr, value, num_elements * sizeof(T));
  }

  return retval;
}

template <typename T, typename SizeT>
hipError_t GarenteeAllocation(
  T*      &ptr,
  SizeT   &allocated,
  size_t   target,
  Malloc_t malloc_type = Malloc_t::Default,
  unsigned int flags = hipMemAttachGlobal,
  hipStream_t stream = 0,
  bool     keep_content = false,
  bool     init_to_zero = false)
{
  hipError_t retval = hipSuccess;
  if (allocated >= target)
    return retval;

  //if (stream != 0)
  //{
  //  GUARD_CU2("hipStreamSynchronize",
  //    hipStreamSynchronize(stream));
  //}
  if (!keep_content)
  {
    auto temp_ptr = ptr;
    GUARD_CU(Free<T> (temp_ptr, malloc_type));
    GUARD_CU(Malloc(ptr, target, malloc_type, flags));
    if (init_to_zero)
    {
      GUARD_CU(Memset(ptr, 0, target, malloc_type, stream));
    }
  } else {
    T* temp_ptr = NULL;
    GUARD_CU(Malloc(temp_ptr, target, malloc_type, flags));
    GUARD_CU(Memcpy(temp_ptr, ptr, allocated, malloc_type, stream));
    if (init_to_zero)
    {
      GUARD_CU(Memset(temp_ptr + allocated, 0, target - allocated,
        malloc_type, stream));
    }
    GUARD_CU(Free(ptr, malloc_type));
    ptr = temp_ptr;
    temp_ptr = NULL;
  }
  allocated = target;
  return retval;
}

// ****************************
// DGC Functions
// ****************************

void DgcConfig::Set(std::string key, std::string value)
{
  if (key == "dgc_sparsity_warmup_epochs")
    warmup_epochs = std::stoi(value);

  else if (key == "dgc_init_sparsity")
    init_sparsity = std::stod(value);

  else if (key == "dgc_final_sparsity")
    final_sparsity = std::stod(value);

  else if (key == "dgc_sampling_rate")
    sampling_rate = std::stod(value);

  else if (key == "dgc_rand_seed")
    rand_seed = std::stoi(value);

  else if (key == "dgc_grid_size")
    grid_size = std::stoi(value);

  else if (key == "dgc_block_size")
    block_size = std::stoi(value);

  else if (key == "dgc_min_sampling_num")
    min_sampling_num = std::stoi(value);

  else if (key == "dgc_local_gradient_clipping")
  {
    if (value == "True")
      local_gradient_clipping = true;
    else if (value == "False")
      local_gradient_clipping = false;
  }

  else if (key == "dgc_clipping_threshold")
    clipping_threshold = std::stof(value);

  else if (key == "dgc_use_allreduce")
  {
    if (value == "True")
      use_allReduce = true;
    else if (value == "False")
      use_allReduce = false;
  }

  else if (key == "dgc_use_hierarchical_allreduce")
  {
    if (value == "True")
      use_hierarchical_allreduce = true;
    else if (value == "False")
      use_hierarchical_allreduce = false;
  }

  else if (key == "momentum")
    momentum = std::stof(value);

  else if (key == "num_examples_per_epoch")
    num_examples_per_epoch = std::stoi(value);

  else if (key == "batch_size")
    batch_size_per_gpu = std::stoi(value);

  //printf("%s = %s\n", key.c_str(), value.c_str());
}

template <typename T, typename SizeT, typename Compare>
hipError_t Sort(
  T           *elements,
  SizeT        num_elements,
  Compare      compare,
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  hipError_t retval = hipSuccess;

  if (malloc_type == Raw)
  {
    std::sort(elements, elements + num_elements, compare);
    return retval;
  }

  // Not using thrust for now;
  // if sort becomes performance bottleneck, change to cub
  // Note: thrust::sort hit a bug that produced illegal memory access
  //thrust::sort(thrust::cuda::par.on(stream),
  //  elements, elements + num_elements, compare);

  // Cub sorting
  bool temp_storage_allocated = false;
  if (temp_storage == NULL && temp_storage_bytes == NULL)
  {
    temp_storage = new char*;
    temp_storage[0] = NULL;
    temp_storage_bytes = new size_t;
    temp_storage_bytes[0] = 0;
    temp_storage_allocated = true;
  }

  size_t required_bytes = 0;
  GUARD_CU(hipcub::DeviceRadixSort::SortKeys(
    (char*)NULL, required_bytes,
    elements, elements,
    num_elements, 0, sizeof(T) * 8, stream));

  GUARD_CU(GarenteeAllocation(temp_storage[0],
    temp_storage_bytes[0], required_bytes, malloc_type, flags));
  //GUARD_CU2("hipDeviceSynchronize",
  //  hipDeviceSynchronize());

  GUARD_CU(hipcub::DeviceRadixSort::SortKeys(
    temp_storage[0], temp_storage_bytes[0],
    elements, elements,
    num_elements, 0, sizeof(T) * 8, stream));

  if (temp_storage_allocated)
  {
    GUARD_CU(Free(temp_storage[0], malloc_type));
    free(temp_storage);
    free(temp_storage_bytes);
    temp_storage = NULL;
    temp_storage_bytes = NULL;
    temp_storage_allocated = false;
  }

  return retval;
}

template <typename T, typename SizeT>
hipError_t Sort(
  T      *elements,
  SizeT   num_elements,
  hipStream_t stream = 0,
  Malloc_t malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  return Sort(elements, num_elements,
    [] __host__ __device__ (T a, T b){ return a < b;},
    stream, malloc_type, temp_storage, temp_storage_bytes, flags);
}

template <typename SizeT>
hipError_t Sort(
  ncclDataType_t nccl_type,
  void        *elements,
  SizeT        num_elements,
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  hipError_t retval = hipSuccess;

  switch (nccl_type)
  {
  case ncclFloat32:
    retval = Sort<float> ((float*)elements, num_elements,
      stream, malloc_type, temp_storage, temp_storage_bytes, flags);
    break;

  case ncclFloat64:
    retval = Sort<double> ((double*)elements, num_elements,
      stream, malloc_type, temp_storage, temp_storage_bytes, flags);
    break;

  case ncclInt32:
    retval = Sort<int32_t> ((int32_t*)elements, num_elements,
      stream, malloc_type, temp_storage, temp_storage_bytes, flags);
    break;

  case ncclInt64:
    retval = Sort<int64_t> ((int64_t*)elements, num_elements,
      stream, malloc_type, temp_storage, temp_storage_bytes, flags);
    break;

  default:
    break;
  }
  return retval;
}

template <typename T, typename SizeT, typename Compare>
hipError_t SegSort(
  T           *elements,
  SizeT        num_elements,
  SizeT       *seg_starts,
  int          num_segments,
  Compare      compare,
  hipStream_t stream = 0,
  Malloc_t     malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  hipError_t retval = hipSuccess;

  if (malloc_type == Raw)
  {
    for (int i = 0; i < num_segments; i++)
      std::sort(elements + seg_starts[i], elements + seg_starts[i+1], compare);
    return retval;
  }

  // Not using thrust for now;
  // if sort becomes performance bottleneck, change to cub
  // Note: thrust::sort hit a bug that produced illegal memory access
  //thrust::sort(thrust::cuda::par.on(stream),
  //  elements, elements + num_elements, compare);

  // Cub sorting
  bool temp_storage_allocated = false;
  if (temp_storage == NULL && temp_storage_bytes == NULL)
  {
    temp_storage = new char*;
    temp_storage[0] = NULL;
    temp_storage_bytes = new size_t;
    temp_storage_bytes[0] = 0;
    temp_storage_allocated = true;
  }

  size_t required_bytes = 0;
  GUARD_CU(hipcub::DeviceSegmentedRadixSort::SortKeys(
    (char*)NULL, required_bytes,
    elements, elements, num_elements,
    num_segments, seg_starts, seg_starts + 1,
    0, sizeof(T) * 8, stream));

  GUARD_CU(GarenteeAllocation(temp_storage[0],
    temp_storage_bytes[0], required_bytes, malloc_type, flags));
  //GUARD_CU2("hipDeviceSynchronize",
  //  hipDeviceSynchronize());

  GUARD_CU(hipcub::DeviceSegmentedRadixSort::SortKeys(
    temp_storage[0], temp_storage_bytes[0],
    elements, elements, num_elements,
    num_segments, seg_starts, seg_starts + 1,
    0, sizeof(T) * 8, stream));

  if (temp_storage_allocated)
  {
    GUARD_CU(Free(temp_storage[0], malloc_type));
    free(temp_storage);
    free(temp_storage_bytes);
    temp_storage = NULL;
    temp_storage_bytes = NULL;
    temp_storage_allocated = false;
  }

  return retval;
}

template <typename T, typename SizeT>
hipError_t SegSort(
  T      *elements,
  SizeT   num_elements,
  SizeT  *seg_starts,
  int     num_segments,
  hipStream_t stream = 0,
  Malloc_t malloc_type = Malloc_t::Default,
  char       **temp_storage = NULL,
  size_t      *temp_storage_bytes = NULL,
  unsigned int flags = hipMemAttachGlobal)
{
  return SegSort(elements, num_elements, seg_starts, num_segments,
    [] __host__ __device__ (T a, T b){ return a < b;},
    stream, malloc_type, temp_storage, temp_storage_bytes, flags);
}

template <typename T>
hipError_t ClipGradient(
  T          *gradients,
  //uint64_t   *layer_offsets,
  //int         num_layers,
  std::vector<std::pair<std::string, uint64_t> > &layers,
             // <name, #elements> of layers
  DgcConfig  &config,
  DgcState   &state)
{
  hipError_t retval = hipSuccess;

  // skip first step, because total number of layers are unknown
  if (state.step == 0)
    return retval;

  int num_layers = layers.size();
  GUARD_CU(GarenteeAllocation(state.temp_storage, state.temp_storage_bytes,
    sizeof(T) * 2 * num_layers + sizeof(uint32_t) * (num_layers + 1)));
  GUARD_CU(GarenteeAllocation(state.h_layer_starts, state.h_layer_starts_allocated,
    num_layers + 1, Malloc_t::Host));
  uint32_t start_counter = 0;
  for (int i = 0; i < num_layers; i++)
  {
    state.h_layer_starts[i] = start_counter;
    start_counter += layers[i].second;
  }
  state.h_layer_starts[num_layers] = start_counter;

  T* sums         = (T*)(state.temp_storage);
  T* coefficients = (T*)(state.temp_storage + sizeof(T) * num_layers);
  uint32_t* layer_starts = (uint32_t*)(state.temp_storage + sizeof(T) * 2 * num_layers);
  auto stream     = config.stream;
  int  grid_size  = config.grid_size;
  int  block_size = config.block_size;
  auto clipping_threshold = config.clipping_threshold;

  GUARD_CU(Memset(sums, 0, num_layers, Malloc_t::Default, stream));
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(layer_starts, state.h_layer_starts, sizeof(uint32_t) * (num_layers + 1),
      hipMemcpyHostToDevice, stream));

  // loop_kernel<<<grid_size, block_size, 0, stream>>>(layer_offsets[num_layers],
  //   [offsets, sums, gradients, num_layers] __device__ (const uint64_t &i)
  //   {
  //     int layer = binarySearch(offsets, 0, num_layers, i);
  //     //if (i < offsets[layer] || i >= offsets[layer + 1])
  //     //  printf("offset mismatch: i = %ld, layer = %d, offsets = %ld, %ld, %ld\n",
  //     //      i, layer, layer > 0 ? offsets[layer -1] : -1,
  //     //      offsets[layer], layer < num_layers ? offsets[layer + 1] : -1);
  //
  //     auto gradient = gradients[i];
  //     atomicAdd(sums + layer, gradient * gradient);
  //   });
  L2norm_kernel<<<grid_size, block_size, 0, stream>>>(
    gradients, layer_starts, num_layers, sums);

  int total_num_layers = state.layer_offset_bytes.size();
  uint64_t total_num_gradients = state.offset_byte_counter / sizeof(T);

  loop_kernel<<<grid_size, block_size, 0, stream>>>(num_layers,
    [sums, coefficients, total_num_layers, total_num_gradients,
    clipping_threshold, layer_starts]
    __device__ (const int &layer)
    {
      coefficients[layer] = clipping_threshold /
        // (sqrt(sums[layer] * total_num_gradients / (offsets[layer + 1] - offsets[layer])) + 1e-6);
        // (sqrt(sums[layer]) + 1e-6);
        (sqrt(sums[layer]) * total_num_layers + 1e-6);
        //(sqrt(sums[layer]) * total_num_gradients / (offsets[layer + 1] - offsets[layer]) + 1e-6);
      //printf("Layer %3d: L2 norm = %3.6f, #gradients = %6ld, coef = %3.6f\n",
      //  layer, sqrt(sums[layer]), (long)(offsets[layer+1] - offsets[layer]),
      //  coefficients[layer]);
    });

  loop_kernel<<<grid_size, block_size, 0, stream>>>(start_counter,
    [layer_starts, gradients, coefficients, num_layers] __device__ (const uint32_t &i)
    {
      int layer = binarySearch(layer_starts, 0, num_layers, i);
      auto coefficient = coefficients[layer];
      if (coefficient < 1)
        gradients[i] *= coefficient;
    });

  return retval;
}

// Main DGC routine
template <typename T, typename SizeT>
hipError_t GradientAllReduce(
  T              *input_gradients,     // GPU pointer to the input_gradients
  T              *output_gradients,     // GPU pointer to the output_gradients
  std::vector<std::pair<std::string, uint64_t> > &layers,
                                  // <name, #elements> of layers
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  hipError_t retval = hipSuccess;
  //SizeT num_samples  = 0;
  auto  block_size   = config.block_size;
  auto  grid_size    = config.grid_size;
  auto  stream       = config.stream;
  int   num_layers   = layers.size();
  SizeT num_gradients = 0;

  //GUARD_CU2("hipStreamSynchronize before",
  //  hipStreamSynchronize(stream));

  if (config.local_gradient_clipping)
    GUARD_CU(ClipGradient(input_gradients, layers, config, state));
  //GUARD_CU2("hipStreamSynchronize after clipping",
  //  hipStreamSynchronize(stream));

  GUARD_CU(GarenteeAllocation(state.h_layer_starts, state.h_layer_starts_allocated,
    num_layers + 1, Malloc_t::Host));
  // find which step is currently in and look for unallocated layers
  std::vector<std::pair<std::string, uint64_t> > layers_to_allocate;
  SizeT num_gradients_to_allocate = 0;
  for (auto &layer : layers)
  {
    auto name = layer.first;
    num_gradients += layer.second;
    // finds step number
    auto counter_it = state.step_counters.find(name);
    if (counter_it == state.step_counters.end())
      state.step_counters[name] = 0;
    else {
      auto step = counter_it -> second;
      counter_it -> second ++;
      if (state.step < step)
        state.step = step;
    }

    auto offset_it = state.layer_offset_bytes.find(name);
    if (offset_it == state.layer_offset_bytes.end()) {
      layers_to_allocate.push_back(layer);
      num_gradients_to_allocate += layer.second;
    }
  } // end of for layers

  // allocate new layers
  if (num_gradients_to_allocate > 0) {
    GUARD_CU(GarenteeAllocation(state.pervious_verlocity,
      state.pervious_verlocity_allocated,
      state.offset_byte_counter + sizeof(T) * num_gradients_to_allocate,
      Malloc_t::Default, hipMemAttachGlobal, stream, true, true));
    GUARD_CU(GarenteeAllocation(state.pervious_accumulated_verlocity,
      state.pervious_accumulated_verlocity_allocated,
      state.offset_byte_counter + sizeof(T) * num_gradients_to_allocate,
      Malloc_t::Default, hipMemAttachGlobal, stream, true, true));

    for (auto& layer : layers_to_allocate) {
      state.layer_offset_bytes[layer.first] = state.offset_byte_counter;
      state.offset_byte_counter += layer.second * sizeof(T);
    }
  }

  // find continous layers as chunks
  // <start, size, offset> of chunks
  std::vector<std::tuple<SizeT, SizeT, size_t> > chunks;
  size_t chunk_offset_bytes = state.layer_offset_bytes[layers.begin() -> first];
  SizeT  layer_start = 0;
  SizeT  chunk_start = 0;
  SizeT  chunk_size  = 0;
  for (int i = 0; i < num_layers; i++) {
    auto &layer = layers[i];
    state.h_layer_starts[i] = layer_start;
    if (chunk_offset_bytes + chunk_size * sizeof(T) !=
      state.layer_offset_bytes[layer.first]) {
      // mismatch
      chunks.push_back(std::make_tuple(
        chunk_start, chunk_size, chunk_offset_bytes));
      chunk_size  = 0;
      chunk_start = layer_start;
      chunk_offset_bytes = state.layer_offset_bytes[layer.first];
    }

    chunk_size  += layer.second;
    layer_start += layer.second;
  } // end of for layers
  state.h_layer_starts[num_layers] = layer_start;
  if (chunk_size != 0)
    chunks.push_back(std::make_tuple(
      chunk_start, chunk_size, chunk_offset_bytes));

  auto &layer_starts = state.layer_starts;
  GUARD_CU(GarenteeAllocation(state.layer_starts,
    state.layer_starts_allocated, num_layers + 1));
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(state.layer_starts, state.h_layer_starts,
      sizeof(uint32_t) * (num_layers + 1), hipMemcpyHostToDevice, stream));

  // Memory allocation and type conversion
  GUARD_CU(GarenteeAllocation(state.verlocity,
    state.verlocity_allocated, num_gradients * sizeof(T)));
  GUARD_CU(GarenteeAllocation(state.accumulated_verlocity,
    state.accumulated_verlocity_allocated, num_gradients * sizeof(T)));
  T* verlocity = (T*)(state.verlocity);
  T* accumulated_verlocity = (T*)(state.accumulated_verlocity);

  // momentum correction by chunks
  for (auto& chunk : chunks)
  {
    SizeT chunk_start = std::get<0>(chunk);
    SizeT chunk_size  = std::get<1>(chunk);
    size_t chunk_offset = std::get<2>(chunk);

    T* pervious_verlocity
      = (T*)(state.pervious_verlocity + chunk_offset);
    T* pervious_accumulated_verlocity
      = (T*)(state.pervious_accumulated_verlocity + chunk_offset);
    auto &momentum = config.momentum;

    //printf("input_gradients = %p, gradient_chunk = [%ld, %ld), "
    //  "pervious_verlocity = %p, verlocity = %p, "
    //  "pervious_accumulated_verlocity = %p, accumulated_verlocity = %p\n",
    //  input_gradients, gradient_start_chunk,
    //  gradient_start_chunk + num_gradients_chunk,
    //  pervious_verlocity, verlocity,
    //  pervious_accumulated_verlocity, accumulated_verlocity);

    loop_kernel<<<grid_size, block_size, 0, stream>>>(chunk_size,
      [momentum, input_gradients, chunk_start,
      pervious_verlocity, verlocity,
      accumulated_verlocity, pervious_accumulated_verlocity]
      __device__ (const SizeT &i) {
        auto pos = i + chunk_start;
        auto u = pervious_verlocity[i] * momentum + input_gradients[pos];
        accumulated_verlocity[pos] = pervious_accumulated_verlocity[i] + u;
        verlocity[pos] = u;
      });
  }
  //GUARD_CU2("hipStreamSynchronize after momentum correction",
  //  hipStreamSynchronize(stream));

  // Sampling
  auto &samp_starts = state.samp_starts;
  GUARD_CU(GarenteeAllocation(state.samp_starts, state.samp_starts_allocated,
    num_layers + 1));
  GUARD_CU(GarenteeAllocation(state.h_samp_starts, state.h_samp_starts_allocated,
    num_layers + 1, Malloc_t::Host));
  uint32_t samp_counter = 0;
  for (int i = 0; i < num_layers; i++)
  {
    auto &layer = layers[i];
    state.h_samp_starts[i] = samp_counter;

    uint32_t num_samples = 0;
    if (config.sampling_rate < 1 &&
        layer.second > config.min_sampling_num) {

      num_samples = layer.second * config.sampling_rate;
      if (num_samples < config.min_sampling_num)
        num_samples = config.min_sampling_num;
      if (num_samples > layer.second)
        num_samples = layer.second;
    }

    else { // no sampling
      num_samples = layer.second;
      //GUARD_CU(GarenteeAllocation(state.samp_data, state.samp_allocated,
      //  num_samples * sizeof(T)));

      //GUARD_CU2("hipMemcpyAsync",
      //  hipMemcpyAsync(state.samp_data, gradients,
      //    sizeof(T) * num_samples, hipMemcpyDeviceToDevice, stream));
      //T* samp_data = (T*)(state.samp_data);
      //loop_kernel<<<grid_size, block_size, 0, stream>>>(num_samples,
      //  [samp_data, accumulated_verlocity] __device__ (const SizeT &i){
      //    samp_data[i] = abs(accumulated_verlocity[i]);
      //  });
    }
    samp_counter += num_samples;
  }
  state.h_samp_starts[num_layers] = samp_counter;
  GUARD_CU2("hipMemcpyAsync",
    hipMemcpyAsync(state.samp_starts, state.h_samp_starts,
      sizeof(uint32_t) * (num_layers + 1), hipMemcpyHostToDevice, stream));

  auto &rand_states = state.rand_states;
  auto &rand_seed   = config.rand_seed;
  if (rand_states == NULL) {
    GUARD_CU(Malloc(rand_states, grid_size * block_size));

    loop_kernel<<<grid_size, block_size, 0, stream>>>(
      (SizeT)grid_size * block_size,
      [rand_states, rand_seed] __device__ (const SizeT &i){
        hiprand_init(rand_seed, i, 0, rand_states + i);
      });
  }

  GUARD_CU(GarenteeAllocation(state.samp_data, state.samp_allocated,
    samp_counter * sizeof(T)));
  T* samp_data = (T*)(state.samp_data);

  //sample_kernel <T, SizeT>
  //  <<<grid_size, block_size, 0, stream>>>(
  //  accumulated_verlocity, num_gradients,
  //  samp_data, num_samples,
  //  state.rand_states);
  sample_kernel2<<<grid_size, block_size, 0, stream>>>(
    accumulated_verlocity, num_gradients,
    state.layer_starts, num_layers,
    state.samp_starts, samp_data, state.rand_states);

  //GUARD_CU2("hipStreamSynchronize after sampling",
  //  hipStreamSynchronize(stream));
  //GUARD_CU2("hipDeviceSynchronize before Sort",
  //  hipDeviceSynchronize());

  // Sort the samples
  //GUARD_CU(Sort(samp_data, num_samples, stream, Malloc_t::Default,
  //  &(state.temp_storage), &(state.temp_storage_bytes)));
  GUARD_CU(SegSort(samp_data, samp_counter, state.samp_starts, num_layers,
    stream, Malloc_t::Default, &(state.temp_storage), &(state.temp_storage_bytes)));
  //GUARD_CU2("hipDeviceSynchronize after Sort",
  //  hipDeviceSynchronize());
  //GUARD_CU2("hipStreamSynchronize after Sort",
  //  hipStreamSynchronize(stream));

  // Determine the threshold
  uint64_t num_examples_per_step = config.batch_size_per_gpu * config.global_num_gpus;
  uint64_t steps_per_epoch = config.num_examples_per_epoch / num_examples_per_step;
  if (steps_per_epoch * num_examples_per_step < config.num_examples_per_epoch)
    steps_per_epoch ++;
  uint64_t epoch    = state.step * 1.0 / steps_per_epoch;
  double sparsity   = config.final_sparsity;
  if (epoch < config.warmup_epochs) {
    sparsity = config.init_sparsity * exp(
      log(config.final_sparsity / config.init_sparsity)
      / (config.warmup_epochs - 1) * epoch);
    //if (epoch * steps_per_epoch == state.step)
    //  printf("Epoch %ld, Step %ld, sparsity = %lf\n",
    //    epoch, state.step, sparsity);
  }
  SizeT  target_num = num_gradients * (1 - sparsity);
  //auto &threshold = state.gradient_threshold;
  //if (threshold == NULL) {
  //  GUARD_CU(Malloc(threshold, 1));
  //}

  //loop_kernel<<<1, 1, 0, stream>>>((SizeT)1,
  //  [threshold, samp_data, num_samples, sparsity] __device__ (const SizeT &i){
  //    SizeT pos = num_samples * sparsity;
  //    if (pos >= num_samples)
  //      pos = num_samples - 1;
  //    threshold[0] = samp_data[pos];
  //    //printf("selecting samp[%d] from [%d] {%f, %f, ... %f, %f, %f, ... %f, %f}\n",
  //    //  pos, num_samples,
  //    //  num_samples > 0 ? samp_data[0] : -1,
  //    //  num_samples > 1 ? samp_data[1] : -1,
  //    //  num_samples + 1 > pos  && pos > 0 ? samp_data[pos - 1] : -1,
  //    //  num_samples > pos && pos >= 0 ? samp_data[pos] : -1,
  //    //  num_samples > pos + 1 && pos + 1 >= 0 ? samp_data[pos + 1] : -1,
  //    //  num_samples > 1 ? samp_data[num_samples - 2] : -1,
  //    //  num_samples > 0 ? samp_data[num_samples - 1] : -1);
  //  });
  auto &thresholds = state.thresholds;
  GUARD_CU(GarenteeAllocation(thresholds, state.thresholds_allocated, num_layers));

  loop_kernel<<<grid_size, block_size, 0, stream>>>(num_layers,
    [thresholds, samp_data, samp_starts, sparsity]
    __device__ (const int &layer){
      auto samp_start = samp_starts[layer];
      auto samp_end   = samp_starts[layer + 1];
      auto samp_size  = samp_end - samp_start;
      SizeT pos = samp_size * sparsity;
      if (pos >= samp_size)
        pos = samp_size;
      thresholds[layer] = samp_data[samp_start + pos];
    });

  if (config.use_allReduce) {
    // use allReduce on mask to communicate

    SizeT num_masks = num_gradients / 32;
    if (num_masks * 32 < num_gradients)
      num_masks ++;

    auto mask_allocated_ = state.mask_allocated;
    GUARD_CU(GarenteeAllocation(state.send_masks  , mask_allocated_, num_masks));
    mask_allocated_ = state.mask_allocated;
    GUARD_CU(GarenteeAllocation(state.h_send_masks, mask_allocated_, num_masks, Malloc_t::Host));
    mask_allocated_ = state.mask_allocated;
    GUARD_CU(GarenteeAllocation(state.h_recv_masks, mask_allocated_, num_masks, Malloc_t::Host));
    mask_allocated_ = state.mask_allocated;
    GUARD_CU(GarenteeAllocation(state.recv_masks  , mask_allocated_, num_masks));
    if (state.mask_allocated < num_masks)
      state.mask_allocated = num_masks;

    auto &mask_counters = state.mask_counters;
    auto &mask_offsets  = state.mask_offsets;
    GUARD_CU(GarenteeAllocation(
        mask_counters, state.mask_counters_allocated, (num_masks + 1)));
    GUARD_CU(GarenteeAllocation(
        mask_offsets , state.mask_offsets_allocated , (num_masks + 1)));

    size_t required_bytes = 0;
    GUARD_CU(hipcub::DeviceScan::InclusiveSum(
      (char*)NULL, required_bytes,
      mask_counters, mask_counters, num_masks));
    GUARD_CU(GarenteeAllocation(
      state.temp_storage, state.temp_storage_bytes, required_bytes));

    if (state.h_num_gradients_to_communicate == NULL)
        GUARD_CU(Malloc(state.h_num_gradients_to_communicate, 1, Malloc_t::Host));

    //GUARD_CU2("hipStreamSynchronize after allocation",
    //  hipStreamSynchronize(stream));

    auto &send_masks = state.send_masks;
    auto &recv_masks = state.recv_masks;
    loop_kernel<<<grid_size, block_size, 0, stream>>>(num_masks,
      [send_masks, num_gradients, thresholds, layer_starts, num_layers,
      accumulated_verlocity]
      __device__ (const SizeT &i)
      {
        uint32_t mask = 0;
        SizeT offset = i * 32;
        int end_j = 32, j = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;
        while (j < end_j)
        {
          auto pos = j + offset;
          T element = accumulated_verlocity[pos];
          int layer = binarySearch(layer_starts, 0, num_layers, pos);
          if (!isfinite(element * 1.0f))
          {
            j ++;
            continue;
          }

          if (!(abs(element) < thresholds[layer]))
          {
            mask |= (((uint32_t)1) << j);
          }
          j++;
        }
        send_masks[i] = mask;
      });

    GUARD_CU2("hipMemcpyAsync",
      hipMemcpyAsync(state.h_send_masks, send_masks, sizeof(uint32_t) * num_masks,
        hipMemcpyDeviceToHost, stream));
    GUARD_CU2("hipStreamSynchronize after mask",
      hipStreamSynchronize(stream));

    MPI_Allreduce(state.h_send_masks, state.h_recv_masks, (int)num_masks,
      PreDefinedValues<uint32_t>::getMpiDataType(), MPI_BOR,
      config.use_hierarchical_allreduce ? config.cross_comm : config.mpi_comm);

    GUARD_CU2("hipMemcpyAsync",
      hipMemcpyAsync(recv_masks, state.h_recv_masks, sizeof(uint32_t) * num_masks,
        hipMemcpyHostToDevice, stream));

    loop_kernel<<<grid_size, block_size, 0, stream>>>(num_masks,
      [recv_masks, mask_counters] __device__ (const SizeT &i)
      {
        mask_counters[i] = __popc(recv_masks[i]);
      });

    GUARD_CU(Memset(mask_offsets, 0, 1, Malloc_t::Default, stream));
    GUARD_CU(hipcub::DeviceScan::InclusiveSum(
      state.temp_storage, required_bytes,
      mask_counters, mask_offsets + 1, num_masks, stream));

    GUARD_CU2("hipMemcpyAsync",
      hipMemcpyAsync(state.h_num_gradients_to_communicate,
        mask_offsets + num_masks, sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
    GUARD_CU2("hipStreamSynchronize after InclusiveSum",
      hipStreamSynchronize(stream));

    auto num_gradients_comm = state.h_num_gradients_to_communicate[0];
    if (config.global_gpu_rank == 0)
      printf("%d #gradients to comm = %ld, #gradients = %ld, rate = %f\n",
        state.step, (long)num_gradients_comm, (long)num_gradients,
        1.0f * num_gradients_comm / num_gradients);

    auto send_allocated_ = state.send_allocated * sizeof(T);
    GUARD_CU(GarenteeAllocation(
      state.send_data, send_allocated_, sizeof(T) * num_gradients_comm));
    if (state.send_allocated < num_gradients_comm)
      state.send_allocated = num_gradients_comm;
    auto recv_allocated_ = state.recv_allocated * sizeof(T);
    GUARD_CU(GarenteeAllocation(
      state.recv_data, recv_allocated_, sizeof(T) * num_gradients_comm));
    if (state.recv_allocated < num_gradients_comm)
      state.recv_allocated = num_gradients_comm;

    T* send_data = (T*)(state.send_data);
    T* recv_data = (T*)(state.recv_data);
    auto global_num_gpus = config.global_num_gpus;
    loop_kernel<<<grid_size, block_size, 0, stream>>>(num_masks,
      [recv_masks, mask_offsets, send_data, global_num_gpus,
      num_gradients, accumulated_verlocity]
      __device__ (const SizeT &i)
      {
        uint32_t mask = recv_masks[i];
        if (mask == 0)
          return;

        SizeT offset = i * 32, output_offset = mask_offsets[i];
        int end_j = 32, j = 0, output_count = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;
        while (j < end_j)
        {
          if ((mask & (((uint32_t)1) << j)) == 0)
          {
            j ++;
            continue;
          }
          T element = accumulated_verlocity[j + offset];
          if (!isfinite(element * 1.0f))
            element = 0;

          send_data[output_offset + output_count] = element; // / global_num_gpus;
          output_count ++;
          j++;
        }
      });

    //GUARD_CU2("hipStreamSynchronize after send_data forming",
    //  hipStreamSynchronize(stream));

    GUARD_NCCL2("ncclAllReduce",
      ncclAllReduce(send_data   , (void*)recv_data,
        (size_t)num_gradients_comm,
        PreDefinedValues<T>::NCCLDataType, ncclSum,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));

    GUARD_CU(Memset(output_gradients, 0, num_gradients, Malloc_t::Default, stream));
    loop_kernel<<<grid_size, block_size, 0, stream>>>(num_masks,
      [recv_masks, mask_offsets, recv_data, output_gradients, num_gradients]
      __device__ (const SizeT &i)
      {
        uint32_t mask = recv_masks[i];
        if (mask == 0)
          return;

        SizeT offset = i * 32, output_offset = mask_offsets[i];
        int end_j = 32, j = 0, output_count = 0;
        if (offset + end_j > num_gradients)
          end_j = num_gradients - offset;
        while (j < end_j)
        {
          if ((mask & (((uint32_t)1) << j)) == 0)
          {
            j ++;
            continue;
          }

          output_gradients[j + offset] = recv_data[output_offset + output_count];
          output_count ++;
          j++;
        }
      });

    //GUARD_CU2("hipStreamSynchronize after output_gradient calculation",
    //  hipStreamSynchronize(stream));

    // Updates pervious_verlocity and pervious_accumulated_verlocity
    // Can be overlap with communication
    for (auto& chunk : chunks)
    {
      SizeT  chunk_start  = std::get<0>(chunk);
      SizeT  chunk_size   = std::get<1>(chunk);
      size_t chunk_offset = std::get<2>(chunk);

      T* pervious_verlocity
        = (T*)(state.pervious_verlocity + chunk_offset);
      T* pervious_accumulated_verlocity
        = (T*)(state.pervious_accumulated_verlocity + chunk_offset);

      loop_kernel <<<grid_size, block_size, 0, stream>>>(chunk_size,
        [recv_masks, chunk_start, chunk_size,
         verlocity, pervious_verlocity,
         accumulated_verlocity, pervious_accumulated_verlocity]
        __device__ (const SizeT &i)
        {
          //if (i == 0)
          //  printf("gradient [%ld...%ld) \n",
          //    (long)gradient_start_chunk,
          //    (long)(gradient_start_chunk + num_gradients_chunk));
          auto gradient_pos = i + chunk_start;
          auto mask_pos = gradient_pos / 32;
          auto mask = recv_masks[mask_pos];
          auto mask_offset = (gradient_pos & ((uint32_t)31));

          if ((mask & (((uint32_t)1) << mask_offset)) != 0)
          {
            pervious_verlocity[i] = 0;
            pervious_accumulated_verlocity[i] = 0;
          } else {
            pervious_verlocity[i] = verlocity[gradient_pos];
            pervious_accumulated_verlocity[i] = accumulated_verlocity[gradient_pos];
          }
        });
    }
  } // end of if (use_allReduce)

  else {
    // use allGather to communicate
    //auto &samp_counter = state.samp_counter;
    //loop_kernel <<<1, 1, 0, stream>>>((SizeT)1,
    //  [samp_counter] __device__ (const SizeT &i)
    //  {
    //    samp_counter[0] = 0;
    //  });
    //loop_kernel <<<grid_size, block_size, 0, stream>>>(num_samples,
    //  [samp_data, num_samples, samp_counter, threshold] __device__ (const SizeT &i)
    //  {
    //    if (!(samp_data[i] < threshold[0]))
    //    {
    //      atomicAdd(samp_counter, (uint64_t)1);
    //    }
    //  });
    //loop_kernel <<<1, 1, 0, stream>>>((SizeT)1,
    //  [samp_counter] __device__ (const SizeT &i)
    //  {
    //    printf("Recount = %d\n", samp_counter[0]);
    //  });

    // Pick those larger than threshold
    auto &send_counter   = state.send_counter;
    //auto &send_data      = state.send_data;
    auto &send_indices   = state.send_indices;
    auto &send_allocated = state.send_allocated;
    auto send_allocated_ = send_allocated * sizeof(T);
    if (send_counter == NULL) {
      GUARD_CU(Malloc(send_counter, 1));
    }

    GUARD_CU(GarenteeAllocation(
      state.send_data, send_allocated_, target_num * sizeof(T)));
    GUARD_CU(GarenteeAllocation(
      send_indices, send_allocated , target_num));
    if (state.max_gradient == NULL) {
      GUARD_CU(Malloc(state.max_gradient, 1));
    }
    //loop_kernel <<<1, 1, 0, stream>>>((SizeT)1,
    //  [send_counter] __device__ (const SizeT &i)
    //  {
    //    send_counter[0] = 0;
    //  });
    GUARD_CU(Memset(send_counter, 0, 1, Malloc_t::Default, stream));
    GUARD_CU(Memset(state.max_gradient, 0, 1, Malloc_t::Default, stream));

    T* send_data = (T*)(state.send_data);
    // select at most target_num gradients
    //select_kernel
    //  <<<grid_size, block_size, 0, stream>>>
    //  (accumulated_verlocity, num_gradients, config.global_num_gpus,
    //  threshold, target_num, send_data, send_indices, send_counter,
    //  state.max_gradient);
    select_kernel3
      <<<grid_size, block_size, 0, stream>>>
      (accumulated_verlocity, config.global_num_gpus,
      thresholds, layer_starts, num_layers, target_num,
      send_data, send_indices, send_counter, state.max_gradient);

    // pad if num_slected < target_num
    pad_kernel
      <<<grid_size, block_size, 0, stream>>>
      ((T*)send_data, send_indices, target_num, send_counter, state.max_gradient);

    // Reallocate if not enough
    SizeT recv_count      = target_num * config.global_num_gpus;
    auto &recv_allocated  = state.recv_allocated;
    auto  recv_allocated_ = state.recv_allocated * sizeof(T);
    //auto &recv_data       = state.recv_data;
    auto &recv_indices    = state.recv_indices;

    //printf("recv_count = %lld\n", (long long)recv_count);
    GUARD_CU(GarenteeAllocation(
        state.recv_data, recv_allocated_, recv_count * sizeof(T)));
    GUARD_CU(GarenteeAllocation(
        recv_indices, recv_allocated, recv_count));

    //GUARD_CU2("hipStreamSynchronize after send data forming",
    //    hipStreamSynchronize(stream));

    T* recv_data = (T*)(state.recv_data);
    // Collect selected data & indices from all peers
    GUARD_NCCL2("ncclAllGather",
      ncclAllGather(send_data   , (void*)recv_data,
        (size_t)target_num, PreDefinedValues<T       >::NCCLDataType,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));
    GUARD_NCCL2("ncclAllGather",
      ncclAllGather(send_indices, (void*)recv_indices,
        (size_t)target_num, PreDefinedValues<uint32_t>::NCCLDataType,
        config.use_hierarchical_allreduce ? config.nccl_cross_comm :
          config.nccl_comm, stream));
    //GUARD_CU2("hipStreamSynchronize after AllGather",
    //    hipStreamSynchronize(stream));

    //auto &global_gradients_= state.global_gradients;
    //auto &global_allocated = state.global_allocated;
    //GUARD_CU(GarenteeAllocation(
    //  state.global_gradients, global_allocated, num_gradients * sizeof(T)));
    //T* global_gradients = (T*)(state.global_gradients);

    // Post process gradients
    //loop_kernel <<<grid_size, block_size, 0, stream>>>(num_gradients,
    //  [global_gradients] __device__ (const SizeT &i)
    //  {
    //    global_gradients[i] = 0;
    //  });
    GUARD_CU(Memset(output_gradients, 0, num_gradients, Malloc_t::Default, stream));

    // Unpack recv data
    loop_kernel <<<grid_size, block_size, 0, stream>>>(recv_count,
      [recv_data, recv_indices, output_gradients] __device__ (const SizeT &i)
      {
        T     element = recv_data   [i];
        SizeT index   = recv_indices[i];
        if (isValid(index))
          atomicAdd(output_gradients + index, element);
      });

    // Updates pervious_verlocity and pervious_accumulated_verlocity
    // Can be overlap with communication
    for (auto &chunk : chunks)
    {
      SizeT  chunk_start  = std::get<0>(chunk);
      SizeT  chunk_size   = std::get<1>(chunk);
      size_t chunk_offset = std::get<2>(chunk);

      T* pervious_verlocity
        = (T*)(state.pervious_verlocity + chunk_offset);
      T* pervious_accumulated_verlocity
        = (T*)(state.pervious_accumulated_verlocity + chunk_offset);

      loop_kernel <<<grid_size, block_size, 0, stream>>>(chunk_size,
        [thresholds, chunk_start, chunk_size,
        verlocity, pervious_verlocity,
        accumulated_verlocity, pervious_accumulated_verlocity,
        layer_starts, num_layers]
        __device__ (const SizeT &i)
        {
          //if (i == 0)
          //  printf("gradient [%ld...%ld) \n",
          //    (long)gradient_start_chunk,
          //    (long)(gradient_start_chunk + num_gradients_chunk));
          auto gradient_pos = i + chunk_start;
          auto v = accumulated_verlocity[gradient_pos];
          int layer = binarySearch(layer_starts, 0, num_layers, gradient_pos);
          if (isfinite(v * 1.0f) && abs(v) > thresholds[layer])
          {
            pervious_verlocity[i] = 0;
            pervious_accumulated_verlocity[i] = 0;
          } else {
            pervious_verlocity[i] = verlocity[gradient_pos];
            pervious_accumulated_verlocity[i] = v;
          }
        });
    }
  }

  //GUARD_CU2("hipStreamSynchronize after",
  //  hipStreamSynchronize(stream));

  return retval;
}

// Entry warper function
hipError_t ClipGradient(
  ncclDataType_t  gradient_type, // type of gradient
  void           *gradients,     // GPU pointer to the gradients
  //uint64_t       *layer_offsets, // gradient layer offsets, on host
  //int             num_layers,    // The number of layers in the gradients
  std::vector<std::pair<std::string, uint64_t> > &layers,
                                // <name, #elements> of layers
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  typedef uint32_t SizeT;
  hipError_t retval = hipSuccess;

  switch (gradient_type)
  {
  case ncclFloat32:
    retval = ClipGradient <float> (
      //(float*)gradients, layer_offsets, num_layers, config, state);
      (float*)gradients, layers, config, state);
    break;

  case ncclFloat64:
    retval = ClipGradient <double> (
      //(double*)gradients, layer_offsets, num_layers, config, state);
      (double*)gradients, layers, config, state);
    break;

  case ncclInt32:
    retval = ClipGradient <int32_t> (
      //(int32_t*)gradients, layer_offsets, num_layers, config, state);
      (int32_t*)gradients, layers, config, state);
    break;

  case ncclInt64:
    retval = ClipGradient <int64_t> (
      //(int64_t*)gradients, layer_offsets, num_layers, config, state);
      (int64_t*)gradients, layers, config, state);
    break;

  default:
    break;
  }
  return retval;
}

hipError_t GradientAllReduce(
  ncclDataType_t  gradient_type, // type of gradient
  void           *input_gradients, // GPU pointer to the input graients
  void           *output_gradients,// GPU pointer to the output gradients
  //uint64_t        num_gradients, // number of gradients
  //std::vector<std::tuple<uint64_t, uint64_t, size_t> >
  //               &offset_map,    // <start, length, offset> mappings for
                                 // continous chunks of gradients
  std::vector<std::pair<std::string, uint64_t> > &layers,
                                 // <name, #elements> of layers
  DgcConfig      &config,        // DGC configuration
  DgcState       &state)         // DGC running states
{
  typedef uint32_t SizeT;
  hipError_t retval = hipSuccess;

  if (config.use_hierarchical_allreduce &&
      !config.cross_comm_inited) {
    ncclUniqueId nccl_cross_id;
    if (config.global_node_rank == 0) {
      GUARD_NCCL2("ncclGetUniqueId",
        ncclGetUniqueId(&nccl_cross_id));
    }

    MPI_Bcast((void*)&nccl_cross_id, sizeof(nccl_cross_id), MPI_BYTE, 0,
      config.cross_comm);

    ncclComm_t new_nccl_comm;
    GUARD_NCCL2("ncclCommInitRank",
      ncclCommInitRank(&new_nccl_comm, config.global_num_nodes,
        nccl_cross_id, config.global_node_rank));
    config.nccl_cross_comm = new_nccl_comm;

    ncclUniqueId nccl_local_id;
    if (config.local_gpu_rank == 0) {
      GUARD_NCCL2("ncclGetUniqueId",
        ncclGetUniqueId(&nccl_local_id));
    }

    MPI_Bcast((void*)&nccl_local_id, sizeof(nccl_local_id), MPI_BYTE, 0,
      config.local_comm);

    ncclComm_t new_nccl_comm2;
    GUARD_NCCL2("ncclCommInitRank",
      ncclCommInitRank(&new_nccl_comm2, config.local_num_gpus,
        nccl_local_id, config.local_gpu_rank));
    config.nccl_local_comm = new_nccl_comm2;

    MPI_Barrier(config.mpi_comm);
    config.cross_comm_inited = true;
  }

  size_t num_gradients = 0;
  if (config.use_hierarchical_allreduce) {
    for (auto& layer : layers)
      num_gradients += layer.second;

    GUARD_NCCL2("ncclReduce",
      ncclReduce(input_gradients, input_gradients, num_gradients,
        gradient_type, ncclSum, 0, config.nccl_local_comm, config.stream));
  }

  if ((config.use_hierarchical_allreduce && config.local_gpu_rank == 0) ||
      !config.use_hierarchical_allreduce) {
    switch (gradient_type)
    {
    case ncclFloat32:
      retval = GradientAllReduce <float, SizeT> (
        (float*)input_gradients, (float*)output_gradients,
        //(SizeT)num_gradients, offset_map, config, state);
        layers, config, state);
      break;

    case ncclFloat64:
      retval = GradientAllReduce<double, SizeT> (
        (double*)input_gradients, (double*)output_gradients,
        //(SizeT)num_gradients, offset_map, config, state);
        layers, config, state);
      break;

    case ncclInt32:
      retval = GradientAllReduce<int32_t, SizeT> (
        (int32_t*)input_gradients, (int32_t*)output_gradients,
        //(SizeT)num_gradients, offset_map, config, state);
        layers, config, state);
      break;

    case ncclInt64:
      retval = GradientAllReduce<int64_t, SizeT> (
        (int64_t*)input_gradients, (int64_t*)output_gradients,
        //(SizeT)num_gradients, offset_map, config, state);
        layers, config, state);
      break;

    default:
      break;
    }
  }

  if (config.use_hierarchical_allreduce) {
    GUARD_NCCL2("ncclBcast",
      ncclBcast(output_gradients, num_gradients,
        gradient_type, 0, config.nccl_local_comm, config.stream));
  }
  return retval;
}

} // end of namespace dgc
} // end of namespace horovod
